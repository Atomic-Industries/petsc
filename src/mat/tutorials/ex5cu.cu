#include "hip/hip_runtime.h"
static char help[] = "Test of CUDA matrix assemble with simple matrix.\n\n";

// This a minimal example of the use of the CUDA MatAIJ metadata for assembly.
//
// The matrix must be a type 'aijcusparse' and must first be assembled on the CPU to provide the nonzero pattern.
// Next, get a pointer to a simple CSR mirror (PetscSplitCSRDataStructure) of the matrix data on
//    the GPU with MatCUSPARSEGetDeviceMatWrite().
// Then use this object to populate the matrix on the GPU with MatSetValuesDevice().
// Finally call MatAssemblyBegin/End() and the matrix is ready to use on the GPU without matrix data movement between the
//    host and GPU.

#include <petscconf.h>
#include <petscmat.h>
#include <petscaijdevice.h>
#include <petsccublas.h>
#include <assert.h>

__global__
void assemble_on_gpu(PetscSplitCSRDataStructure *d_mat, PetscInt start, PetscInt end, PetscInt Ne, PetscMPIInt rank)
{
  const PetscInt  inc = blockDim.x, my0 = threadIdx.x;
  PetscInt        i;
  PetscScalar     values[] = {1,-1,-1,3.1};
  PetscErrorCode  ierr;

  for (i=start+my0; i<end; i+=inc) {
    PetscInt js[] = {i-1, i};
    ierr = MatSetValuesDevice(d_mat,2,js,2,js,values,ADD_VALUES);if (ierr) assert(0);
  }
}

PetscErrorCode assemble_on_cpu(Mat A, PetscInt start, PetscInt end, PetscInt Ne, PetscMPIInt rank)
{
  PetscInt        i;
  PetscScalar     values[] = {1,-1,-1,1.1};
  PetscErrorCode  ierr;

  for (i=start; i<end; i++) {
    PetscInt js[] = {i-1, i};
    ierr = MatSetValues(A,2,js,2,js,values,ADD_VALUES);CHKERRQ(ierr);
  }
  return 0;
}

int main(int argc,char **args)
{
  PetscErrorCode               ierr;
  Mat                          A;
  PetscInt                     N=11, nz=3, Istart, Iend, num_threads = 128;
  PetscSplitCSRDataStructure   *d_mat;
  PetscLogEvent                event;
  Vec                          x,y;
  hipError_t                  cerr;
  PetscMPIInt                  rank;

  ierr = PetscInitialize(&argc,&args,(char*)0,help);if (ierr) return ierr;
  ierr = PetscOptionsGetInt(NULL,NULL, "-n", &N, NULL);CHKERRQ(ierr);
  ierr = PetscOptionsGetInt(NULL,NULL, "-num_threads", &num_threads, NULL);CHKERRQ(ierr);
  ierr = PetscOptionsGetInt(NULL,NULL, "-nz_row", &nz, NULL);CHKERRQ(ierr); // for debugging, will be wrong if nz<3
  if (nz>N+1) nz=N+1;
  ierr = MPI_Comm_rank(PETSC_COMM_WORLD,&rank);CHKERRQ(ierr);

  ierr = PetscLogEventRegister("GPU operator", MAT_CLASSID, &event);CHKERRQ(ierr);
  ierr = MatCreateAIJCUSPARSE(PETSC_COMM_WORLD,PETSC_DECIDE,PETSC_DECIDE,N,N,nz,NULL,nz-1,NULL,&A);CHKERRQ(ierr);
  ierr = MatSetFromOptions(A);CHKERRQ(ierr);
  ierr = MatCreateVecs(A,&x,&y);CHKERRQ(ierr);
  ierr = MatGetOwnershipRange(A,&Istart,&Iend);CHKERRQ(ierr);

  assemble_on_cpu(A, Istart, Iend, N, rank);CHKERRQ(ierr);
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);

  ierr = VecSet(x,1.0);CHKERRQ(ierr);
  ierr = MatMult(A,x,y);CHKERRQ(ierr);
  ierr = VecViewFromOptions(y,NULL,"-vec_view");CHKERRQ(ierr);

  if (Iend<N) Iend++; // elements, ignore off processor entries so do redundent
  ierr = PetscLogEventBegin(event,0,0,0,0);CHKERRQ(ierr);
  ierr = MatZeroEntries(A);CHKERRQ(ierr);
  ierr = MatCUSPARSEGetDeviceMatWrite(A,&d_mat);CHKERRQ(ierr);
  assemble_on_gpu<<<1,num_threads>>>(d_mat, Istart, Iend, N, rank);
  cerr = hipDeviceSynchronize();CHKERRCUDA(cerr);
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);

  ierr = VecSet(x,1.0);CHKERRQ(ierr);
  ierr = MatMult(A,x,y);CHKERRQ(ierr);
  ierr = VecViewFromOptions(y,NULL,"-vec_view");CHKERRQ(ierr);
  ierr = PetscLogEventEnd(event,0,0,0,0);CHKERRQ(ierr);

  ierr = MatDestroy(&A);CHKERRQ(ierr);
  ierr = VecDestroy(&x);CHKERRQ(ierr);
  ierr = VecDestroy(&y);CHKERRQ(ierr);
  ierr = PetscFinalize();
  return ierr;
}

/*TEST

   build:
      requires: cuda !define(PETSC_USE_CTABLE)

   test:
      suffix: 0
      args: -n 11 -vec_view
      nsize:  2

TEST*/
