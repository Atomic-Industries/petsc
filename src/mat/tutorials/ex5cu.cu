#include "hip/hip_runtime.h"
static char help[] = "Test of CUDA matrix assemble with simple matrix.\n\n";

// This a minimal example of the use of the CUDA MatAIJ metadata for assembly.
//
// The matrix must be a type 'aijcusparse' and must first be assembled on the CPU to provide the nonzero pattern.
// Next, get a pointer to a simple CSR mirror (PetscSplitCSRDataStructure) of the matrix data on
//    the GPU with MatCUSPARSEGetDeviceMatWrite().
// Then use this object to populate the matrix on the GPU with MatSetValuesDevice().
// Finally call MatAssemblyBegin/End() and the matrix is ready to use on the GPU without matrix data movement between the
//    host and GPU.

#include <petscconf.h>
#include <petscmat.h>
#include <petscdevice.h>
#include <assert.h>

#include <petscaijdevice.h>
__global__
void assemble_on_gpu(PetscSplitCSRDataStructure d_mat, PetscInt start, PetscInt end, PetscInt N, PetscMPIInt rank)
{
  const PetscInt  inc = blockDim.x, my0 = threadIdx.x;
  PetscInt        i;
  PetscErrorCode  ierr;

  for (i=start+my0; i<end+1; i+=inc) {
    PetscInt    js[] = {i-1, i}, nn = (i==N) ? 1 : 2; // negative indices are igored but >= N are not, so clip end
    PetscScalar values[] = {1,1,1,1};
    ierr = MatSetValuesDevice(d_mat,nn,js,nn,js,values,ADD_VALUES);if (ierr) assert(0);
  }
}

PetscErrorCode assemble_on_cpu(Mat A, PetscInt start, PetscInt end, PetscInt N, PetscMPIInt rank)
{
  PetscInt       i;
  PetscErrorCode ierr;

  PetscFunctionBeginUser;
  for (i=start; i<end+1; i++) {
    PetscInt    js[] = {i-1, i}, nn = (i==N) ? 1 : 2;
    PetscScalar values[] = {1,1,1,1};
    CHKERRQ(MatSetValues(A,nn,js,nn,js,values,ADD_VALUES));
  }
  PetscFunctionReturn(0);
}

int main(int argc,char **args)
{
  PetscErrorCode             ierr;
  Mat                        A;
  PetscInt                   N=11, nz=3, Istart, Iend, num_threads = 128;
  PetscSplitCSRDataStructure d_mat;
  PetscLogEvent              event;
  PetscMPIInt                rank,size;
  PetscBool                  testmpiseq = PETSC_FALSE;
  Vec                        x,y;

  ierr = PetscInitialize(&argc,&args,(char*)0,help);if (ierr) return ierr;
  CHKERRQ(PetscOptionsGetInt(NULL,NULL, "-n", &N, NULL));
  CHKERRQ(PetscOptionsGetInt(NULL,NULL, "-num_threads", &num_threads, NULL));
  CHKERRQ(PetscOptionsGetInt(NULL,NULL, "-nz_row", &nz, NULL));
  CHKERRQ(PetscOptionsGetBool(NULL,NULL, "-testmpiseq", &testmpiseq, NULL));
  if (nz<3)   nz=3;
  if (nz>N+1) nz=N+1;
  CHKERRMPI(MPI_Comm_rank(PETSC_COMM_WORLD,&rank));
  CHKERRMPI(MPI_Comm_size(PETSC_COMM_WORLD,&size));

  CHKERRQ(PetscLogEventRegister("GPU operator", MAT_CLASSID, &event));
  CHKERRQ(MatCreateAIJCUSPARSE(PETSC_COMM_WORLD,PETSC_DECIDE,PETSC_DECIDE,N,N,nz,NULL,nz-1,NULL,&A));
  CHKERRQ(MatSetFromOptions(A));
  CHKERRQ(MatSetOption(A,MAT_IGNORE_OFF_PROC_ENTRIES,PETSC_TRUE));
  CHKERRQ(MatCreateVecs(A,&x,&y));
  CHKERRQ(MatGetOwnershipRange(A,&Istart,&Iend));
  /* current GPU assembly code does not support offprocessor values insertion */
  CHKERRQ(assemble_on_cpu(A, Istart, Iend, N, rank));
  CHKERRQ(MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY));
  CHKERRQ(MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY));

  // test
  CHKERRQ(VecSet(x,1.0));
  CHKERRQ(MatMult(A,x,y));
  CHKERRQ(VecViewFromOptions(y,NULL,"-ex5_vec_view"));

  if (testmpiseq && size == 1) {
    CHKERRQ(MatConvert(A,MATSEQAIJ,MAT_INPLACE_MATRIX,&A));
    CHKERRQ(MatConvert(A,MATMPIAIJCUSPARSE,MAT_INPLACE_MATRIX,&A));
  }
  CHKERRQ(PetscLogEventBegin(event,0,0,0,0));
  CHKERRQ(MatCUSPARSEGetDeviceMatWrite(A,&d_mat));
  assemble_on_gpu<<<1,num_threads>>>(d_mat, Istart, Iend, N, rank);
  CHKERRCUDA(hipDeviceSynchronize());
  CHKERRQ(MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY));
  CHKERRQ(MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY));
  CHKERRQ(PetscLogEventEnd(event,0,0,0,0));

  // test
  CHKERRQ(VecSet(x,1.0));
  CHKERRQ(MatMult(A,x,y));
  CHKERRQ(VecViewFromOptions(y,NULL,"-ex5_vec_view"));

  CHKERRQ(MatDestroy(&A));
  CHKERRQ(VecDestroy(&x));
  CHKERRQ(VecDestroy(&y));
  ierr = PetscFinalize();
  return ierr;
}

/*TEST

   build:
      requires: cuda

   test:
      suffix: 0
      diff_args: -j
      args: -n 11 -ex5_vec_view
      nsize: 1

   test:
      suffix: 1
      diff_args: -j
      args: -n 11 -ex5_vec_view
      nsize: 2

   test:
      suffix: 2
      diff_args: -j
      args: -n 11 -testmpiseq -ex5_vec_view
      nsize: 1

TEST*/
