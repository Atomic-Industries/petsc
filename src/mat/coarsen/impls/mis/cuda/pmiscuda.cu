#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <petsc/private/matimpl.h>               /*I "petscmatcoarsen.h" I*/
#include <../src/mat/impls/aij/seq/aij.h>
#include <../src/mat/impls/aij/mpi/mpiaij.h>
#include <petscsf.h>
#include <petsccublas.h>

#define MIS_NOT_DONE 0
#define MIS_COARSE 1
#define MIS_FINE -1
#define MIS_APPENDED -2

__global__ void pmis_init_random(PetscInt *perm_ix,
                                 PetscInt *lid_random,
                                 PetscInt size)
{
  PetscInt lid,i;
  PetscInt global_id   = blockDim.x*blockIdx.x+threadIdx.x;
  PetscInt global_size = gridDim.x*blockDim.x;
  hiprandState state;

  hiprand_init(global_id,global_id,0,&state);
  for (i=global_id; i<size; i+=global_size) {
    lid = perm_ix[i];
    lid_random[lid] = hiprand_uniform(&state)*size;
  }
}

__global__ void pmis_init_workdata(PetscInt my0,
                                   PetscInt *lid_index,
                                   PetscInt *lid_parent_gid,
                                   PetscInt size)
{
  PetscInt i;
  PetscInt global_id   = blockDim.x*blockIdx.x+threadIdx.x;
  PetscInt global_size = gridDim.x*blockDim.x;

  for (i=global_id; i<size; i+=global_size) {
    lid_index[i] = i+my0;
    lid_parent_gid[i] = -1;
  }
}

__global__ void pmis_max_neighborhood(PetscInt const *lid_cprowID,
                                      PetscInt const *lid_state,
                                      PetscInt const *lid_random,
                                      PetscInt const *lid_index,
                                      PetscInt const *lid_state_ghosts,
                                      PetscInt const *lid_random_ghosts,
                                      PetscInt const *lid_index_ghosts,
                                      PetscInt       *lid_state2,
                                      PetscInt       *lid_random2,
                                      PetscInt       *lid_index2,
                                      PetscInt const *ii,
                                      PetscInt const *jj,
                                      PetscInt const *ii_ghosts,
                                      PetscInt const *jj_ghosts,
                                      PetscInt size)
{
  PetscInt global_id   = blockDim.x*blockIdx.x+threadIdx.x;
  PetscInt global_size = gridDim.x*blockDim.x;
  PetscInt i,j,lidj,max_state,max_random,max_index;

  for (i=global_id; i<size; i+=global_size) {
    max_state  = lid_state[i];
    max_random = lid_random[i];
    max_index  = lid_index[i];

    for (j=ii[i]; j<ii[i+1]; j++) { /* matA */
      lidj = jj[j];
      /* lexigraphical triple-max */
      if (max_state < lid_state[lidj]) {
        max_state  = lid_state[lidj];
        max_random = lid_random[lidj];
        max_index  = lid_index[lidj];
      } else if (max_state == lid_state[lidj]) {
        if (max_random < lid_random[lidj]) {
          max_state  = lid_state[lidj];
          max_random = lid_random[lidj];
          max_index  = lid_index[lidj];
        } else if (max_random == lid_random[lidj]) {
          if (max_index < lid_index[lidj]) {
            max_state  = lid_state[lidj];
            max_random = lid_random[lidj];
            max_index  = lid_index[lidj];
          }
        }
      }
    }
    if (lid_cprowID && lid_cprowID[i] != -1) {
      for (j=ii_ghosts[lid_cprowID[i]]; j<ii_ghosts[lid_cprowID[i]+1]; j++) { /* matB */
        lidj = jj_ghosts[j];
        /* lexigraphical triple-max */
        if (max_state < lid_state_ghosts[lidj]) {
          max_state  = lid_state_ghosts[lidj];
          max_random = lid_random_ghosts[lidj];
          max_index  = lid_index_ghosts[lidj];
        } else if (max_state == lid_state_ghosts[lidj]) {
          if (max_random < lid_random_ghosts[lidj]) {
            max_state  = lid_state_ghosts[lidj];
            max_random = lid_random_ghosts[lidj];
            max_index  = lid_index_ghosts[lidj];
          } else if (max_random == lid_random_ghosts[lidj]) {
            if (max_index < lid_index_ghosts[lidj]) {
              max_state  = lid_state_ghosts[lidj];
              max_random = lid_random_ghosts[lidj];
              max_index  = lid_index_ghosts[lidj];
            }
          }
        }
      }
    }
    lid_state2[i]  = max_state;
    lid_random2[i] = max_random;
    lid_index2[i]  = max_index;
  }
}

__global__ void pmis_mark_mis_nodes(PetscInt       my0,
                                    PetscInt const *lid_state2,
                                    PetscInt const *lid_index2,
                                    PetscInt       *lid_state,
                                    PetscInt       *lid_type,
                                    PetscInt       *undecided_buffer,
                                    PetscInt       size)
{
  PetscInt global_id   = blockDim.x*blockIdx.x+threadIdx.x;
  PetscInt global_size = gridDim.x*blockDim.x;
  PetscInt num_undecided = 0;
  PetscInt max_state,max_index,i;

  for (i=global_id; i<size; i+=global_size) {
    max_state = lid_state2[i];
    max_index = lid_index2[i];
    if (lid_type[i] == MIS_NOT_DONE) {
      if (i+my0 == max_index) { /* MIS node */
        lid_type[i] = MIS_COARSE;
        lid_state[i] = 1;
      } else if (max_state == 1) { /* can be removed */
        lid_type[i] = MIS_FINE;
        lid_state[i] = -1;
      } else num_undecided += 1;
    }
  }
  /* reduction of the number of undecided nodes inside a block */
  __shared__ PetscInt shared_buffer[256];
  shared_buffer[threadIdx.x] = num_undecided;
  for (PetscInt stride=blockDim.x/2; stride>0; stride/=2)
  {
    __syncthreads();
    if (threadIdx.x < stride) shared_buffer[threadIdx.x] += shared_buffer[threadIdx.x+stride];
  }
  if (threadIdx.x == 0) undecided_buffer[blockIdx.x] = shared_buffer[0];
}

__global__ void pmis_ghost_nodes_parents(PetscInt const *lid_cprowID,
                                         PetscInt const *lid_state_ghosts,
                                         PetscInt const *lid_type,
                                         PetscInt const *cpcol_gid,
                                         PetscInt       *lid_parent_gid,
                                         PetscInt const *ii_ghosts,
                                         PetscInt const *jj_ghosts,
                                         PetscInt       size)
{
  PetscInt global_id   = blockDim.x*blockIdx.x+threadIdx.x;
  PetscInt global_size = gridDim.x*blockDim.x;
  PetscInt i,j,lidj;

  for (i=global_id; i<size; i+=global_size) {
    if (lid_type[i] == MIS_FINE && lid_cprowID && lid_cprowID[i] != -1) {
      /* check the ghost neighbors */
      for (j=ii_ghosts[lid_cprowID[i]]; j<ii_ghosts[lid_cprowID[i]+1]; j++) { /* matB */
        lidj = jj_ghosts[j];
        if (lid_state_ghosts[lidj] == 1) {
          lid_parent_gid[i] = cpcol_gid[lidj];
          break;
        }
      }
    }
  }
}

/*
   maxIndSetAgg - parallel maximal independent set (MIS) with data locality info. MatAIJ specific!!!

   Input Parameter:
   . perm - serial permutation of rows of local to process in MIS
   . Gmat - glabal matrix of graph (data not defined)
   . strict_aggs - flag for whether to keep strict (non overlapping) aggregates in 'llist';

   Output Parameter:
   . a_selected - IS of selected vertices, includes 'ghost' nodes at end with natural local indices
   . a_locals_llist - array of list of nodes rooted at selected nodes
*/
PETSC_EXTERN PetscErrorCode maxIndSetAggCUDA(IS perm,Mat Gmat,PetscBool strict_aggs,PetscCoarsenData **a_locals_llist)
{
  PetscErrorCode   ierr;
  Mat_SeqAIJ       *matA,*matB=NULL;
  Mat_MPIAIJ       *mpimat=NULL;
  MPI_Comm         comm;
  PetscInt         i,j,num_fine_ghosts,iter,Iend,my0,lid,num_undecided,num_undecided2,nselected;
  PetscInt         *dev_lid_type,*lid_type;
  PetscInt         undecided_buffer[256];
  PetscInt         *lid_gid,*dev_cpcol_gid,*dev_lid_parent_gid,*dev_lid_state,*dev_lid_random,*dev_lid_index,*dev_lid_state2,*dev_lid_random2,*dev_lid_index2,*dev_tmp;
  PetscInt         *dev_lid_state_ghosts = NULL,*dev_lid_random_ghosts = NULL,*dev_lid_index_ghosts = NULL;
  PetscInt         *dev_undecided_buffer;
  PetscInt         *dev_matAi,*dev_matAj,*dev_matBi,*dev_matBj;
  PetscInt         *cpcol_gid,*lid_cprowID = NULL,*dev_lid_cprowID = NULL;
  PetscBool        isMPI,isAIJ;
  const PetscInt   *perm_ix;
  const PetscInt   nloc = Gmat->rmap->n; /* number of local points (exclude ghost points) */
  PetscCoarsenData *agg_lists;
  PetscLayout      layout;
  PetscSF          sf;
  hipError_t      cerr;

  PetscFunctionBegin;
  if (!strict_aggs) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_USER,"Not support strict_aggs=false on CUDA.");
  ierr = PetscObjectGetComm((PetscObject)Gmat,&comm);CHKERRQ(ierr);
  ierr = PetscCDCreate(nloc,&agg_lists);CHKERRQ(ierr);
  if (a_locals_llist) *a_locals_llist = agg_lists;

  /* get submatrices */
  ierr = PetscObjectBaseTypeCompare((PetscObject)Gmat,MATMPIAIJ,&isMPI);CHKERRQ(ierr);
  if (isMPI) {
    mpimat = (Mat_MPIAIJ*)Gmat->data;
    matA   = (Mat_SeqAIJ*)mpimat->A->data;
    matB   = (Mat_SeqAIJ*)mpimat->B->data;
    /* force compressed storage of B */
    ierr   = MatCheckCompressedRow(mpimat->B,matB->nonzerorowcnt,&matB->compressedrow,matB->i,Gmat->rmap->n,-1.0);CHKERRQ(ierr);
  } else {
    ierr = PetscObjectBaseTypeCompare((PetscObject)Gmat,MATSEQAIJ,&isAIJ);CHKERRQ(ierr);
    if (!isAIJ) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_USER,"Require AIJ matrix.");
    matA = (Mat_SeqAIJ*)Gmat->data;
  }
  ierr = MatGetOwnershipRange(Gmat,&my0,&Iend);CHKERRQ(ierr);
  ierr = ISGetIndices(perm, &perm_ix);CHKERRQ(ierr);
  cerr = hipMalloc((void**)&dev_matAi,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMemcpy(dev_matAi,matA->i,nloc*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_matAj,matA->nz*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMemcpy(dev_matAj,matA->j,matA->nz*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);

  ierr = PetscMalloc1(nloc,&lid_type);CHKERRQ(ierr);
  cerr = hipMalloc((void**)&dev_lid_type,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_state,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_random,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_index,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_state2,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_random2,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_index2,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMalloc((void**)&dev_lid_parent_gid,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);

  /* initialize the data */
  cerr = hipMemset(dev_lid_type,0,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMemset(dev_lid_state,0,nloc*sizeof(PetscInt));CHKERRCUDA(cerr);
  cerr = hipMemcpy(dev_lid_random,perm_ix,nloc*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  pmis_init_workdata<<<128, 128>>>(my0, 
                                   dev_lid_index,
                                   dev_lid_parent_gid,
                                   nloc
                                  );


  PetscMPIInt     rank;
  ierr = MPI_Comm_rank(comm,&rank);CHKERRQ(ierr);
  if (mpimat) {
    ierr = VecGetLocalSize(mpimat->lvec,&num_fine_ghosts);CHKERRQ(ierr);
    cerr = hipMalloc((void**)&dev_lid_state_ghosts,num_fine_ghosts*sizeof(PetscInt));CHKERRCUDA(cerr);
    cerr = hipMalloc((void**)&dev_lid_random_ghosts,num_fine_ghosts*sizeof(PetscInt));CHKERRCUDA(cerr);
    cerr = hipMalloc((void**)&dev_lid_index_ghosts,num_fine_ghosts*sizeof(PetscInt));CHKERRCUDA(cerr);
    ierr = PetscSFCreate(PetscObjectComm((PetscObject)Gmat),&sf);CHKERRQ(ierr);
    ierr = MatGetLayouts(Gmat,&layout,NULL);CHKERRQ(ierr);
    ierr = PetscSFSetGraphLayout(sf,layout,num_fine_ghosts,NULL,PETSC_COPY_VALUES,mpimat->garray);CHKERRQ(ierr);
    cerr = hipMalloc((void**)&dev_matBi,(matB->compressedrow.nrows+1)*sizeof(PetscInt));CHKERRCUDA(cerr);
    cerr = hipMalloc((void**)&dev_matBj,matB->nz*sizeof(PetscInt));CHKERRCUDA(cerr);
    cerr = hipMemcpy(dev_matBi,matB->compressedrow.i,(matB->compressedrow.nrows+1)*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    cerr = hipMemcpy(dev_matBj,matB->j,matB->nz*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);

    /* gid table is needed only for building aggregates */
    ierr = PetscMalloc1(nloc,&lid_gid);CHKERRQ(ierr);
    ierr = PetscMalloc1(num_fine_ghosts,&cpcol_gid);CHKERRQ(ierr);
    cerr = hipMalloc((void**)&dev_cpcol_gid,num_fine_ghosts*sizeof(PetscInt));CHKERRCUDA(cerr);
    for (i=0; i<nloc; i++) lid_gid[i] = i+my0;
    ierr = PetscSFBcastBegin(sf,MPIU_INT,lid_gid,cpcol_gid);CHKERRQ(ierr);
    ierr = PetscSFBcastEnd(sf,MPIU_INT,lid_gid,cpcol_gid);CHKERRQ(ierr);
    cerr = hipMemcpy(dev_cpcol_gid,cpcol_gid,num_fine_ghosts*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = PetscFree(lid_gid);CHKERRQ(ierr);
  } else num_fine_ghosts = 0;

  if (matB) {
    ierr = PetscMalloc1(nloc,&lid_cprowID);CHKERRQ(ierr);
    ierr = hipMalloc((void**)&dev_lid_cprowID,nloc*sizeof(PetscInt));CHKERRQ(ierr);
    for (i=0; i<nloc; i++) lid_cprowID[i] = -1;
    /* set index into cmpressed row 'lid_cprowID' */
    for (i=0; i<matB->compressedrow.nrows; i++) {
      lid = matB->compressedrow.rindex[i];
      lid_cprowID[lid] = i;
    }
    cerr = hipMemcpy(dev_lid_cprowID,lid_cprowID,nloc*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
  }

  cerr = hipMalloc((void**)&dev_undecided_buffer,256*sizeof(PetscInt));CHKERRCUDA(cerr);
  /* MIS */
  iter = 0;
  num_undecided2 = nloc;
  while (num_undecided2) {
    PetscInt r;
    iter++;

    if (mpimat) {
      ierr = PetscSFBcastBegin(sf,MPIU_INT,dev_lid_state,dev_lid_state_ghosts);CHKERRQ(ierr);
      ierr = PetscSFBcastEnd(sf,MPIU_INT,dev_lid_state,dev_lid_state_ghosts);CHKERRQ(ierr);
      ierr = PetscSFBcastBegin(sf,MPIU_INT,dev_lid_random,dev_lid_random_ghosts);CHKERRQ(ierr);
      ierr = PetscSFBcastEnd(sf,MPIU_INT,dev_lid_random,dev_lid_random_ghosts);CHKERRQ(ierr);
      ierr = PetscSFBcastBegin(sf,MPIU_INT,dev_lid_index,dev_lid_index_ghosts);CHKERRQ(ierr);
      ierr = PetscSFBcastEnd(sf,MPIU_INT,dev_lid_index,dev_lid_index_ghosts);CHKERRQ(ierr);
    }

    for (r=0; r<1; r++) { /* only work for MIS(1)*/
      if (r>0) {
        /* copy work array (can be fused into a single kernel if needed. Previous kernel is in most cases sufficiently heavy) */
        dev_tmp = dev_lid_state; dev_lid_state  = dev_lid_state2; dev_lid_state2 = dev_tmp;
        dev_tmp = dev_lid_random; dev_lid_random = dev_lid_random2; dev_lid_random2 = dev_tmp;
        dev_tmp = dev_lid_index; dev_lid_index  = dev_lid_index2; dev_lid_index2 = dev_tmp;
      }
      /* max operation over neighborhood */
      pmis_max_neighborhood<<<128, 128>>>(dev_lid_cprowID,
                                          dev_lid_state,
                                          dev_lid_random,
                                          dev_lid_index,
                                          dev_lid_state_ghosts,
                                          dev_lid_random_ghosts,
                                          dev_lid_index_ghosts,
                                          dev_lid_state2,
                                          dev_lid_random2,
                                          dev_lid_index2,
                                          dev_matAi,
                                          dev_matAj,
                                          dev_matBi,
                                          dev_matBj,
                                          nloc
                                         );

    }

    pmis_mark_mis_nodes<<<128, 128>>>(my0,
                                      dev_lid_state2,
                                      dev_lid_index2,
                                      dev_lid_state,
                                      dev_lid_type,
                                      dev_undecided_buffer,
                                      nloc
                                     );

    cerr = hipMemcpy(undecided_buffer,dev_undecided_buffer,256*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);

    /* reduction among all blocks */
    num_undecided = 0;
    for (i=0; i<256; i++) {
      num_undecided += undecided_buffer[i];
    }
    if (mpimat) {
      /* all done? */
      ierr = MPIU_Allreduce(&num_undecided,&num_undecided2,1,MPIU_INT,MPI_SUM,comm);CHKERRQ(ierr); /* synchronous version */
      if (!num_undecided2) break;
    } else break; /* all done */
  } /* outer parallel MIS loop */

  cerr = hipMemcpy(lid_type,dev_lid_type,nloc*sizeof(PetscInt),hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
  for (i=0; i<nloc; i++) {
    if (lid_type[i] == MIS_COARSE) {
      nselected++;
      ierr = PetscCDAppendID(agg_lists,i,i+my0);CHKERRQ(ierr);
      lid_type[i] = MIS_APPENDED;
      /* append local adjacient nodes that are not selected */
      for (j=0; j< matA->i[i+1]-matA->i[i]; j++) {
        PetscInt lidj = matA->j[matA->i[i]+j];
        if (lid_type[lidj] != MIS_APPENDED) {
          ierr = PetscCDAppendID(agg_lists,i,lidj+my0);CHKERRQ(ierr);
          lid_type[lidj] = MIS_APPENDED;
        }
      }
    }
  }

  /* tell adj who my lid_parent_gid vertices belong to - fill in agg_lists selected ghost lists */
  if (matB) {
    PetscInt *cpcol_sel_gid,sgid,gid;

    /* find the parents of the ghost nodes */
    ierr = PetscSFBcastBegin(sf,MPIU_INT,dev_lid_state,dev_lid_state_ghosts);CHKERRQ(ierr);
    ierr = PetscSFBcastEnd(sf,MPIU_INT,dev_lid_state,dev_lid_state_ghosts);CHKERRQ(ierr);
    cerr = hipMemcpy(dev_lid_type,lid_type,nloc*sizeof(PetscInt),hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    pmis_ghost_nodes_parents<<<128,128>>>(dev_lid_cprowID,
                                          dev_lid_state_ghosts,
                                          dev_lid_type,
                                          dev_cpcol_gid,
                                          dev_lid_parent_gid,
                                          dev_matBi,
                                          dev_matBj,
                                          nloc
                                         );

    ierr = PetscMalloc1(num_fine_ghosts, &cpcol_sel_gid);CHKERRQ(ierr);
    /* get proc of the ghost to be appended */
    ierr = PetscSFBcastBegin(sf,MPIU_INT,dev_lid_parent_gid,cpcol_sel_gid);CHKERRQ(ierr);
    ierr = PetscSFBcastEnd(sf,MPIU_INT,dev_lid_parent_gid,cpcol_sel_gid);CHKERRQ(ierr);
    for (i=0; i<num_fine_ghosts; i++) {
      sgid = cpcol_sel_gid[i];
      gid  = cpcol_gid[i];
      if (sgid >= my0 && sgid < Iend) {
        ierr = PetscCDAppendID(agg_lists, sgid-my0, gid);CHKERRQ(ierr);
      }
    }
    ierr = PetscFree(cpcol_sel_gid);CHKERRQ(ierr);
  }

  ierr = ISRestoreIndices(perm,&perm_ix);CHKERRQ(ierr);
  ierr = PetscInfo2(Gmat,"\t selected %D of %D vertices.\n",nselected,nloc);CHKERRQ(ierr);

  if (mpimat) {
    ierr = PetscSFDestroy(&sf);CHKERRQ(ierr);
    cerr = hipFree(dev_lid_state_ghosts);CHKERRCUDA(cerr);
    cerr = hipFree(dev_lid_random_ghosts);CHKERRCUDA(cerr);
    cerr = hipFree(dev_lid_index_ghosts);CHKERRCUDA(cerr);
    cerr = hipFree(dev_matBi);CHKERRCUDA(cerr);
    cerr = hipFree(dev_matBj);CHKERRCUDA(cerr);
    cerr = hipFree(dev_cpcol_gid);CHKERRCUDA(cerr);
    cerr = hipFree(dev_lid_parent_gid);CHKERRCUDA(cerr);
    ierr = PetscFree(cpcol_gid);CHKERRQ(ierr);
  }
  if (matB) {
    ierr = PetscFree(lid_cprowID);CHKERRQ(ierr);
    cerr = hipFree(dev_lid_cprowID);CHKERRCUDA(cerr);
  }
  ierr = PetscFree(lid_type);CHKERRQ(ierr);
  cerr = hipFree(dev_lid_type);CHKERRCUDA(cerr);
  cerr = hipFree(dev_lid_state);CHKERRCUDA(cerr);
  cerr = hipFree(dev_lid_random);CHKERRCUDA(cerr);
  cerr = hipFree(dev_lid_index);CHKERRCUDA(cerr);
  cerr = hipFree(dev_lid_state2);CHKERRCUDA(cerr);
  cerr = hipFree(dev_lid_random2);CHKERRCUDA(cerr);
  cerr = hipFree(dev_lid_index2);CHKERRCUDA(cerr);
  cerr = hipFree(dev_matAi);CHKERRCUDA(cerr);
  cerr = hipFree(dev_matAj);CHKERRCUDA(cerr);
  PetscFunctionReturn(0);
}
