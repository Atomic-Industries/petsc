#include "hip/hip_runtime.h"
/*
  Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format using the CUSPARSE library,
*/
#define PETSC_SKIP_SPINLOCK
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1

#include <petscconf.h>
#include <../src/mat/impls/aij/seq/aij.h>          /*I "petscmat.h" I*/
#include <../src/mat/impls/sbaij/seq/sbaij.h>
#include <../src/vec/vec/impls/dvecimpl.h>
#include <petsc/private/vecimpl.h>
#undef VecType
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>
#include <thrust/adjacent_difference.h>
#include <thrust/async/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

const char *const MatCUSPARSEStorageFormats[]    = {"CSR","ELL","HYB","MatCUSPARSEStorageFormat","MAT_CUSPARSE_",0};
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  /* The following are copied from hipsparse.h in CUDA-11.0. In MatCUSPARSESpMVAlgorithms[] etc, we copy them in
    0-based integer value order, since we want to use PetscOptionsEnum() to parse user command line options for them.

  typedef enum {
      HIPSPARSE_MV_ALG_DEFAULT = 0,
      HIPSPARSE_COOMV_ALG      = 1,
      HIPSPARSE_CSRMV_ALG1     = 2,
      HIPSPARSE_CSRMV_ALG2     = 3
  } hipsparseSpMVAlg_t;

  typedef enum {
      HIPSPARSE_MM_ALG_DEFAULT     CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_ALG_DEFAULT) = 0,
      HIPSPARSE_COOMM_ALG1         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_COO_ALG1)    = 1,
      HIPSPARSE_COOMM_ALG2         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_COO_ALG2)    = 2,
      HIPSPARSE_COOMM_ALG3         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_COO_ALG3)    = 3,
      HIPSPARSE_CSRMM_ALG1         CUSPARSE_DEPRECATED_ENUM(HIPSPARSE_SPMM_CSR_ALG1)    = 4,
      HIPSPARSE_SPMM_ALG_DEFAULT = 0,
      HIPSPARSE_SPMM_COO_ALG1    = 1,
      HIPSPARSE_SPMM_COO_ALG2    = 2,
      HIPSPARSE_SPMM_COO_ALG3    = 3,
      HIPSPARSE_SPMM_COO_ALG4    = 5,
      HIPSPARSE_SPMM_CSR_ALG1    = 4,
      HIPSPARSE_SPMM_CSR_ALG2    = 6,
  } hipsparseSpMMAlg_t;

  typedef enum {
      HIPSPARSE_CSR2CSC_ALG1 = 1, // faster than V2 (in general), deterministc
      HIPSPARSE_CSR2CSC_ALG2 = 2  // low memory requirement, non-deterministc
  } hipsparseCsr2CscAlg_t;
  */
  const char *const MatCUSPARSESpMVAlgorithms[]    = {"MV_ALG_DEFAULT","COOMV_ALG", "CSRMV_ALG1","CSRMV_ALG2", "hipsparseSpMVAlg_t","CUSPARSE_",0};
  const char *const MatCUSPARSESpMMAlgorithms[]    = {"ALG_DEFAULT","COO_ALG1","COO_ALG2","COO_ALG3","CSR_ALG1","COO_ALG4","CSR_ALG2","hipsparseSpMMAlg_t","CUSPARSE_SPMM_",0};
  const char *const MatCUSPARSECsr2CscAlgorithms[] = {"INVALID"/*cusparse does not have enum 0! We created one*/,"ALG1","ALG2","hipsparseCsr2CscAlg_t","CUSPARSE_CSR2CSC_",0};
#endif

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,const MatFactorInfo*);
static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo*);

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat);
static PetscErrorCode MatAXPY_SeqAIJCUSPARSE(Mat,PetscScalar,Mat,MatStructure);
static PetscErrorCode MatScale_SeqAIJCUSPARSE(Mat,PetscScalar);
static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultHermitianTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
static PetscErrorCode MatMultHermitianTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
static PetscErrorCode MatMultAddKernel_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec,PetscBool,PetscBool);

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix**);
static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct**);
static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct**,MatCUSPARSEStorageFormat);
static PetscErrorCode MatSeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors**);
static PetscErrorCode MatSeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE**);

static PetscErrorCode MatSeqAIJCUSPARSECopyFromGPU(Mat);
static PetscErrorCode MatSeqAIJCUSPARSEInvalidateTranspose(Mat,PetscBool);

static PetscErrorCode MatSeqAIJCopySubArray_SeqAIJCUSPARSE(Mat,PetscInt,const PetscInt[],PetscScalar[]);
static PetscErrorCode MatSetPreallocationCOO_SeqAIJCUSPARSE(Mat,PetscCount,const PetscInt[],const PetscInt[]);
static PetscErrorCode MatSetValuesCOO_SeqAIJCUSPARSE(Mat,const PetscScalar[],InsertMode);

PetscErrorCode MatCUSPARSESetStream(Mat A,const hipStream_t stream)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  PetscCheck(cusparsestruct,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing spptr");
  cusparsestruct->stream = stream;
  CHKERRCUSPARSE(hipsparseSetStream(cusparsestruct->handle,cusparsestruct->stream));
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSESetHandle(Mat A,const hipsparseHandle_t handle)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  PetscCheck(cusparsestruct,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing spptr");
  if (cusparsestruct->handle != handle) {
    if (cusparsestruct->handle) {
      CHKERRCUSPARSE(hipsparseDestroy(cusparsestruct->handle));
    }
    cusparsestruct->handle = handle;
  }
  CHKERRCUSPARSE(hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatCUSPARSEClearHandle(Mat A)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscBool          flg;

  PetscFunctionBegin;
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg));
  if (!flg || !cusparsestruct) PetscFunctionReturn(0);
  if (cusparsestruct->handle) cusparsestruct->handle = 0;
  PetscFunctionReturn(0);
}

PetscErrorCode MatFactorGetSolverType_seqaij_cusparse(Mat A,MatSolverType *type)
{
  PetscFunctionBegin;
  *type = MATSOLVERCUSPARSE;
  PetscFunctionReturn(0);
}

/*MC
  MATSOLVERCUSPARSE = "cusparse" - A matrix type providing triangular solvers for seq matrices
  on a single GPU of type, seqaijcusparse, aijcusparse, or seqaijcusp, aijcusp. Currently supported
  algorithms are ILU(k) and ICC(k). Typically, deeper factorizations (larger k) results in poorer
  performance in the triangular solves. Full LU, and Cholesky decompositions can be solved through the
  CUSPARSE triangular solve algorithm. However, the performance can be quite poor and thus these
  algorithms are not recommended. This class does NOT support direct solver operations.

  Level: beginner

.seealso: PCFactorSetMatSolverType(), MatSolverType, MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse(Mat A,MatFactorType ftype,Mat *B)
{
  PetscInt       n = A->rmap->n;

  PetscFunctionBegin;
  CHKERRQ(MatCreate(PetscObjectComm((PetscObject)A),B));
  CHKERRQ(MatSetSizes(*B,n,n,n,n));
  (*B)->factortype = ftype;
  CHKERRQ(MatSetType(*B,MATSEQAIJCUSPARSE));

  if (A->boundtocpu && A->bindingpropagates) CHKERRQ(MatBindToCPU(*B,PETSC_TRUE));
  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT) {
    CHKERRQ(MatSetBlockSizesFromMats(*B,A,A));
    if (!A->boundtocpu) {
      (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJCUSPARSE;
      (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJCUSPARSE;
    } else {
      (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJ;
      (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJ;
    }
    CHKERRQ(PetscStrallocpy(MATORDERINGND,(char**)&(*B)->preferredordering[MAT_FACTOR_LU]));
    CHKERRQ(PetscStrallocpy(MATORDERINGNATURAL,(char**)&(*B)->preferredordering[MAT_FACTOR_ILU]));
    CHKERRQ(PetscStrallocpy(MATORDERINGNATURAL,(char**)&(*B)->preferredordering[MAT_FACTOR_ILUDT]));
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    if (!A->boundtocpu) {
      (*B)->ops->iccfactorsymbolic      = MatICCFactorSymbolic_SeqAIJCUSPARSE;
      (*B)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqAIJCUSPARSE;
    } else {
      (*B)->ops->iccfactorsymbolic      = MatICCFactorSymbolic_SeqAIJ;
      (*B)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqAIJ;
    }
    CHKERRQ(PetscStrallocpy(MATORDERINGND,(char**)&(*B)->preferredordering[MAT_FACTOR_CHOLESKY]));
    CHKERRQ(PetscStrallocpy(MATORDERINGNATURAL,(char**)&(*B)->preferredordering[MAT_FACTOR_ICC]));
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported for CUSPARSE Matrix Types");

  CHKERRQ(MatSeqAIJSetPreallocation(*B,MAT_SKIP_ALLOCATION,NULL));
  (*B)->canuseordering = PETSC_TRUE;
  CHKERRQ(PetscObjectComposeFunction((PetscObject)(*B),"MatFactorGetSolverType_C",MatFactorGetSolverType_seqaij_cusparse));
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatCUSPARSESetFormat_SeqAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT:
    cusparsestruct->format = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparsestruct->format = format;
    break;
  default:
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);
}

/*@
   MatCUSPARSESetFormat - Sets the storage format of CUSPARSE matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for MPIAIJCUSPARSE matrices.
   Not Collective

   Input Parameters:
+  A - Matrix of type SEQAIJCUSPARSE
.  op - MatCUSPARSEFormatOperation. SEQAIJCUSPARSE matrices support MAT_CUSPARSE_MULT and MAT_CUSPARSE_ALL. MPIAIJCUSPARSE matrices support MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_OFFDIAG, and MAT_CUSPARSE_ALL.
-  format - MatCUSPARSEStorageFormat (one of MAT_CUSPARSE_CSR, MAT_CUSPARSE_ELL, MAT_CUSPARSE_HYB. The latter two require CUDA 4.2)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
@*/
PetscErrorCode MatCUSPARSESetFormat(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  CHKERRQ(PetscTryMethod(A,"MatCUSPARSESetFormat_C",(Mat,MatCUSPARSEFormatOperation,MatCUSPARSEStorageFormat),(A,op,format)));
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatCUSPARSESetUseCPUSolve_SeqAIJCUSPARSE(Mat A,PetscBool use_cpu)
{
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  cusparsestruct->use_cpu_solve = use_cpu;
  PetscFunctionReturn(0);
}

/*@
   MatCUSPARSESetUseCPUSolve - Sets use CPU MatSolve.

   Input Parameters:
+  A - Matrix of type SEQAIJCUSPARSE
-  use_cpu - set flag for using the built-in CPU MatSolve

   Output Parameter:

   Notes:
   The cuSparse LU solver currently computes the factors with the built-in CPU method
   and moves the factors to the GPU for the solve. We have observed better performance keeping the data on the CPU and computing the solve there.
   This method to specify if the solve is done on the CPU or GPU (GPU is the default).

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
@*/
PetscErrorCode MatCUSPARSESetUseCPUSolve(Mat A,PetscBool use_cpu)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  CHKERRQ(PetscTryMethod(A,"MatCUSPARSESetUseCPUSolve_C",(Mat,PetscBool),(A,use_cpu)));
  PetscFunctionReturn(0);
}

PetscErrorCode MatSetOption_SeqAIJCUSPARSE(Mat A,MatOption op,PetscBool flg)
{
  PetscFunctionBegin;
  switch (op) {
    case MAT_FORM_EXPLICIT_TRANSPOSE:
      /* need to destroy the transpose matrix if present to prevent from logic errors if flg is set to true later */
      if (A->form_explicit_transpose && !flg) CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE));
      A->form_explicit_transpose = flg;
      break;
    default:
      CHKERRQ(MatSetOption_SeqAIJ(A,op,flg));
      break;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A);

static PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             isrow = b->row,iscol = b->col;
  PetscBool      row_identity,col_identity;
  Mat_SeqAIJCUSPARSE *cusparsestruct = (Mat_SeqAIJCUSPARSE*)B->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSECopyFromGPU(A));
  CHKERRQ(MatLUFactorNumeric_SeqAIJ(B,A,info));
  B->offloadmask = PETSC_OFFLOAD_CPU;
  /* determine which version of MatSolve needs to be used. */
  CHKERRQ(ISIdentity(isrow,&row_identity));
  CHKERRQ(ISIdentity(iscol,&col_identity));
  if (row_identity && col_identity) {
    if (!cusparsestruct->use_cpu_solve) {
      B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
      B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
    }
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  } else {
    if (!cusparsestruct->use_cpu_solve) {
      B->ops->solve = MatSolve_SeqAIJCUSPARSE;
      B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
    }
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  }

  /* get the triangular factors */
  if (!cusparsestruct->use_cpu_solve) {
    CHKERRQ(MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(B));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(PetscOptionItems *PetscOptionsObject,Mat A)
{
  PetscErrorCode           ierr;
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;
  Mat_SeqAIJCUSPARSE       *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  CHKERRQ(PetscOptionsHead(PetscOptionsObject,"SeqAIJCUSPARSE options"));
  if (A->factortype == MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) CHKERRQ(MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT,format));

    ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV and TriSolve",
                            "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)cusparsestruct->format,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) CHKERRQ(MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format));
    CHKERRQ(PetscOptionsBool("-mat_cusparse_use_cpu_solve","Use CPU (I)LU solve","MatCUSPARSESetUseCPUSolve",cusparsestruct->use_cpu_solve,&cusparsestruct->use_cpu_solve,&flg));
    if (flg) CHKERRQ(MatCUSPARSESetUseCPUSolve(A,cusparsestruct->use_cpu_solve));
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    ierr = PetscOptionsEnum("-mat_cusparse_spmv_alg","sets cuSPARSE algorithm used in sparse-mat dense-vector multiplication (SpMV)",
                            "hipsparseSpMVAlg_t",MatCUSPARSESpMVAlgorithms,(PetscEnum)cusparsestruct->spmvAlg,(PetscEnum*)&cusparsestruct->spmvAlg,&flg);CHKERRQ(ierr);
    /* If user did use this option, check its consistency with cuSPARSE, since PetscOptionsEnum() sets enum values based on their position in MatCUSPARSESpMVAlgorithms[] */
#if PETSC_PKG_CUDA_VERSION_GE(11,2,0)
    PetscCheckFalse(flg && HIPSPARSE_SPMV_CSR_ALG1 != 2,PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseSpMVAlg_t has been changed but PETSc has not been updated accordingly");
#else
    PetscCheckFalse(flg && HIPSPARSE_CSRMV_ALG1 != 2,PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseSpMVAlg_t has been changed but PETSc has not been updated accordingly");
#endif
    ierr = PetscOptionsEnum("-mat_cusparse_spmm_alg","sets cuSPARSE algorithm used in sparse-mat dense-mat multiplication (SpMM)",
                            "hipsparseSpMMAlg_t",MatCUSPARSESpMMAlgorithms,(PetscEnum)cusparsestruct->spmmAlg,(PetscEnum*)&cusparsestruct->spmmAlg,&flg);CHKERRQ(ierr);
    PetscCheckFalse(flg && HIPSPARSE_SPMM_CSR_ALG1 != 4,PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseSpMMAlg_t has been changed but PETSc has not been updated accordingly");

    ierr = PetscOptionsEnum("-mat_cusparse_csr2csc_alg","sets cuSPARSE algorithm used in converting CSR matrices to CSC matrices",
                            "hipsparseCsr2CscAlg_t",MatCUSPARSECsr2CscAlgorithms,(PetscEnum)cusparsestruct->csr2cscAlg,(PetscEnum*)&cusparsestruct->csr2cscAlg,&flg);CHKERRQ(ierr);
    PetscCheckFalse(flg && HIPSPARSE_CSR2CSC_ALG1 != 1,PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE enum hipsparseCsr2CscAlg_t has been changed but PETSc has not been updated accordingly");
   #endif
  }
  CHKERRQ(PetscOptionsTail());
  PetscFunctionReturn(0);
}

static PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors));
  CHKERRQ(MatILUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info));
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors));
  CHKERRQ(MatLUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info));
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatICCFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors));
  CHKERRQ(MatICCFactorSymbolic_SeqAIJ(B,A,perm,info));
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS perm,const MatFactorInfo *info)
{
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)B->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSETriFactors_Reset(&cusparseTriFactors));
  CHKERRQ(MatCholeskyFactorSymbolic_SeqAIJ(B,A,perm,info));
  B->ops->choleskyfactornumeric = MatCholeskyFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildILULowerTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  const PetscInt                    *ai = a->i,*aj = a->j,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiLo, *AjLo;
  PetscInt                          i,nz, nzLower, offset, rowOffset;

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    try {
      /* first figure out the number of nonzeros in the lower triangular matrix including 1's on the diagonal. */
      nzLower=n+ai[n]-ai[1];
      if (!loTriFactor) {
        PetscScalar                       *AALo;

        CHKERRCUDA(hipHostMalloc((void**) &AALo, nzLower*sizeof(PetscScalar)));

        /* Allocate Space for the lower triangular matrix */
        CHKERRCUDA(hipHostMalloc((void**) &AiLo, (n+1)*sizeof(PetscInt)));
        CHKERRCUDA(hipHostMalloc((void**) &AjLo, nzLower*sizeof(PetscInt)));

        /* Fill the lower triangular matrix */
        AiLo[0]  = (PetscInt) 0;
        AiLo[n]  = nzLower;
        AjLo[0]  = (PetscInt) 0;
        AALo[0]  = (MatScalar) 1.0;
        v        = aa;
        vi       = aj;
        offset   = 1;
        rowOffset= 1;
        for (i=1; i<n; i++) {
          nz = ai[i+1] - ai[i];
          /* additional 1 for the term on the diagonal */
          AiLo[i]    = rowOffset;
          rowOffset += nz+1;

          CHKERRQ(PetscArraycpy(&(AjLo[offset]), vi, nz));
          CHKERRQ(PetscArraycpy(&(AALo[offset]), v, nz));

          offset      += nz;
          AjLo[offset] = (PetscInt) i;
          AALo[offset] = (MatScalar) 1.0;
          offset      += 1;

          v  += nz;
          vi += nz;
        }

        /* allocate space for the triangular factor information */
        CHKERRQ(PetscNew(&loTriFactor));
        loTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
        /* Create the matrix description */
        CHKERRCUSPARSE(hipsparseCreateMatDescr(&loTriFactor->descr));
        CHKERRCUSPARSE(hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO));
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
       #else
        CHKERRCUSPARSE(hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
       #endif
        CHKERRCUSPARSE(hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_LOWER));
        CHKERRCUSPARSE(hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT));

        /* set the operation */
        loTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        /* set the matrix */
        loTriFactor->csrMat = new CsrMatrix;
        loTriFactor->csrMat->num_rows = n;
        loTriFactor->csrMat->num_cols = n;
        loTriFactor->csrMat->num_entries = nzLower;

        loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
        loTriFactor->csrMat->row_offsets->assign(AiLo, AiLo+n+1);

        loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzLower);
        loTriFactor->csrMat->column_indices->assign(AjLo, AjLo+nzLower);

        loTriFactor->csrMat->values = new THRUSTARRAY(nzLower);
        loTriFactor->csrMat->values->assign(AALo, AALo+nzLower);

        /* Create the solve analysis information */
        CHKERRQ(PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0));
        CHKERRCUSPARSE(cusparse_create_analysis_info(&loTriFactor->solveInfo));
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(cusparse_get_svbuffsize(cusparseTriFactors->handle, loTriFactor->solveOp,
                                               loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                               loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                               loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo,
                                               &loTriFactor->solveBufferSize));
        CHKERRCUDA(hipMalloc(&loTriFactor->solveBuffer,loTriFactor->solveBufferSize));
      #endif

        /* perform the solve analysis */
        CHKERRCUSPARSE(cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                                         loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                         loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                         loTriFactor->csrMat->column_indices->data().get(),
                                         #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                         loTriFactor->solveInfo,
                                         loTriFactor->solvePolicy, loTriFactor->solveBuffer));
                                         #else
                                         loTriFactor->solveInfo));
                                         #endif
        CHKERRCUDA(WaitForCUDA());
        CHKERRQ(PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0));

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;
        loTriFactor->AA_h = AALo;
        CHKERRCUDA(hipHostFree(AiLo));
        CHKERRCUDA(hipHostFree(AjLo));
        CHKERRQ(PetscLogCpuToGpu((n+1+nzLower)*sizeof(int)+nzLower*sizeof(PetscScalar)));
      } else { /* update values only */
        if (!loTriFactor->AA_h) {
          CHKERRCUDA(hipHostMalloc((void**) &loTriFactor->AA_h, nzLower*sizeof(PetscScalar)));
        }
        /* Fill the lower triangular matrix */
        loTriFactor->AA_h[0]  = 1.0;
        v        = aa;
        vi       = aj;
        offset   = 1;
        for (i=1; i<n; i++) {
          nz = ai[i+1] - ai[i];
          CHKERRQ(PetscArraycpy(&(loTriFactor->AA_h[offset]), v, nz));
          offset      += nz;
          loTriFactor->AA_h[offset] = 1.0;
          offset      += 1;
          v  += nz;
        }
        loTriFactor->csrMat->values->assign(loTriFactor->AA_h, loTriFactor->AA_h+nzLower);
        CHKERRQ(PetscLogCpuToGpu(nzLower*sizeof(PetscScalar)));
      }
    } catch(char *ex) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  PetscInt                          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  const PetscInt                    *aj = a->j,*adiag = a->diag,*vi;
  const MatScalar                   *aa = a->a,*v;
  PetscInt                          *AiUp, *AjUp;
  PetscInt                          i,nz, nzUpper, offset;

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    try {
      /* next, figure out the number of nonzeros in the upper triangular matrix. */
      nzUpper = adiag[0]-adiag[n];
      if (!upTriFactor) {
        PetscScalar *AAUp;

        CHKERRCUDA(hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar)));

        /* Allocate Space for the upper triangular matrix */
        CHKERRCUDA(hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt)));
        CHKERRCUDA(hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt)));

        /* Fill the upper triangular matrix */
        AiUp[0]=(PetscInt) 0;
        AiUp[n]=nzUpper;
        offset = nzUpper;
        for (i=n-1; i>=0; i--) {
          v  = aa + adiag[i+1] + 1;
          vi = aj + adiag[i+1] + 1;

          /* number of elements NOT on the diagonal */
          nz = adiag[i] - adiag[i+1]-1;

          /* decrement the offset */
          offset -= (nz+1);

          /* first, set the diagonal elements */
          AjUp[offset] = (PetscInt) i;
          AAUp[offset] = (MatScalar)1./v[nz];
          AiUp[i]      = AiUp[i+1] - (nz+1);

          CHKERRQ(PetscArraycpy(&(AjUp[offset+1]), vi, nz));
          CHKERRQ(PetscArraycpy(&(AAUp[offset+1]), v, nz));
        }

        /* allocate space for the triangular factor information */
        CHKERRQ(PetscNew(&upTriFactor));
        upTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        /* Create the matrix description */
        CHKERRCUSPARSE(hipsparseCreateMatDescr(&upTriFactor->descr));
        CHKERRCUSPARSE(hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO));
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
       #else
        CHKERRCUSPARSE(hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
       #endif
        CHKERRCUSPARSE(hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER));
        CHKERRCUSPARSE(hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));

        /* set the operation */
        upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        /* set the matrix */
        upTriFactor->csrMat = new CsrMatrix;
        upTriFactor->csrMat->num_rows = n;
        upTriFactor->csrMat->num_cols = n;
        upTriFactor->csrMat->num_entries = nzUpper;

        upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(n+1);
        upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+n+1);

        upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(nzUpper);
        upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+nzUpper);

        upTriFactor->csrMat->values = new THRUSTARRAY(nzUpper);
        upTriFactor->csrMat->values->assign(AAUp, AAUp+nzUpper);

        /* Create the solve analysis information */
        CHKERRQ(PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0));
        CHKERRCUSPARSE(cusparse_create_analysis_info(&upTriFactor->solveInfo));
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(cusparse_get_svbuffsize(cusparseTriFactors->handle, upTriFactor->solveOp,
                                               upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                               upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                               upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo,
                                               &upTriFactor->solveBufferSize));
        CHKERRCUDA(hipMalloc(&upTriFactor->solveBuffer,upTriFactor->solveBufferSize));
      #endif

        /* perform the solve analysis */
        CHKERRCUSPARSE(cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                                         upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                         upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                         upTriFactor->csrMat->column_indices->data().get(),
                                         #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                         upTriFactor->solveInfo,
                                         upTriFactor->solvePolicy, upTriFactor->solveBuffer));
                                         #else
                                         upTriFactor->solveInfo));
                                         #endif
        CHKERRCUDA(WaitForCUDA());
        CHKERRQ(PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0));

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;
        upTriFactor->AA_h = AAUp;
        CHKERRCUDA(hipHostFree(AiUp));
        CHKERRCUDA(hipHostFree(AjUp));
        CHKERRQ(PetscLogCpuToGpu((n+1+nzUpper)*sizeof(int)+nzUpper*sizeof(PetscScalar)));
      } else {
        if (!upTriFactor->AA_h) {
          CHKERRCUDA(hipHostMalloc((void**) &upTriFactor->AA_h, nzUpper*sizeof(PetscScalar)));
        }
        /* Fill the upper triangular matrix */
        offset = nzUpper;
        for (i=n-1; i>=0; i--) {
          v  = aa + adiag[i+1] + 1;

          /* number of elements NOT on the diagonal */
          nz = adiag[i] - adiag[i+1]-1;

          /* decrement the offset */
          offset -= (nz+1);

          /* first, set the diagonal elements */
          upTriFactor->AA_h[offset] = 1./v[nz];
          CHKERRQ(PetscArraycpy(&(upTriFactor->AA_h[offset+1]), v, nz));
        }
        upTriFactor->csrMat->values->assign(upTriFactor->AA_h, upTriFactor->AA_h+nzUpper);
        CHKERRQ(PetscLogCpuToGpu(nzUpper*sizeof(PetscScalar)));
      }
    } catch(char *ex) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEILUAnalysisAndCopyToGPU(Mat A)
{
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           isrow = a->row,iscol = a->icol;
  PetscBool                    row_identity,col_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  PetscCheck(cusparseTriFactors,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
  CHKERRQ(MatSeqAIJCUSPARSEBuildILULowerTriMatrix(A));
  CHKERRQ(MatSeqAIJCUSPARSEBuildILUUpperTriMatrix(A));

  if (!cusparseTriFactors->workVector) { cusparseTriFactors->workVector = new THRUSTARRAY(n); }
  cusparseTriFactors->nnz=a->nz;

  A->offloadmask = PETSC_OFFLOAD_BOTH;
  /* lower triangular indices */
  CHKERRQ(ISIdentity(isrow,&row_identity));
  if (!row_identity && !cusparseTriFactors->rpermIndices) {
    const PetscInt *r;

    CHKERRQ(ISGetIndices(isrow,&r));
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(r, r+n);
    CHKERRQ(ISRestoreIndices(isrow,&r));
    CHKERRQ(PetscLogCpuToGpu(n*sizeof(PetscInt)));
  }

  /* upper triangular indices */
  CHKERRQ(ISIdentity(iscol,&col_identity));
  if (!col_identity && !cusparseTriFactors->cpermIndices) {
    const PetscInt *c;

    CHKERRQ(ISGetIndices(iscol,&c));
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(c, c+n);
    CHKERRQ(ISRestoreIndices(iscol,&c));
    CHKERRQ(PetscLogCpuToGpu(n*sizeof(PetscInt)));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEBuildICCTriMatrices(Mat A)
{
  Mat_SeqAIJ                        *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  PetscInt                          *AiUp, *AjUp;
  PetscScalar                       *AAUp;
  PetscScalar                       *AALo;
  PetscInt                          nzUpper = a->nz,n = A->rmap->n,i,offset,nz,j;
  Mat_SeqSBAIJ                      *b = (Mat_SeqSBAIJ*)A->data;
  const PetscInt                    *ai = b->i,*aj = b->j,*vj;
  const MatScalar                   *aa = b->a,*v;

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    try {
      CHKERRCUDA(hipHostMalloc((void**) &AAUp, nzUpper*sizeof(PetscScalar)));
      CHKERRCUDA(hipHostMalloc((void**) &AALo, nzUpper*sizeof(PetscScalar)));
      if (!upTriFactor && !loTriFactor) {
        /* Allocate Space for the upper triangular matrix */
        CHKERRCUDA(hipHostMalloc((void**) &AiUp, (n+1)*sizeof(PetscInt)));
        CHKERRCUDA(hipHostMalloc((void**) &AjUp, nzUpper*sizeof(PetscInt)));

        /* Fill the upper triangular matrix */
        AiUp[0]=(PetscInt) 0;
        AiUp[n]=nzUpper;
        offset = 0;
        for (i=0; i<n; i++) {
          /* set the pointers */
          v  = aa + ai[i];
          vj = aj + ai[i];
          nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

          /* first, set the diagonal elements */
          AjUp[offset] = (PetscInt) i;
          AAUp[offset] = (MatScalar)1.0/v[nz];
          AiUp[i]      = offset;
          AALo[offset] = (MatScalar)1.0/v[nz];

          offset+=1;
          if (nz>0) {
            CHKERRQ(PetscArraycpy(&(AjUp[offset]), vj, nz));
            CHKERRQ(PetscArraycpy(&(AAUp[offset]), v, nz));
            for (j=offset; j<offset+nz; j++) {
              AAUp[j] = -AAUp[j];
              AALo[j] = AAUp[j]/v[nz];
            }
            offset+=nz;
          }
        }

        /* allocate space for the triangular factor information */
        CHKERRQ(PetscNew(&upTriFactor));
        upTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        /* Create the matrix description */
        CHKERRCUSPARSE(hipsparseCreateMatDescr(&upTriFactor->descr));
        CHKERRCUSPARSE(hipsparseSetMatIndexBase(upTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO));
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
       #else
        CHKERRCUSPARSE(hipsparseSetMatType(upTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
       #endif
        CHKERRCUSPARSE(hipsparseSetMatFillMode(upTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER));
        CHKERRCUSPARSE(hipsparseSetMatDiagType(upTriFactor->descr, HIPSPARSE_DIAG_TYPE_UNIT));

        /* set the matrix */
        upTriFactor->csrMat = new CsrMatrix;
        upTriFactor->csrMat->num_rows = A->rmap->n;
        upTriFactor->csrMat->num_cols = A->cmap->n;
        upTriFactor->csrMat->num_entries = a->nz;

        upTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
        upTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

        upTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
        upTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

        upTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
        upTriFactor->csrMat->values->assign(AAUp, AAUp+a->nz);

        /* set the operation */
        upTriFactor->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        /* Create the solve analysis information */
        CHKERRQ(PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0));
        CHKERRCUSPARSE(cusparse_create_analysis_info(&upTriFactor->solveInfo));
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(cusparse_get_svbuffsize(cusparseTriFactors->handle, upTriFactor->solveOp,
                                               upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                               upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                               upTriFactor->csrMat->column_indices->data().get(), upTriFactor->solveInfo,
                                               &upTriFactor->solveBufferSize));
        CHKERRCUDA(hipMalloc(&upTriFactor->solveBuffer,upTriFactor->solveBufferSize));
      #endif

        /* perform the solve analysis */
        CHKERRCUSPARSE(cusparse_analysis(cusparseTriFactors->handle, upTriFactor->solveOp,
                                         upTriFactor->csrMat->num_rows, upTriFactor->csrMat->num_entries, upTriFactor->descr,
                                         upTriFactor->csrMat->values->data().get(), upTriFactor->csrMat->row_offsets->data().get(),
                                         upTriFactor->csrMat->column_indices->data().get(),
                                         #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                         upTriFactor->solveInfo,
                                         upTriFactor->solvePolicy, upTriFactor->solveBuffer));
                                         #else
                                         upTriFactor->solveInfo));
                                         #endif
        CHKERRCUDA(WaitForCUDA());
        CHKERRQ(PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0));

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = upTriFactor;

        /* allocate space for the triangular factor information */
        CHKERRQ(PetscNew(&loTriFactor));
        loTriFactor->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

        /* Create the matrix description */
        CHKERRCUSPARSE(hipsparseCreateMatDescr(&loTriFactor->descr));
        CHKERRCUSPARSE(hipsparseSetMatIndexBase(loTriFactor->descr, HIPSPARSE_INDEX_BASE_ZERO));
       #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
       #else
        CHKERRCUSPARSE(hipsparseSetMatType(loTriFactor->descr, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
       #endif
        CHKERRCUSPARSE(hipsparseSetMatFillMode(loTriFactor->descr, HIPSPARSE_FILL_MODE_UPPER));
        CHKERRCUSPARSE(hipsparseSetMatDiagType(loTriFactor->descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));

        /* set the operation */
        loTriFactor->solveOp = HIPSPARSE_OPERATION_TRANSPOSE;

        /* set the matrix */
        loTriFactor->csrMat = new CsrMatrix;
        loTriFactor->csrMat->num_rows = A->rmap->n;
        loTriFactor->csrMat->num_cols = A->cmap->n;
        loTriFactor->csrMat->num_entries = a->nz;

        loTriFactor->csrMat->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
        loTriFactor->csrMat->row_offsets->assign(AiUp, AiUp+A->rmap->n+1);

        loTriFactor->csrMat->column_indices = new THRUSTINTARRAY32(a->nz);
        loTriFactor->csrMat->column_indices->assign(AjUp, AjUp+a->nz);

        loTriFactor->csrMat->values = new THRUSTARRAY(a->nz);
        loTriFactor->csrMat->values->assign(AALo, AALo+a->nz);

        /* Create the solve analysis information */
        CHKERRQ(PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0));
        CHKERRCUSPARSE(cusparse_create_analysis_info(&loTriFactor->solveInfo));
      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
        CHKERRCUSPARSE(cusparse_get_svbuffsize(cusparseTriFactors->handle, loTriFactor->solveOp,
                                               loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                               loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                               loTriFactor->csrMat->column_indices->data().get(), loTriFactor->solveInfo,
                                               &loTriFactor->solveBufferSize));
        CHKERRCUDA(hipMalloc(&loTriFactor->solveBuffer,loTriFactor->solveBufferSize));
      #endif

        /* perform the solve analysis */
        CHKERRCUSPARSE(cusparse_analysis(cusparseTriFactors->handle, loTriFactor->solveOp,
                                         loTriFactor->csrMat->num_rows, loTriFactor->csrMat->num_entries, loTriFactor->descr,
                                         loTriFactor->csrMat->values->data().get(), loTriFactor->csrMat->row_offsets->data().get(),
                                         loTriFactor->csrMat->column_indices->data().get(),
                                         #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                         loTriFactor->solveInfo,
                                         loTriFactor->solvePolicy, loTriFactor->solveBuffer));
                                         #else
                                         loTriFactor->solveInfo));
                                         #endif
        CHKERRCUDA(WaitForCUDA());
        CHKERRQ(PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0));

        /* assign the pointer */
        ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = loTriFactor;

        CHKERRQ(PetscLogCpuToGpu(2*(((A->rmap->n+1)+(a->nz))*sizeof(int)+(a->nz)*sizeof(PetscScalar))));
        CHKERRCUDA(hipHostFree(AiUp));
        CHKERRCUDA(hipHostFree(AjUp));
      } else {
        /* Fill the upper triangular matrix */
        offset = 0;
        for (i=0; i<n; i++) {
          /* set the pointers */
          v  = aa + ai[i];
          nz = ai[i+1] - ai[i] - 1; /* exclude diag[i] */

          /* first, set the diagonal elements */
          AAUp[offset] = 1.0/v[nz];
          AALo[offset] = 1.0/v[nz];

          offset+=1;
          if (nz>0) {
            CHKERRQ(PetscArraycpy(&(AAUp[offset]), v, nz));
            for (j=offset; j<offset+nz; j++) {
              AAUp[j] = -AAUp[j];
              AALo[j] = AAUp[j]/v[nz];
            }
            offset+=nz;
          }
        }
        PetscCheck(upTriFactor,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
        PetscCheck(loTriFactor,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
        upTriFactor->csrMat->values->assign(AAUp, AAUp+a->nz);
        loTriFactor->csrMat->values->assign(AALo, AALo+a->nz);
        CHKERRQ(PetscLogCpuToGpu(2*(a->nz)*sizeof(PetscScalar)));
      }
      CHKERRCUDA(hipHostFree(AAUp));
      CHKERRCUDA(hipHostFree(AALo));
    } catch(char *ex) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(Mat A)
{
  Mat_SeqAIJ                   *a                  = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS                           ip = a->row;
  PetscBool                    perm_identity;
  PetscInt                     n = A->rmap->n;

  PetscFunctionBegin;
  PetscCheck(cusparseTriFactors,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cusparseTriFactors");
  CHKERRQ(MatSeqAIJCUSPARSEBuildICCTriMatrices(A));
  if (!cusparseTriFactors->workVector) { cusparseTriFactors->workVector = new THRUSTARRAY(n); }
  cusparseTriFactors->nnz=(a->nz-n)*2 + n;

  A->offloadmask = PETSC_OFFLOAD_BOTH;

  /* lower triangular indices */
  CHKERRQ(ISIdentity(ip,&perm_identity));
  if (!perm_identity) {
    IS             iip;
    const PetscInt *irip,*rip;

    CHKERRQ(ISInvertPermutation(ip,PETSC_DECIDE,&iip));
    CHKERRQ(ISGetIndices(iip,&irip));
    CHKERRQ(ISGetIndices(ip,&rip));
    cusparseTriFactors->rpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->rpermIndices->assign(rip, rip+n);
    cusparseTriFactors->cpermIndices = new THRUSTINTARRAY(n);
    cusparseTriFactors->cpermIndices->assign(irip, irip+n);
    CHKERRQ(ISRestoreIndices(iip,&irip));
    CHKERRQ(ISDestroy(&iip));
    CHKERRQ(ISRestoreIndices(ip,&rip));
    CHKERRQ(PetscLogCpuToGpu(2.*n*sizeof(PetscInt)));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat_SeqAIJ     *b = (Mat_SeqAIJ*)B->data;
  IS             ip = b->row;
  PetscBool      perm_identity;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSECopyFromGPU(A));
  CHKERRQ(MatCholeskyFactorNumeric_SeqAIJ(B,A,info));
  B->offloadmask = PETSC_OFFLOAD_CPU;
  /* determine which version of MatSolve needs to be used. */
  CHKERRQ(ISIdentity(ip,&perm_identity));
  if (perm_identity) {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering;
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  } else {
    B->ops->solve = MatSolve_SeqAIJCUSPARSE;
    B->ops->solvetranspose = MatSolveTranspose_SeqAIJCUSPARSE;
    B->ops->matsolve = NULL;
    B->ops->matsolvetranspose = NULL;
  }

  /* get the triangular factors */
  CHKERRQ(MatSeqAIJCUSPARSEICCAnalysisAndCopyToGPU(B));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(Mat A)
{
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT;
  hipsparseIndexBase_t               indexBase;
  hipsparseMatrixType_t              matrixType;
  hipsparseFillMode_t                fillMode;
  hipsparseDiagType_t                diagType;

  PetscFunctionBegin;
  /* allocate space for the transpose of the lower triangular factor */
  CHKERRQ(PetscNew(&loTriFactorT));
  loTriFactorT->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  /* set the matrix descriptors of the lower triangular factor */
  matrixType = hipsparseGetMatType(loTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(loTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(loTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(loTriFactor->descr);

  /* Create the matrix description */
  CHKERRCUSPARSE(hipsparseCreateMatDescr(&loTriFactorT->descr));
  CHKERRCUSPARSE(hipsparseSetMatIndexBase(loTriFactorT->descr, indexBase));
  CHKERRCUSPARSE(hipsparseSetMatType(loTriFactorT->descr, matrixType));
  CHKERRCUSPARSE(hipsparseSetMatFillMode(loTriFactorT->descr, fillMode));
  CHKERRCUSPARSE(hipsparseSetMatDiagType(loTriFactorT->descr, diagType));

  /* set the operation */
  loTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the lower triangular factor*/
  loTriFactorT->csrMat = new CsrMatrix;
  loTriFactorT->csrMat->num_rows       = loTriFactor->csrMat->num_cols;
  loTriFactorT->csrMat->num_cols       = loTriFactor->csrMat->num_rows;
  loTriFactorT->csrMat->num_entries    = loTriFactor->csrMat->num_entries;
  loTriFactorT->csrMat->row_offsets    = new THRUSTINTARRAY32(loTriFactorT->csrMat->num_rows+1);
  loTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(loTriFactorT->csrMat->num_entries);
  loTriFactorT->csrMat->values         = new THRUSTARRAY(loTriFactorT->csrMat->num_entries);

  /* compute the transpose of the lower triangular factor, i.e. the CSC */
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  CHKERRCUSPARSE(hipsparseCsr2cscEx2_bufferSize(cusparseTriFactors->handle, loTriFactor->csrMat->num_rows,
                                               loTriFactor->csrMat->num_cols, loTriFactor->csrMat->num_entries,
                                               loTriFactor->csrMat->values->data().get(),
                                               loTriFactor->csrMat->row_offsets->data().get(),
                                               loTriFactor->csrMat->column_indices->data().get(),
                                               loTriFactorT->csrMat->values->data().get(),
                                               loTriFactorT->csrMat->row_offsets->data().get(), loTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                                               HIPSPARSE_ACTION_NUMERIC,indexBase,
                                               HIPSPARSE_CSR2CSC_ALG1, &loTriFactor->csr2cscBufferSize));
  CHKERRCUDA(hipMalloc(&loTriFactor->csr2cscBuffer,loTriFactor->csr2cscBufferSize));
#endif

  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0));
  CHKERRCUSPARSE(cusparse_csr2csc(cusparseTriFactors->handle, loTriFactor->csrMat->num_rows,
                                  loTriFactor->csrMat->num_cols, loTriFactor->csrMat->num_entries,
                                  loTriFactor->csrMat->values->data().get(),
                                  loTriFactor->csrMat->row_offsets->data().get(),
                                  loTriFactor->csrMat->column_indices->data().get(),
                                  loTriFactorT->csrMat->values->data().get(),
                                  #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
                                  loTriFactorT->csrMat->row_offsets->data().get(), loTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                                  HIPSPARSE_ACTION_NUMERIC, indexBase,
                                  HIPSPARSE_CSR2CSC_ALG1, loTriFactor->csr2cscBuffer));
                                  #else
                                  loTriFactorT->csrMat->column_indices->data().get(), loTriFactorT->csrMat->row_offsets->data().get(),
                                  HIPSPARSE_ACTION_NUMERIC, indexBase));
                                  #endif
  CHKERRCUDA(WaitForCUDA());
  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0));

  /* Create the solve analysis information */
  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0));
  CHKERRCUSPARSE(cusparse_create_analysis_info(&loTriFactorT->solveInfo));
#if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
  CHKERRCUSPARSE(cusparse_get_svbuffsize(cusparseTriFactors->handle, loTriFactorT->solveOp,
                                         loTriFactorT->csrMat->num_rows, loTriFactorT->csrMat->num_entries, loTriFactorT->descr,
                                         loTriFactorT->csrMat->values->data().get(), loTriFactorT->csrMat->row_offsets->data().get(),
                                         loTriFactorT->csrMat->column_indices->data().get(), loTriFactorT->solveInfo,
                                         &loTriFactorT->solveBufferSize));
  CHKERRCUDA(hipMalloc(&loTriFactorT->solveBuffer,loTriFactorT->solveBufferSize));
#endif

  /* perform the solve analysis */
  CHKERRCUSPARSE(cusparse_analysis(cusparseTriFactors->handle, loTriFactorT->solveOp,
                                   loTriFactorT->csrMat->num_rows, loTriFactorT->csrMat->num_entries, loTriFactorT->descr,
                                   loTriFactorT->csrMat->values->data().get(), loTriFactorT->csrMat->row_offsets->data().get(),
                                   loTriFactorT->csrMat->column_indices->data().get(),
                                   #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                   loTriFactorT->solveInfo,
                                   loTriFactorT->solvePolicy, loTriFactorT->solveBuffer));
                                   #else
                                   loTriFactorT->solveInfo));
                                   #endif
  CHKERRCUDA(WaitForCUDA());
  CHKERRQ(PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0));

  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtrTranspose = loTriFactorT;

  /*********************************************/
  /* Now the Transpose of the Upper Tri Factor */
  /*********************************************/

  /* allocate space for the transpose of the upper triangular factor */
  CHKERRQ(PetscNew(&upTriFactorT));
  upTriFactorT->solvePolicy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  /* set the matrix descriptors of the upper triangular factor */
  matrixType = hipsparseGetMatType(upTriFactor->descr);
  indexBase = hipsparseGetMatIndexBase(upTriFactor->descr);
  fillMode = hipsparseGetMatFillMode(upTriFactor->descr)==HIPSPARSE_FILL_MODE_UPPER ?
    HIPSPARSE_FILL_MODE_LOWER : HIPSPARSE_FILL_MODE_UPPER;
  diagType = hipsparseGetMatDiagType(upTriFactor->descr);

  /* Create the matrix description */
  CHKERRCUSPARSE(hipsparseCreateMatDescr(&upTriFactorT->descr));
  CHKERRCUSPARSE(hipsparseSetMatIndexBase(upTriFactorT->descr, indexBase));
  CHKERRCUSPARSE(hipsparseSetMatType(upTriFactorT->descr, matrixType));
  CHKERRCUSPARSE(hipsparseSetMatFillMode(upTriFactorT->descr, fillMode));
  CHKERRCUSPARSE(hipsparseSetMatDiagType(upTriFactorT->descr, diagType));

  /* set the operation */
  upTriFactorT->solveOp = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  /* allocate GPU space for the CSC of the upper triangular factor*/
  upTriFactorT->csrMat = new CsrMatrix;
  upTriFactorT->csrMat->num_rows       = upTriFactor->csrMat->num_cols;
  upTriFactorT->csrMat->num_cols       = upTriFactor->csrMat->num_rows;
  upTriFactorT->csrMat->num_entries    = upTriFactor->csrMat->num_entries;
  upTriFactorT->csrMat->row_offsets    = new THRUSTINTARRAY32(upTriFactorT->csrMat->num_rows+1);
  upTriFactorT->csrMat->column_indices = new THRUSTINTARRAY32(upTriFactorT->csrMat->num_entries);
  upTriFactorT->csrMat->values         = new THRUSTARRAY(upTriFactorT->csrMat->num_entries);

  /* compute the transpose of the upper triangular factor, i.e. the CSC */
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  CHKERRCUSPARSE(hipsparseCsr2cscEx2_bufferSize(cusparseTriFactors->handle,upTriFactor->csrMat->num_rows,
                                               upTriFactor->csrMat->num_cols, upTriFactor->csrMat->num_entries,
                                               upTriFactor->csrMat->values->data().get(),
                                               upTriFactor->csrMat->row_offsets->data().get(),
                                               upTriFactor->csrMat->column_indices->data().get(),
                                               upTriFactorT->csrMat->values->data().get(),
                                               upTriFactorT->csrMat->row_offsets->data().get(), upTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                                               HIPSPARSE_ACTION_NUMERIC,indexBase,
                                               HIPSPARSE_CSR2CSC_ALG1, &upTriFactor->csr2cscBufferSize));
  CHKERRCUDA(hipMalloc(&upTriFactor->csr2cscBuffer,upTriFactor->csr2cscBufferSize));
#endif

  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0));
  CHKERRCUSPARSE(cusparse_csr2csc(cusparseTriFactors->handle, upTriFactor->csrMat->num_rows,
                                  upTriFactor->csrMat->num_cols, upTriFactor->csrMat->num_entries,
                                  upTriFactor->csrMat->values->data().get(),
                                  upTriFactor->csrMat->row_offsets->data().get(),
                                  upTriFactor->csrMat->column_indices->data().get(),
                                  upTriFactorT->csrMat->values->data().get(),
                                  #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
                                  upTriFactorT->csrMat->row_offsets->data().get(), upTriFactorT->csrMat->column_indices->data().get(), cusparse_scalartype,
                                  HIPSPARSE_ACTION_NUMERIC, indexBase,
                                  HIPSPARSE_CSR2CSC_ALG1, upTriFactor->csr2cscBuffer));
                                  #else
                                  upTriFactorT->csrMat->column_indices->data().get(), upTriFactorT->csrMat->row_offsets->data().get(),
                                 HIPSPARSE_ACTION_NUMERIC, indexBase));
                                 #endif

  CHKERRCUDA(WaitForCUDA());
  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0));

  /* Create the solve analysis information */
  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSESolveAnalysis,A,0,0,0));
  CHKERRCUSPARSE(cusparse_create_analysis_info(&upTriFactorT->solveInfo));
  #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
  CHKERRCUSPARSE(cusparse_get_svbuffsize(cusparseTriFactors->handle, upTriFactorT->solveOp,
                                         upTriFactorT->csrMat->num_rows, upTriFactorT->csrMat->num_entries, upTriFactorT->descr,
                                         upTriFactorT->csrMat->values->data().get(), upTriFactorT->csrMat->row_offsets->data().get(),
                                         upTriFactorT->csrMat->column_indices->data().get(), upTriFactorT->solveInfo,
                                         &upTriFactorT->solveBufferSize));
  CHKERRCUDA(hipMalloc(&upTriFactorT->solveBuffer,upTriFactorT->solveBufferSize));
  #endif

  /* perform the solve analysis */
  /* christ, would it have killed you to put this stuff in a function????????? */
  CHKERRCUSPARSE(cusparse_analysis(cusparseTriFactors->handle, upTriFactorT->solveOp,
                                   upTriFactorT->csrMat->num_rows, upTriFactorT->csrMat->num_entries, upTriFactorT->descr,
                                   upTriFactorT->csrMat->values->data().get(), upTriFactorT->csrMat->row_offsets->data().get(),
                                   upTriFactorT->csrMat->column_indices->data().get(),
                                   #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                                   upTriFactorT->solveInfo,
                                   upTriFactorT->solvePolicy, upTriFactorT->solveBuffer));
                                   #else
                                   upTriFactorT->solveInfo));
                                   #endif

  CHKERRCUDA(WaitForCUDA());
  CHKERRQ(PetscLogEventEnd(MAT_CUSPARSESolveAnalysis,A,0,0,0));

  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtrTranspose = upTriFactorT;
  PetscFunctionReturn(0);
}

struct PetscScalarToPetscInt
{
  __host__ __device__
  PetscInt operator()(PetscScalar s)
  {
    return (PetscInt)PetscRealPart(s);
  }
};

static PetscErrorCode MatSeqAIJCUSPARSEFormExplicitTranspose(Mat A)
{
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct, *matstructT;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  hipsparseStatus_t             stat;
  hipsparseIndexBase_t          indexBase;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
  PetscCheck(matstruct,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing mat struct");
  matstructT = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
  PetscCheckFalse(A->transupdated && !matstructT,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing matTranspose struct");
  if (A->transupdated) PetscFunctionReturn(0);
  CHKERRQ(PetscLogEventBegin(MAT_CUSPARSEGenerateTranspose,A,0,0,0));
  CHKERRQ(PetscLogGpuTimeBegin());
  if (cusparsestruct->format != MAT_CUSPARSE_CSR) {
    CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE));
  }
  if (!cusparsestruct->matTranspose) { /* create cusparse matrix */
    matstructT = new Mat_SeqAIJCUSPARSEMultStruct;
    CHKERRCUSPARSE(hipsparseCreateMatDescr(&matstructT->descr));
    indexBase = hipsparseGetMatIndexBase(matstruct->descr);
    CHKERRCUSPARSE(hipsparseSetMatIndexBase(matstructT->descr, indexBase));
    CHKERRCUSPARSE(hipsparseSetMatType(matstructT->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));

    /* set alpha and beta */
    CHKERRCUDA(hipMalloc((void **)&(matstructT->alpha_one),sizeof(PetscScalar)));
    CHKERRCUDA(hipMalloc((void **)&(matstructT->beta_zero),sizeof(PetscScalar)));
    CHKERRCUDA(hipMalloc((void **)&(matstructT->beta_one), sizeof(PetscScalar)));
    CHKERRCUDA(hipMemcpy(matstructT->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
    CHKERRCUDA(hipMemcpy(matstructT->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice));
    CHKERRCUDA(hipMemcpy(matstructT->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));

    if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
      CsrMatrix *matrixT = new CsrMatrix;
      matstructT->mat = matrixT;
      matrixT->num_rows = A->cmap->n;
      matrixT->num_cols = A->rmap->n;
      matrixT->num_entries = a->nz;
      matrixT->row_offsets = new THRUSTINTARRAY32(matrixT->num_rows+1);
      matrixT->column_indices = new THRUSTINTARRAY32(a->nz);
      matrixT->values = new THRUSTARRAY(a->nz);

      if (!cusparsestruct->rowoffsets_gpu) { cusparsestruct->rowoffsets_gpu = new THRUSTINTARRAY32(A->rmap->n+1); }
      cusparsestruct->rowoffsets_gpu->assign(a->i,a->i+A->rmap->n+1);

     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      #if PETSC_PKG_CUDA_VERSION_GE(11,2,1)
        stat = hipsparseCreateCsr(&matstructT->matDescr,
                               matrixT->num_rows, matrixT->num_cols, matrixT->num_entries,
                               matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(),
                               matrixT->values->data().get(),
                               HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I, /* row offset, col idx type due to THRUSTINTARRAY32 */
                               indexBase,cusparse_scalartype);CHKERRCUSPARSE(stat);
      #else
        /* cusparse-11.x returns errors with zero-sized matrices until 11.2.1,
           see https://docs.nvidia.com/cuda/cuda-toolkit-release-notes/index.html#cusparse-11.2.1

           I don't know what a proper value should be for matstructT->matDescr with empty matrices, so I just set
           it to NULL to blow it up if one relies on it. Per https://docs.nvidia.com/cuda/cusparse/index.html#csr2cscEx2,
           when nnz = 0, matrixT->row_offsets[] should be filled with indexBase. So I also set it accordingly.
        */
        if (matrixT->num_entries) {
          stat = hipsparseCreateCsr(&matstructT->matDescr,
                                 matrixT->num_rows, matrixT->num_cols, matrixT->num_entries,
                                 matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(),
                                 matrixT->values->data().get(),
                                 HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I,
                                 indexBase,cusparse_scalartype);CHKERRCUSPARSE(stat);

        } else {
          matstructT->matDescr = NULL;
          matrixT->row_offsets->assign(matrixT->row_offsets->size(),indexBase);
        }
      #endif
     #endif
    } else if (cusparsestruct->format == MAT_CUSPARSE_ELL || cusparsestruct->format == MAT_CUSPARSE_HYB) {
   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
   #else
      CsrMatrix *temp  = new CsrMatrix;
      CsrMatrix *tempT = new CsrMatrix;
      /* First convert HYB to CSR */
      temp->num_rows = A->rmap->n;
      temp->num_cols = A->cmap->n;
      temp->num_entries = a->nz;
      temp->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      temp->column_indices = new THRUSTINTARRAY32(a->nz);
      temp->values = new THRUSTARRAY(a->nz);

      stat = cusparse_hyb2csr(cusparsestruct->handle,
                              matstruct->descr, (hipsparseHybMat_t)matstruct->mat,
                              temp->values->data().get(),
                              temp->row_offsets->data().get(),
                              temp->column_indices->data().get());CHKERRCUSPARSE(stat);

      /* Next, convert CSR to CSC (i.e. the matrix transpose) */
      tempT->num_rows = A->rmap->n;
      tempT->num_cols = A->cmap->n;
      tempT->num_entries = a->nz;
      tempT->row_offsets = new THRUSTINTARRAY32(A->rmap->n+1);
      tempT->column_indices = new THRUSTINTARRAY32(a->nz);
      tempT->values = new THRUSTARRAY(a->nz);

      stat = cusparse_csr2csc(cusparsestruct->handle, temp->num_rows,
                              temp->num_cols, temp->num_entries,
                              temp->values->data().get(),
                              temp->row_offsets->data().get(),
                              temp->column_indices->data().get(),
                              tempT->values->data().get(),
                              tempT->column_indices->data().get(),
                              tempT->row_offsets->data().get(),
                              HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSPARSE(stat);

      /* Last, convert CSC to HYB */
      hipsparseHybMat_t hybMat;
      CHKERRCUSPARSE(hipsparseCreateHybMat(&hybMat));
      hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
        HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
      stat = cusparse_csr2hyb(cusparsestruct->handle, A->rmap->n, A->cmap->n,
                              matstructT->descr, tempT->values->data().get(),
                              tempT->row_offsets->data().get(),
                              tempT->column_indices->data().get(),
                              hybMat, 0, partition);CHKERRCUSPARSE(stat);

      /* assign the pointer */
      matstructT->mat = hybMat;
      A->transupdated = PETSC_TRUE;
      /* delete temporaries */
      if (tempT) {
        if (tempT->values) delete (THRUSTARRAY*) tempT->values;
        if (tempT->column_indices) delete (THRUSTINTARRAY32*) tempT->column_indices;
        if (tempT->row_offsets) delete (THRUSTINTARRAY32*) tempT->row_offsets;
        delete (CsrMatrix*) tempT;
      }
      if (temp) {
        if (temp->values) delete (THRUSTARRAY*) temp->values;
        if (temp->column_indices) delete (THRUSTINTARRAY32*) temp->column_indices;
        if (temp->row_offsets) delete (THRUSTINTARRAY32*) temp->row_offsets;
        delete (CsrMatrix*) temp;
      }
     #endif
    }
  }
  if (cusparsestruct->format == MAT_CUSPARSE_CSR) { /* transpose mat struct may be already present, update data */
    CsrMatrix *matrix  = (CsrMatrix*)matstruct->mat;
    CsrMatrix *matrixT = (CsrMatrix*)matstructT->mat;
    PetscCheck(matrix,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix");
    PetscCheck(matrix->row_offsets,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix rows");
    PetscCheck(matrix->column_indices,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix cols");
    PetscCheck(matrix->values,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrix values");
    PetscCheck(matrixT,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT");
    PetscCheck(matrixT->row_offsets,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT rows");
    PetscCheck(matrixT->column_indices,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT cols");
    PetscCheck(matrixT->values,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CsrMatrixT values");
    if (!cusparsestruct->rowoffsets_gpu) { /* this may be absent when we did not construct the transpose with csr2csc */
      cusparsestruct->rowoffsets_gpu  = new THRUSTINTARRAY32(A->rmap->n + 1);
      cusparsestruct->rowoffsets_gpu->assign(a->i,a->i + A->rmap->n + 1);
      CHKERRQ(PetscLogCpuToGpu((A->rmap->n + 1)*sizeof(PetscInt)));
    }
    if (!cusparsestruct->csr2csc_i) {
      THRUSTARRAY csr2csc_a(matrix->num_entries);
      PetscStackCallThrust(thrust::sequence(thrust::device, csr2csc_a.begin(), csr2csc_a.end(), 0.0));

      indexBase = hipsparseGetMatIndexBase(matstruct->descr);
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      void   *csr2cscBuffer;
      size_t csr2cscBufferSize;
      stat = hipsparseCsr2cscEx2_bufferSize(cusparsestruct->handle, A->rmap->n,
                                           A->cmap->n, matrix->num_entries,
                                           matrix->values->data().get(),
                                           cusparsestruct->rowoffsets_gpu->data().get(),
                                           matrix->column_indices->data().get(),
                                           matrixT->values->data().get(),
                                           matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(), cusparse_scalartype,
                                           HIPSPARSE_ACTION_NUMERIC,indexBase,
                                           cusparsestruct->csr2cscAlg, &csr2cscBufferSize);CHKERRCUSPARSE(stat);
      CHKERRCUDA(hipMalloc(&csr2cscBuffer,csr2cscBufferSize));
     #endif

      if (matrix->num_entries) {
        /* When there are no nonzeros, this routine mistakenly returns HIPSPARSE_STATUS_INVALID_VALUE in
           mat_tests-ex62_15_mpiaijcusparse on ranks 0 and 2 with CUDA-11. But CUDA-10 is OK.
           I checked every parameters and they were just fine. I have no clue why cusparse complains.

           Per https://docs.nvidia.com/cuda/cusparse/index.html#csr2cscEx2, when nnz = 0, matrixT->row_offsets[]
           should be filled with indexBase. So I just take a shortcut here.
        */
        stat = cusparse_csr2csc(cusparsestruct->handle, A->rmap->n,
                              A->cmap->n,matrix->num_entries,
                              csr2csc_a.data().get(),
                              cusparsestruct->rowoffsets_gpu->data().get(),
                              matrix->column_indices->data().get(),
                              matrixT->values->data().get(),
                             #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
                              matrixT->row_offsets->data().get(), matrixT->column_indices->data().get(), cusparse_scalartype,
                              HIPSPARSE_ACTION_NUMERIC,indexBase,
                              cusparsestruct->csr2cscAlg, csr2cscBuffer);CHKERRCUSPARSE(stat);
                             #else
                              matrixT->column_indices->data().get(), matrixT->row_offsets->data().get(),
                              HIPSPARSE_ACTION_NUMERIC, indexBase);CHKERRCUSPARSE(stat);
                             #endif
      } else {
        matrixT->row_offsets->assign(matrixT->row_offsets->size(),indexBase);
      }

      cusparsestruct->csr2csc_i = new THRUSTINTARRAY(matrix->num_entries);
      PetscStackCallThrust(thrust::transform(thrust::device,matrixT->values->begin(),matrixT->values->end(),cusparsestruct->csr2csc_i->begin(),PetscScalarToPetscInt()));
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      CHKERRCUDA(hipFree(csr2cscBuffer));
     #endif
    }
    PetscStackCallThrust(thrust::copy(thrust::device,thrust::make_permutation_iterator(matrix->values->begin(), cusparsestruct->csr2csc_i->begin()),
                                                     thrust::make_permutation_iterator(matrix->values->begin(), cusparsestruct->csr2csc_i->end()),
                                                     matrixT->values->begin()));
  }
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogEventEnd(MAT_CUSPARSEGenerateTranspose,A,0,0,0));
  /* the compressed row indices is not used for matTranspose */
  matstructT->cprowIndices = NULL;
  /* assign the pointer */
  ((Mat_SeqAIJCUSPARSE*)A->spptr)->matTranspose = matstructT;
  A->transupdated = PETSC_TRUE;
  PetscFunctionReturn(0);
}

/* Why do we need to analyze the transposed matrix again? Can't we just use op(A) = HIPSPARSE_OPERATION_TRANSPOSE in MatSolve_SeqAIJCUSPARSE? */
static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  PetscInt                              n = xx->map->n;
  const PetscScalar                     *barray;
  PetscScalar                           *xarray;
  thrust::device_ptr<const PetscScalar> bGPU;
  thrust::device_ptr<PetscScalar>       xGPU;
  hipsparseStatus_t                      stat;
  Mat_SeqAIJCUSPARSETriFactors          *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct     *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                           *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    CHKERRQ(MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A));
    loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  CHKERRQ(VecCUDAGetArrayWrite(xx,&xarray));
  CHKERRQ(VecCUDAGetArrayRead(bb,&barray));
  xGPU = thrust::device_pointer_cast(xarray);
  bGPU = thrust::device_pointer_cast(barray);

  CHKERRQ(PetscLogGpuTimeBegin());
  /* First, reorder with the row permutation */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU+n, cusparseTriFactors->rpermIndices->end()),
               xGPU);

  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        xarray,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        tempGPU->data().get(),
                        upTriFactorT->solvePolicy, upTriFactorT->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                        tempGPU->data().get());CHKERRCUSPARSE(stat);
                      #endif

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(),
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        xarray,
                        loTriFactorT->solvePolicy, loTriFactorT->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                         xarray);CHKERRCUSPARSE(stat);
                      #endif

  /* Last, copy the solution, xGPU, into a temporary with the column permutation ... can't be done in place. */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(xGPU, cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(xGPU+n, cusparseTriFactors->cpermIndices->end()),
               tempGPU->begin());

  /* Copy the temporary to the full solution. */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),tempGPU->begin(), tempGPU->end(), xGPU);

  /* restore */
  CHKERRQ(VecCUDARestoreArrayRead(bb,&barray));
  CHKERRQ(VecCUDARestoreArrayWrite(xx,&xarray));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                 *barray;
  PetscScalar                       *xarray;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactorT = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;

  PetscFunctionBegin;
  /* Analyze the matrix and create the transpose ... on the fly */
  if (!loTriFactorT && !upTriFactorT) {
    CHKERRQ(MatSeqAIJCUSPARSEAnalyzeTransposeForSolve(A));
    loTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtrTranspose;
    upTriFactorT       = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtrTranspose;
  }

  /* Get the GPU pointers */
  CHKERRQ(VecCUDAGetArrayWrite(xx,&xarray));
  CHKERRQ(VecCUDAGetArrayRead(bb,&barray));

  CHKERRQ(PetscLogGpuTimeBegin());
  /* First, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactorT->solveOp,
                        upTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactorT->descr,
                        upTriFactorT->csrMat->values->data().get(),
                        upTriFactorT->csrMat->row_offsets->data().get(),
                        upTriFactorT->csrMat->column_indices->data().get(),
                        upTriFactorT->solveInfo,
                        barray,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        tempGPU->data().get(),
                        upTriFactorT->solvePolicy, upTriFactorT->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                        tempGPU->data().get());CHKERRCUSPARSE(stat);
                      #endif

  /* Then, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactorT->solveOp,
                        loTriFactorT->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactorT->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactorT->descr,
                        loTriFactorT->csrMat->values->data().get(),
                        loTriFactorT->csrMat->row_offsets->data().get(),
                        loTriFactorT->csrMat->column_indices->data().get(),
                        loTriFactorT->solveInfo,
                        tempGPU->data().get(),
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        xarray,
                        loTriFactorT->solvePolicy, loTriFactorT->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                        xarray);CHKERRCUSPARSE(stat);
                      #endif

  /* restore */
  CHKERRQ(VecCUDARestoreArrayRead(bb,&barray));
  CHKERRQ(VecCUDARestoreArrayWrite(xx,&xarray));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                     *barray;
  PetscScalar                           *xarray;
  thrust::device_ptr<const PetscScalar> bGPU;
  thrust::device_ptr<PetscScalar>       xGPU;
  hipsparseStatus_t                      stat;
  Mat_SeqAIJCUSPARSETriFactors          *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct     *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                           *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;

  PetscFunctionBegin;

  /* Get the GPU pointers */
  CHKERRQ(VecCUDAGetArrayWrite(xx,&xarray));
  CHKERRQ(VecCUDAGetArrayRead(bb,&barray));
  xGPU = thrust::device_pointer_cast(xarray);
  bGPU = thrust::device_pointer_cast(barray);

  CHKERRQ(PetscLogGpuTimeBegin());
  /* First, reorder with the row permutation */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->begin()),
               thrust::make_permutation_iterator(bGPU, cusparseTriFactors->rpermIndices->end()),
               tempGPU->begin());

  /* Next, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        tempGPU->data().get(),
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                         xarray,
                         loTriFactor->solvePolicy, loTriFactor->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                         xarray);CHKERRCUSPARSE(stat);
                      #endif

  /* Then, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,xarray,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        tempGPU->data().get(),
                        upTriFactor->solvePolicy, upTriFactor->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                        tempGPU->data().get());CHKERRCUSPARSE(stat);
                      #endif

  /* Last, reorder with the column permutation */
  thrust::copy(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_permutation_iterator(tempGPU->begin(), cusparseTriFactors->cpermIndices->begin()),
               thrust::make_permutation_iterator(tempGPU->begin(), cusparseTriFactors->cpermIndices->end()),
               xGPU);

  CHKERRQ(VecCUDARestoreArrayRead(bb,&barray));
  CHKERRQ(VecCUDARestoreArrayWrite(xx,&xarray));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  const PetscScalar                 *barray;
  PetscScalar                       *xarray;
  hipsparseStatus_t                  stat;
  Mat_SeqAIJCUSPARSETriFactors      *cusparseTriFactors = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  Mat_SeqAIJCUSPARSETriFactorStruct *loTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->loTriFactorPtr;
  Mat_SeqAIJCUSPARSETriFactorStruct *upTriFactor = (Mat_SeqAIJCUSPARSETriFactorStruct*)cusparseTriFactors->upTriFactorPtr;
  THRUSTARRAY                       *tempGPU = (THRUSTARRAY*)cusparseTriFactors->workVector;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  CHKERRQ(VecCUDAGetArrayWrite(xx,&xarray));
  CHKERRQ(VecCUDAGetArrayRead(bb,&barray));

  CHKERRQ(PetscLogGpuTimeBegin());
  /* First, solve L */
  stat = cusparse_solve(cusparseTriFactors->handle, loTriFactor->solveOp,
                        loTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        loTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, loTriFactor->descr,
                        loTriFactor->csrMat->values->data().get(),
                        loTriFactor->csrMat->row_offsets->data().get(),
                        loTriFactor->csrMat->column_indices->data().get(),
                        loTriFactor->solveInfo,
                        barray,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        tempGPU->data().get(),
                        loTriFactor->solvePolicy,loTriFactor->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                        tempGPU->data().get());CHKERRCUSPARSE(stat);
                      #endif

  /* Next, solve U */
  stat = cusparse_solve(cusparseTriFactors->handle, upTriFactor->solveOp,
                        upTriFactor->csrMat->num_rows,
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        upTriFactor->csrMat->num_entries,
                      #endif
                        &PETSC_CUSPARSE_ONE, upTriFactor->descr,
                        upTriFactor->csrMat->values->data().get(),
                        upTriFactor->csrMat->row_offsets->data().get(),
                        upTriFactor->csrMat->column_indices->data().get(),
                        upTriFactor->solveInfo,
                        tempGPU->data().get(),
                      #if PETSC_PKG_CUDA_VERSION_GE(9,0,0)
                        xarray,
                        upTriFactor->solvePolicy, upTriFactor->solveBuffer);CHKERRCUSPARSE(stat);
                      #else
                        xarray);CHKERRCUSPARSE(stat);
                      #endif

  CHKERRQ(VecCUDARestoreArrayRead(bb,&barray));
  CHKERRQ(VecCUDARestoreArrayWrite(xx,&xarray));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(2.0*cusparseTriFactors->nnz - A->cmap->n));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSECopyFromGPU(Mat A)
{
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (A->offloadmask == PETSC_OFFLOAD_GPU) {
    CsrMatrix *matrix = (CsrMatrix*)cusp->mat->mat;

    CHKERRQ(PetscLogEventBegin(MAT_CUSPARSECopyFromGPU,A,0,0,0));
    CHKERRCUDA(hipMemcpy(a->a, matrix->values->data().get(), a->nz*sizeof(PetscScalar), hipMemcpyDeviceToHost));
    CHKERRCUDA(WaitForCUDA());
    CHKERRQ(PetscLogGpuToCpu(a->nz*sizeof(PetscScalar)));
    CHKERRQ(PetscLogEventEnd(MAT_CUSPARSECopyFromGPU,A,0,0,0));
    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJGetArray_SeqAIJCUSPARSE(Mat A,PetscScalar *array[])
{
  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSECopyFromGPU(A));
  *array = ((Mat_SeqAIJ*)A->data)->a;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJRestoreArray_SeqAIJCUSPARSE(Mat A,PetscScalar *array[])
{
  PetscFunctionBegin;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  *array         = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJGetArrayRead_SeqAIJCUSPARSE(Mat A,const PetscScalar *array[])
{
  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSECopyFromGPU(A));
  *array = ((Mat_SeqAIJ*)A->data)->a;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJRestoreArrayRead_SeqAIJCUSPARSE(Mat A,const PetscScalar *array[])
{
  PetscFunctionBegin;
  *array = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJGetArrayWrite_SeqAIJCUSPARSE(Mat A,PetscScalar *array[])
{
  PetscFunctionBegin;
  *array = ((Mat_SeqAIJ*)A->data)->a;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJRestoreArrayWrite_SeqAIJCUSPARSE(Mat A,PetscScalar *array[])
{
  PetscFunctionBegin;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  *array         = NULL;
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat A)
{
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct = cusparsestruct->mat;
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  PetscInt                     m = A->rmap->n,*ii,*ridx,tmp;
  hipsparseStatus_t             stat;
  PetscBool                    both = PETSC_TRUE;

  PetscFunctionBegin;
  PetscCheck(!A->boundtocpu,PETSC_COMM_SELF,PETSC_ERR_GPU,"Cannot copy to GPU");
  if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask == PETSC_OFFLOAD_CPU) {
    if (A->nonzerostate == cusparsestruct->nonzerostate && cusparsestruct->format == MAT_CUSPARSE_CSR) { /* Copy values only */
      CsrMatrix *matrix;
      matrix = (CsrMatrix*)cusparsestruct->mat->mat;

      PetscCheckFalse(a->nz && !a->a,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CSR values");
      CHKERRQ(PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0));
      matrix->values->assign(a->a, a->a+a->nz);
      CHKERRCUDA(WaitForCUDA());
      CHKERRQ(PetscLogCpuToGpu((a->nz)*sizeof(PetscScalar)));
      CHKERRQ(PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0));
      CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_FALSE));
    } else {
      PetscInt nnz;
      CHKERRQ(PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0));
      CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&cusparsestruct->mat,cusparsestruct->format));
      CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE));
      delete cusparsestruct->workVector;
      delete cusparsestruct->rowoffsets_gpu;
      cusparsestruct->workVector = NULL;
      cusparsestruct->rowoffsets_gpu = NULL;
      try {
        if (a->compressedrow.use) {
          m    = a->compressedrow.nrows;
          ii   = a->compressedrow.i;
          ridx = a->compressedrow.rindex;
        } else {
          m    = A->rmap->n;
          ii   = a->i;
          ridx = NULL;
        }
        PetscCheckFalse(!ii,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CSR row data");
        if (!a->a) { nnz = ii[m]; both = PETSC_FALSE; }
        else nnz = a->nz;
        PetscCheckFalse(nnz && !a->j,PETSC_COMM_SELF,PETSC_ERR_GPU,"Missing CSR column data");

        /* create cusparse matrix */
        cusparsestruct->nrows = m;
        matstruct = new Mat_SeqAIJCUSPARSEMultStruct;
        CHKERRCUSPARSE(hipsparseCreateMatDescr(&matstruct->descr));
        CHKERRCUSPARSE(hipsparseSetMatIndexBase(matstruct->descr, HIPSPARSE_INDEX_BASE_ZERO));
        CHKERRCUSPARSE(hipsparseSetMatType(matstruct->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));

        CHKERRCUDA(hipMalloc((void **)&(matstruct->alpha_one),sizeof(PetscScalar)));
        CHKERRCUDA(hipMalloc((void **)&(matstruct->beta_zero),sizeof(PetscScalar)));
        CHKERRCUDA(hipMalloc((void **)&(matstruct->beta_one), sizeof(PetscScalar)));
        CHKERRCUDA(hipMemcpy(matstruct->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
        CHKERRCUDA(hipMemcpy(matstruct->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice));
        CHKERRCUDA(hipMemcpy(matstruct->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
        CHKERRCUSPARSE(hipsparseSetPointerMode(cusparsestruct->handle, HIPSPARSE_POINTER_MODE_DEVICE));

        /* Build a hybrid/ellpack matrix if this option is chosen for the storage */
        if (cusparsestruct->format==MAT_CUSPARSE_CSR) {
          /* set the matrix */
          CsrMatrix *mat= new CsrMatrix;
          mat->num_rows = m;
          mat->num_cols = A->cmap->n;
          mat->num_entries = nnz;
          mat->row_offsets = new THRUSTINTARRAY32(m+1);
          mat->row_offsets->assign(ii, ii + m+1);

          mat->column_indices = new THRUSTINTARRAY32(nnz);
          mat->column_indices->assign(a->j, a->j+nnz);

          mat->values = new THRUSTARRAY(nnz);
          if (a->a) mat->values->assign(a->a, a->a+nnz);

          /* assign the pointer */
          matstruct->mat = mat;
         #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
          if (mat->num_rows) { /* cusparse errors on empty matrices! */
            stat = hipsparseCreateCsr(&matstruct->matDescr,
                                    mat->num_rows, mat->num_cols, mat->num_entries,
                                    mat->row_offsets->data().get(), mat->column_indices->data().get(),
                                    mat->values->data().get(),
                                    HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I, /* row offset, col idx types due to THRUSTINTARRAY32 */
                                    HIPSPARSE_INDEX_BASE_ZERO,cusparse_scalartype);CHKERRCUSPARSE(stat);
          }
         #endif
        } else if (cusparsestruct->format==MAT_CUSPARSE_ELL || cusparsestruct->format==MAT_CUSPARSE_HYB) {
         #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
          SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
         #else
          CsrMatrix *mat= new CsrMatrix;
          mat->num_rows = m;
          mat->num_cols = A->cmap->n;
          mat->num_entries = nnz;
          mat->row_offsets = new THRUSTINTARRAY32(m+1);
          mat->row_offsets->assign(ii, ii + m+1);

          mat->column_indices = new THRUSTINTARRAY32(nnz);
          mat->column_indices->assign(a->j, a->j+nnz);

          mat->values = new THRUSTARRAY(nnz);
          if (a->a) mat->values->assign(a->a, a->a+nnz);

          hipsparseHybMat_t hybMat;
          CHKERRCUSPARSE(hipsparseCreateHybMat(&hybMat));
          hipsparseHybPartition_t partition = cusparsestruct->format==MAT_CUSPARSE_ELL ?
            HIPSPARSE_HYB_PARTITION_MAX : HIPSPARSE_HYB_PARTITION_AUTO;
          stat = cusparse_csr2hyb(cusparsestruct->handle, mat->num_rows, mat->num_cols,
              matstruct->descr, mat->values->data().get(),
              mat->row_offsets->data().get(),
              mat->column_indices->data().get(),
              hybMat, 0, partition);CHKERRCUSPARSE(stat);
          /* assign the pointer */
          matstruct->mat = hybMat;

          if (mat) {
            if (mat->values) delete (THRUSTARRAY*)mat->values;
            if (mat->column_indices) delete (THRUSTINTARRAY32*)mat->column_indices;
            if (mat->row_offsets) delete (THRUSTINTARRAY32*)mat->row_offsets;
            delete (CsrMatrix*)mat;
          }
         #endif
        }

        /* assign the compressed row indices */
        if (a->compressedrow.use) {
          cusparsestruct->workVector = new THRUSTARRAY(m);
          matstruct->cprowIndices    = new THRUSTINTARRAY(m);
          matstruct->cprowIndices->assign(ridx,ridx+m);
          tmp = m;
        } else {
          cusparsestruct->workVector = NULL;
          matstruct->cprowIndices    = NULL;
          tmp = 0;
        }
        CHKERRQ(PetscLogCpuToGpu(((m+1)+(a->nz))*sizeof(int)+tmp*sizeof(PetscInt)+(3+(a->nz))*sizeof(PetscScalar)));

        /* assign the pointer */
        cusparsestruct->mat = matstruct;
      } catch(char *ex) {
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
      }
      CHKERRCUDA(WaitForCUDA());
      CHKERRQ(PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0));
      cusparsestruct->nonzerostate = A->nonzerostate;
    }
    if (both) A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

struct VecCUDAPlusEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<1>(t) + thrust::get<0>(t);
  }
};

struct VecCUDAEquals
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<0>(t);
  }
};

struct VecCUDAEqualsReverse
{
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<0>(t) = thrust::get<1>(t);
  }
};

struct MatMatCusparse {
  PetscBool             cisdense;
  PetscScalar           *Bt;
  Mat                   X;
  PetscBool             reusesym; /* Cusparse does not have split symbolic and numeric phases for sparse matmat operations */
  PetscLogDouble        flops;
  CsrMatrix             *Bcsr;

#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseSpMatDescr_t  matSpBDescr;
  PetscBool             initialized;   /* C = alpha op(A) op(B) + beta C */
  hipsparseDnMatDescr_t  matBDescr;
  hipsparseDnMatDescr_t  matCDescr;
  PetscInt              Blda,Clda; /* Record leading dimensions of B and C here to detect changes*/
 #if PETSC_PKG_CUDA_VERSION_GE(11,4,0)
  void                  *dBuffer4;
  void                  *dBuffer5;
 #endif
  size_t                mmBufferSize;
  void                  *mmBuffer;
  void                  *mmBuffer2; /* SpGEMM WorkEstimation buffer */
  hipsparseSpGEMMDescr_t spgemmDesc;
#endif
};

static PetscErrorCode MatDestroy_MatMatCusparse(void *data)
{
  MatMatCusparse *mmdata = (MatMatCusparse *)data;

  PetscFunctionBegin;
  CHKERRCUDA(hipFree(mmdata->Bt));
  delete mmdata->Bcsr;
 #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  if (mmdata->matSpBDescr) CHKERRCUSPARSE(hipsparseDestroySpMat(mmdata->matSpBDescr));
  if (mmdata->matBDescr)   CHKERRCUSPARSE(hipsparseDestroyDnMat(mmdata->matBDescr));
  if (mmdata->matCDescr)   CHKERRCUSPARSE(hipsparseDestroyDnMat(mmdata->matCDescr));
  if (mmdata->spgemmDesc)  CHKERRCUSPARSE(hipsparseSpGEMM_destroyDescr(mmdata->spgemmDesc));
 #if PETSC_PKG_CUDA_VERSION_GE(11,4,0)
  if (mmdata->dBuffer4)  CHKERRCUDA(hipFree(mmdata->dBuffer4));
  if (mmdata->dBuffer5)  CHKERRCUDA(hipFree(mmdata->dBuffer5));
 #endif
  if (mmdata->mmBuffer)  CHKERRCUDA(hipFree(mmdata->mmBuffer));
  if (mmdata->mmBuffer2) CHKERRCUDA(hipFree(mmdata->mmBuffer2));
 #endif
  CHKERRQ(MatDestroy(&mmdata->X));
  CHKERRQ(PetscFree(data));
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(Mat,Mat,Mat,PetscBool,PetscBool);

static PetscErrorCode MatProductNumeric_SeqAIJCUSPARSE_SeqDENSECUDA(Mat C)
{
  Mat_Product                  *product = C->product;
  Mat                          A,B;
  PetscInt                     m,n,blda,clda;
  PetscBool                    flg,biscuda;
  Mat_SeqAIJCUSPARSE           *cusp;
  hipsparseStatus_t             stat;
  hipsparseOperation_t          opA;
  const PetscScalar            *barray;
  PetscScalar                  *carray;
  MatMatCusparse               *mmdata;
  Mat_SeqAIJCUSPARSEMultStruct *mat;
  CsrMatrix                    *csrmat;

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  PetscCheck(C->product->data,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data empty");
  mmdata = (MatMatCusparse*)product->data;
  A    = product->A;
  B    = product->B;
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)A),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  /* currently CopyToGpu does not copy if the matrix is bound to CPU
     Instead of silently accepting the wrong answer, I prefer to raise the error */
  PetscCheck(!A->boundtocpu,PetscObjectComm((PetscObject)A),PETSC_ERR_ARG_WRONG,"Cannot bind to CPU a CUSPARSE matrix between MatProductSymbolic and MatProductNumeric phases");
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  cusp   = (Mat_SeqAIJCUSPARSE*)A->spptr;
  switch (product->type) {
  case MATPRODUCT_AB:
  case MATPRODUCT_PtAP:
    mat = cusp->mat;
    opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    m   = A->rmap->n;
    n   = B->cmap->n;
    break;
  case MATPRODUCT_AtB:
    if (!A->form_explicit_transpose) {
      mat = cusp->mat;
      opA = HIPSPARSE_OPERATION_TRANSPOSE;
    } else {
      CHKERRQ(MatSeqAIJCUSPARSEFormExplicitTranspose(A));
      mat  = cusp->matTranspose;
      opA  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    }
    m = A->cmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_ABt:
  case MATPRODUCT_RARt:
    mat = cusp->mat;
    opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    m   = A->rmap->n;
    n   = B->rmap->n;
    break;
  default:
    SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }
  PetscCheck(mat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csrmat = (CsrMatrix*)mat->mat;
  /* if the user passed a CPU matrix, copy the data to the GPU */
  CHKERRQ(PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&biscuda));
  if (!biscuda) CHKERRQ(MatConvert(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B));
  CHKERRQ(MatDenseCUDAGetArrayRead(B,&barray));

  CHKERRQ(MatDenseGetLDA(B,&blda));
  if (product->type == MATPRODUCT_RARt || product->type == MATPRODUCT_PtAP) {
    CHKERRQ(MatDenseCUDAGetArrayWrite(mmdata->X,&carray));
    CHKERRQ(MatDenseGetLDA(mmdata->X,&clda));
  } else {
    CHKERRQ(MatDenseCUDAGetArrayWrite(C,&carray));
    CHKERRQ(MatDenseGetLDA(C,&clda));
  }

  CHKERRQ(PetscLogGpuTimeBegin());
 #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseOperation_t opB = (product->type == MATPRODUCT_ABt || product->type == MATPRODUCT_RARt) ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
  /* (re)allocate mmBuffer if not initialized or LDAs are different */
  if (!mmdata->initialized || mmdata->Blda != blda || mmdata->Clda != clda) {
    size_t mmBufferSize;
    if (mmdata->initialized && mmdata->Blda != blda) {CHKERRCUSPARSE(hipsparseDestroyDnMat(mmdata->matBDescr)); mmdata->matBDescr = NULL;}
    if (!mmdata->matBDescr) {
      CHKERRCUSPARSE(hipsparseCreateDnMat(&mmdata->matBDescr,B->rmap->n,B->cmap->n,blda,(void*)barray,cusparse_scalartype,HIPSPARSE_ORDER_COL));
      mmdata->Blda = blda;
    }

    if (mmdata->initialized && mmdata->Clda != clda) {CHKERRCUSPARSE(hipsparseDestroyDnMat(mmdata->matCDescr)); mmdata->matCDescr = NULL;}
    if (!mmdata->matCDescr) { /* matCDescr is for C or mmdata->X */
      CHKERRCUSPARSE(hipsparseCreateDnMat(&mmdata->matCDescr,m,n,clda,(void*)carray,cusparse_scalartype,HIPSPARSE_ORDER_COL));
      mmdata->Clda = clda;
    }

    if (!mat->matDescr) {
      stat = hipsparseCreateCsr(&mat->matDescr,
                               csrmat->num_rows, csrmat->num_cols, csrmat->num_entries,
                               csrmat->row_offsets->data().get(), csrmat->column_indices->data().get(),
                               csrmat->values->data().get(),
                               HIPSPARSE_INDEX_32I,HIPSPARSE_INDEX_32I, /* row offset, col idx types due to THRUSTINTARRAY32 */
                               HIPSPARSE_INDEX_BASE_ZERO,cusparse_scalartype);CHKERRCUSPARSE(stat);
    }
    stat = hipsparseSpMM_bufferSize(cusp->handle,opA,opB,mat->alpha_one,
                                   mat->matDescr,mmdata->matBDescr,mat->beta_zero,
                                   mmdata->matCDescr,cusparse_scalartype,
                                   cusp->spmmAlg,&mmBufferSize);CHKERRCUSPARSE(stat);
    if ((mmdata->mmBuffer && mmdata->mmBufferSize < mmBufferSize) || !mmdata->mmBuffer) {
      CHKERRCUDA(hipFree(mmdata->mmBuffer));
      CHKERRCUDA(hipMalloc(&mmdata->mmBuffer,mmBufferSize));
      mmdata->mmBufferSize = mmBufferSize;
    }
    mmdata->initialized = PETSC_TRUE;
  } else {
    /* to be safe, always update pointers of the mats */
    CHKERRCUSPARSE(hipsparseSpMatSetValues(mat->matDescr,csrmat->values->data().get()));
    CHKERRCUSPARSE(hipsparseDnMatSetValues(mmdata->matBDescr,(void*)barray));
    CHKERRCUSPARSE(hipsparseDnMatSetValues(mmdata->matCDescr,(void*)carray));
  }

  /* do hipsparseSpMM, which supports transpose on B */
  stat = hipsparseSpMM(cusp->handle,opA,opB,mat->alpha_one,
                      mat->matDescr,mmdata->matBDescr,mat->beta_zero,
                      mmdata->matCDescr,cusparse_scalartype,
                      cusp->spmmAlg,mmdata->mmBuffer);CHKERRCUSPARSE(stat);
 #else
  PetscInt k;
  /* cusparseXcsrmm does not support transpose on B */
  if (product->type == MATPRODUCT_ABt || product->type == MATPRODUCT_RARt) {
    hipblasHandle_t cublasv2handle;
    hipblasStatus_t cerr;

    CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
    cerr = cublasXgeam(cublasv2handle,HIPBLAS_OP_T,HIPBLAS_OP_T,
                       B->cmap->n,B->rmap->n,
                       &PETSC_CUSPARSE_ONE ,barray,blda,
                       &PETSC_CUSPARSE_ZERO,barray,blda,
                       mmdata->Bt,B->cmap->n);CHKERRCUBLAS(cerr);
    blda = B->cmap->n;
    k    = B->cmap->n;
  } else {
    k    = B->rmap->n;
  }

  /* perform the MatMat operation, op(A) is m x k, op(B) is k x n */
  stat = cusparse_csr_spmm(cusp->handle,opA,m,n,k,
                           csrmat->num_entries,mat->alpha_one,mat->descr,
                           csrmat->values->data().get(),
                           csrmat->row_offsets->data().get(),
                           csrmat->column_indices->data().get(),
                           mmdata->Bt ? mmdata->Bt : barray,blda,mat->beta_zero,
                           carray,clda);CHKERRCUSPARSE(stat);
 #endif
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(n*2.0*csrmat->num_entries));
  CHKERRQ(MatDenseCUDARestoreArrayRead(B,&barray));
  if (product->type == MATPRODUCT_RARt) {
    CHKERRQ(MatDenseCUDARestoreArrayWrite(mmdata->X,&carray));
    CHKERRQ(MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(B,mmdata->X,C,PETSC_FALSE,PETSC_FALSE));
  } else if (product->type == MATPRODUCT_PtAP) {
    CHKERRQ(MatDenseCUDARestoreArrayWrite(mmdata->X,&carray));
    CHKERRQ(MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(B,mmdata->X,C,PETSC_TRUE,PETSC_FALSE));
  } else {
    CHKERRQ(MatDenseCUDARestoreArrayWrite(C,&carray));
  }
  if (mmdata->cisdense) {
    CHKERRQ(MatConvert(C,MATSEQDENSE,MAT_INPLACE_MATRIX,&C));
  }
  if (!biscuda) {
    CHKERRQ(MatConvert(B,MATSEQDENSE,MAT_INPLACE_MATRIX,&B));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatProductSymbolic_SeqAIJCUSPARSE_SeqDENSECUDA(Mat C)
{
  Mat_Product        *product = C->product;
  Mat                A,B;
  PetscInt           m,n;
  PetscBool          cisdense,flg;
  MatMatCusparse     *mmdata;
  Mat_SeqAIJCUSPARSE *cusp;

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  PetscCheck(!C->product->data,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data not empty");
  A    = product->A;
  B    = product->B;
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscCheckFalse(cusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  switch (product->type) {
  case MATPRODUCT_AB:
    m = A->rmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_AtB:
    m = A->cmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_ABt:
    m = A->rmap->n;
    n = B->rmap->n;
    break;
  case MATPRODUCT_PtAP:
    m = B->cmap->n;
    n = B->cmap->n;
    break;
  case MATPRODUCT_RARt:
    m = B->rmap->n;
    n = B->rmap->n;
    break;
  default:
    SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }
  CHKERRQ(MatSetSizes(C,m,n,m,n));
  /* if C is of type MATSEQDENSE (CPU), perform the operation on the GPU and then copy on the CPU */
  CHKERRQ(PetscObjectTypeCompare((PetscObject)C,MATSEQDENSE,&cisdense));
  CHKERRQ(MatSetType(C,MATSEQDENSECUDA));

  /* product data */
  CHKERRQ(PetscNew(&mmdata));
  mmdata->cisdense = cisdense;
 #if PETSC_PKG_CUDA_VERSION_LT(11,0,0)
  /* cusparseXcsrmm does not support transpose on B, so we allocate buffer to store B^T */
  if (product->type == MATPRODUCT_ABt || product->type == MATPRODUCT_RARt) {
    CHKERRCUDA(hipMalloc((void**)&mmdata->Bt,(size_t)B->rmap->n*(size_t)B->cmap->n*sizeof(PetscScalar)));
  }
 #endif
  /* for these products we need intermediate storage */
  if (product->type == MATPRODUCT_RARt || product->type == MATPRODUCT_PtAP) {
    CHKERRQ(MatCreate(PetscObjectComm((PetscObject)C),&mmdata->X));
    CHKERRQ(MatSetType(mmdata->X,MATSEQDENSECUDA));
    if (product->type == MATPRODUCT_RARt) { /* do not preallocate, since the first call to MatDenseCUDAGetArray will preallocate on the GPU for us */
      CHKERRQ(MatSetSizes(mmdata->X,A->rmap->n,B->rmap->n,A->rmap->n,B->rmap->n));
    } else {
      CHKERRQ(MatSetSizes(mmdata->X,A->rmap->n,B->cmap->n,A->rmap->n,B->cmap->n));
    }
  }
  C->product->data    = mmdata;
  C->product->destroy = MatDestroy_MatMatCusparse;

  C->ops->productnumeric = MatProductNumeric_SeqAIJCUSPARSE_SeqDENSECUDA;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatProductNumeric_SeqAIJCUSPARSE_SeqAIJCUSPARSE(Mat C)
{
  Mat_Product                  *product = C->product;
  Mat                          A,B;
  Mat_SeqAIJCUSPARSE           *Acusp,*Bcusp,*Ccusp;
  Mat_SeqAIJ                   *c = (Mat_SeqAIJ*)C->data;
  Mat_SeqAIJCUSPARSEMultStruct *Amat,*Bmat,*Cmat;
  CsrMatrix                    *Acsr,*Bcsr,*Ccsr;
  PetscBool                    flg;
  hipsparseStatus_t             stat;
  MatProductType               ptype;
  MatMatCusparse               *mmdata;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  hipsparseSpMatDescr_t         BmatSpDescr;
#endif
  hipsparseOperation_t          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE,opB = HIPSPARSE_OPERATION_NON_TRANSPOSE; /* cuSPARSE spgemm doesn't support transpose yet */

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  PetscCheck(C->product->data,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data empty");
  CHKERRQ(PetscObjectTypeCompare((PetscObject)C,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for C of type %s",((PetscObject)C)->type_name);
  mmdata = (MatMatCusparse*)C->product->data;
  A = product->A;
  B = product->B;
  if (mmdata->reusesym) { /* this happens when api_user is true, meaning that the matrix values have been already computed in the MatProductSymbolic phase */
    mmdata->reusesym = PETSC_FALSE;
    Ccusp = (Mat_SeqAIJCUSPARSE*)C->spptr;
    PetscCheckFalse(Ccusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
    Cmat = Ccusp->mat;
    PetscCheck(Cmat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C mult struct for product type %s",MatProductTypes[C->product->type]);
    Ccsr = (CsrMatrix*)Cmat->mat;
    PetscCheck(Ccsr,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C CSR struct");
    goto finalize;
  }
  if (!c->nz) goto finalize;
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  CHKERRQ(PetscObjectTypeCompare((PetscObject)B,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for B of type %s",((PetscObject)B)->type_name);
  PetscCheck(!A->boundtocpu,PetscObjectComm((PetscObject)C),PETSC_ERR_ARG_WRONG,"Cannot bind to CPU a CUSPARSE matrix between MatProductSymbolic and MatProductNumeric phases");
  PetscCheck(!B->boundtocpu,PetscObjectComm((PetscObject)C),PETSC_ERR_ARG_WRONG,"Cannot bind to CPU a CUSPARSE matrix between MatProductSymbolic and MatProductNumeric phases");
  Acusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Bcusp = (Mat_SeqAIJCUSPARSE*)B->spptr;
  Ccusp = (Mat_SeqAIJCUSPARSE*)C->spptr;
  PetscCheckFalse(Acusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  PetscCheckFalse(Bcusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  PetscCheckFalse(Ccusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(B));

  ptype = product->type;
  if (A->symmetric && ptype == MATPRODUCT_AtB) {
    ptype = MATPRODUCT_AB;
    PetscCheck(product->symbolic_used_the_fact_A_is_symmetric,PetscObjectComm((PetscObject)C),PETSC_ERR_PLIB,"Symbolic should have been built using the fact that A is symmetric");
  }
  if (B->symmetric && ptype == MATPRODUCT_ABt) {
    ptype = MATPRODUCT_AB;
    PetscCheck(product->symbolic_used_the_fact_B_is_symmetric,PetscObjectComm((PetscObject)C),PETSC_ERR_PLIB,"Symbolic should have been built using the fact that B is symmetric");
  }
  switch (ptype) {
  case MATPRODUCT_AB:
    Amat = Acusp->mat;
    Bmat = Bcusp->mat;
    break;
  case MATPRODUCT_AtB:
    Amat = Acusp->matTranspose;
    Bmat = Bcusp->mat;
    break;
  case MATPRODUCT_ABt:
    Amat = Acusp->mat;
    Bmat = Bcusp->matTranspose;
    break;
  default:
    SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }
  Cmat = Ccusp->mat;
  PetscCheck(Amat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A mult struct for product type %s",MatProductTypes[ptype]);
  PetscCheck(Bmat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B mult struct for product type %s",MatProductTypes[ptype]);
  PetscCheck(Cmat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C mult struct for product type %s",MatProductTypes[ptype]);
  Acsr = (CsrMatrix*)Amat->mat;
  Bcsr = mmdata->Bcsr ? mmdata->Bcsr : (CsrMatrix*)Bmat->mat; /* B may be in compressed row storage */
  Ccsr = (CsrMatrix*)Cmat->mat;
  PetscCheck(Acsr,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A CSR struct");
  PetscCheck(Bcsr,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B CSR struct");
  PetscCheck(Ccsr,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing C CSR struct");
  CHKERRQ(PetscLogGpuTimeBegin());
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  BmatSpDescr = mmdata->Bcsr ? mmdata->matSpBDescr : Bmat->matDescr; /* B may be in compressed row storage */
  CHKERRCUSPARSE(hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_DEVICE));
  #if PETSC_PKG_CUDA_VERSION_GE(11,4,0)
    stat = hipsparseSpGEMMreuse_compute(Ccusp->handle, opA, opB,
                               Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                               cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                               mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
  #else
    stat = hipsparseSpGEMM_compute(Ccusp->handle, opA, opB,
                               Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                               cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                               mmdata->spgemmDesc, &mmdata->mmBufferSize, mmdata->mmBuffer);CHKERRCUSPARSE(stat);
    stat = hipsparseSpGEMM_copy(Ccusp->handle, opA, opB,
                               Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                               cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
  #endif
#else
  stat = cusparse_csr_spgemm(Ccusp->handle, opA, opB,
                             Acsr->num_rows, Bcsr->num_cols, Acsr->num_cols,
                             Amat->descr, Acsr->num_entries, Acsr->values->data().get(), Acsr->row_offsets->data().get(), Acsr->column_indices->data().get(),
                             Bmat->descr, Bcsr->num_entries, Bcsr->values->data().get(), Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                             Cmat->descr, Ccsr->values->data().get(), Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get());CHKERRCUSPARSE(stat);
#endif
  CHKERRQ(PetscLogGpuFlops(mmdata->flops));
  CHKERRCUDA(WaitForCUDA());
  CHKERRQ(PetscLogGpuTimeEnd());
  C->offloadmask = PETSC_OFFLOAD_GPU;
finalize:
  /* shorter version of MatAssemblyEnd_SeqAIJ */
  CHKERRQ(PetscInfo(C,"Matrix size: %" PetscInt_FMT " X %" PetscInt_FMT "; storage space: 0 unneeded,%" PetscInt_FMT " used\n",C->rmap->n,C->cmap->n,c->nz));
  CHKERRQ(PetscInfo(C,"Number of mallocs during MatSetValues() is 0\n"));
  CHKERRQ(PetscInfo(C,"Maximum nonzeros in any row is %" PetscInt_FMT "\n",c->rmax));
  c->reallocs         = 0;
  C->info.mallocs    += 0;
  C->info.nz_unneeded = 0;
  C->assembled = C->was_assembled = PETSC_TRUE;
  C->num_ass++;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatProductSymbolic_SeqAIJCUSPARSE_SeqAIJCUSPARSE(Mat C)
{
  Mat_Product                  *product = C->product;
  Mat                          A,B;
  Mat_SeqAIJCUSPARSE           *Acusp,*Bcusp,*Ccusp;
  Mat_SeqAIJ                   *a,*b,*c;
  Mat_SeqAIJCUSPARSEMultStruct *Amat,*Bmat,*Cmat;
  CsrMatrix                    *Acsr,*Bcsr,*Ccsr;
  PetscInt                     i,j,m,n,k;
  PetscBool                    flg;
  hipsparseStatus_t             stat;
  MatProductType               ptype;
  MatMatCusparse               *mmdata;
  PetscLogDouble               flops;
  PetscBool                    biscompressed,ciscompressed;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  int64_t                      C_num_rows1, C_num_cols1, C_nnz1;
  hipsparseSpMatDescr_t         BmatSpDescr;
#else
  int                          cnz;
#endif
  hipsparseOperation_t          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE,opB = HIPSPARSE_OPERATION_NON_TRANSPOSE; /* cuSPARSE spgemm doesn't support transpose yet */

  PetscFunctionBegin;
  MatCheckProduct(C,1);
  PetscCheck(!C->product->data,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Product data not empty");
  A    = product->A;
  B    = product->B;
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for type %s",((PetscObject)A)->type_name);
  CHKERRQ(PetscObjectTypeCompare((PetscObject)B,MATSEQAIJCUSPARSE,&flg));
  PetscCheck(flg,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Not for B of type %s",((PetscObject)B)->type_name);
  a = (Mat_SeqAIJ*)A->data;
  b = (Mat_SeqAIJ*)B->data;
  /* product data */
  CHKERRQ(PetscNew(&mmdata));
  C->product->data    = mmdata;
  C->product->destroy = MatDestroy_MatMatCusparse;

  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(B));
  Acusp = (Mat_SeqAIJCUSPARSE*)A->spptr; /* Access spptr after MatSeqAIJCUSPARSECopyToGPU, not before */
  Bcusp = (Mat_SeqAIJCUSPARSE*)B->spptr;
  PetscCheckFalse(Acusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");
  PetscCheckFalse(Bcusp->format != MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Only for MAT_CUSPARSE_CSR format");

  ptype = product->type;
  if (A->symmetric && ptype == MATPRODUCT_AtB) {
    ptype = MATPRODUCT_AB;
    product->symbolic_used_the_fact_A_is_symmetric = PETSC_TRUE;
  }
  if (B->symmetric && ptype == MATPRODUCT_ABt) {
    ptype = MATPRODUCT_AB;
    product->symbolic_used_the_fact_B_is_symmetric = PETSC_TRUE;
  }
  biscompressed = PETSC_FALSE;
  ciscompressed = PETSC_FALSE;
  switch (ptype) {
  case MATPRODUCT_AB:
    m = A->rmap->n;
    n = B->cmap->n;
    k = A->cmap->n;
    Amat = Acusp->mat;
    Bmat = Bcusp->mat;
    if (a->compressedrow.use) ciscompressed = PETSC_TRUE;
    if (b->compressedrow.use) biscompressed = PETSC_TRUE;
    break;
  case MATPRODUCT_AtB:
    m = A->cmap->n;
    n = B->cmap->n;
    k = A->rmap->n;
    CHKERRQ(MatSeqAIJCUSPARSEFormExplicitTranspose(A));
    Amat = Acusp->matTranspose;
    Bmat = Bcusp->mat;
    if (b->compressedrow.use) biscompressed = PETSC_TRUE;
    break;
  case MATPRODUCT_ABt:
    m = A->rmap->n;
    n = B->rmap->n;
    k = A->cmap->n;
    CHKERRQ(MatSeqAIJCUSPARSEFormExplicitTranspose(B));
    Amat = Acusp->mat;
    Bmat = Bcusp->matTranspose;
    if (a->compressedrow.use) ciscompressed = PETSC_TRUE;
    break;
  default:
    SETERRQ(PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Unsupported product type %s",MatProductTypes[product->type]);
  }

  /* create cusparse matrix */
  CHKERRQ(MatSetSizes(C,m,n,m,n));
  CHKERRQ(MatSetType(C,MATSEQAIJCUSPARSE));
  c     = (Mat_SeqAIJ*)C->data;
  Ccusp = (Mat_SeqAIJCUSPARSE*)C->spptr;
  Cmat  = new Mat_SeqAIJCUSPARSEMultStruct;
  Ccsr  = new CsrMatrix;

  c->compressedrow.use = ciscompressed;
  if (c->compressedrow.use) { /* if a is in compressed row, than c will be in compressed row format */
    c->compressedrow.nrows = a->compressedrow.nrows;
    CHKERRQ(PetscMalloc2(c->compressedrow.nrows+1,&c->compressedrow.i,c->compressedrow.nrows,&c->compressedrow.rindex));
    CHKERRQ(PetscArraycpy(c->compressedrow.rindex,a->compressedrow.rindex,c->compressedrow.nrows));
    Ccusp->workVector  = new THRUSTARRAY(c->compressedrow.nrows);
    Cmat->cprowIndices = new THRUSTINTARRAY(c->compressedrow.nrows);
    Cmat->cprowIndices->assign(c->compressedrow.rindex,c->compressedrow.rindex + c->compressedrow.nrows);
  } else {
    c->compressedrow.nrows  = 0;
    c->compressedrow.i      = NULL;
    c->compressedrow.rindex = NULL;
    Ccusp->workVector       = NULL;
    Cmat->cprowIndices      = NULL;
  }
  Ccusp->nrows    = ciscompressed ? c->compressedrow.nrows : m;
  Ccusp->mat      = Cmat;
  Ccusp->mat->mat = Ccsr;
  Ccsr->num_rows    = Ccusp->nrows;
  Ccsr->num_cols    = n;
  Ccsr->row_offsets = new THRUSTINTARRAY32(Ccusp->nrows+1);
  CHKERRCUSPARSE(hipsparseCreateMatDescr(&Cmat->descr));
  CHKERRCUSPARSE(hipsparseSetMatIndexBase(Cmat->descr, HIPSPARSE_INDEX_BASE_ZERO));
  CHKERRCUSPARSE(hipsparseSetMatType(Cmat->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CHKERRCUDA(hipMalloc((void **)&(Cmat->alpha_one),sizeof(PetscScalar)));
  CHKERRCUDA(hipMalloc((void **)&(Cmat->beta_zero),sizeof(PetscScalar)));
  CHKERRCUDA(hipMalloc((void **)&(Cmat->beta_one), sizeof(PetscScalar)));
  CHKERRCUDA(hipMemcpy(Cmat->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
  CHKERRCUDA(hipMemcpy(Cmat->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice));
  CHKERRCUDA(hipMemcpy(Cmat->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
  if (!Ccsr->num_rows || !Ccsr->num_cols || !a->nz || !b->nz) { /* cusparse raise errors in different calls when matrices have zero rows/columns! */
    thrust::fill(thrust::device,Ccsr->row_offsets->begin(),Ccsr->row_offsets->end(),0);
    c->nz = 0;
    Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
    Ccsr->values = new THRUSTARRAY(c->nz);
    goto finalizesym;
  }

  PetscCheck(Amat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A mult struct for product type %s",MatProductTypes[ptype]);
  PetscCheck(Bmat,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B mult struct for product type %s",MatProductTypes[ptype]);
  Acsr = (CsrMatrix*)Amat->mat;
  if (!biscompressed) {
    Bcsr = (CsrMatrix*)Bmat->mat;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    BmatSpDescr = Bmat->matDescr;
#endif
  } else { /* we need to use row offsets for the full matrix */
    CsrMatrix *cBcsr = (CsrMatrix*)Bmat->mat;
    Bcsr = new CsrMatrix;
    Bcsr->num_rows       = B->rmap->n;
    Bcsr->num_cols       = cBcsr->num_cols;
    Bcsr->num_entries    = cBcsr->num_entries;
    Bcsr->column_indices = cBcsr->column_indices;
    Bcsr->values         = cBcsr->values;
    if (!Bcusp->rowoffsets_gpu) {
      Bcusp->rowoffsets_gpu  = new THRUSTINTARRAY32(B->rmap->n + 1);
      Bcusp->rowoffsets_gpu->assign(b->i,b->i + B->rmap->n + 1);
      CHKERRQ(PetscLogCpuToGpu((B->rmap->n + 1)*sizeof(PetscInt)));
    }
    Bcsr->row_offsets = Bcusp->rowoffsets_gpu;
    mmdata->Bcsr = Bcsr;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    if (Bcsr->num_rows && Bcsr->num_cols) {
      stat = hipsparseCreateCsr(&mmdata->matSpBDescr, Bcsr->num_rows, Bcsr->num_cols, Bcsr->num_entries,
                               Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                               Bcsr->values->data().get(),
                               HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                               HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
    }
    BmatSpDescr = mmdata->matSpBDescr;
#endif
  }
  PetscCheck(Acsr,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing A CSR struct");
  PetscCheck(Bcsr,PetscObjectComm((PetscObject)C),PETSC_ERR_GPU,"Missing B CSR struct");
  /* precompute flops count */
  if (ptype == MATPRODUCT_AB) {
    for (i=0, flops = 0; i<A->rmap->n; i++) {
      const PetscInt st = a->i[i];
      const PetscInt en = a->i[i+1];
      for (j=st; j<en; j++) {
        const PetscInt brow = a->j[j];
        flops += 2.*(b->i[brow+1] - b->i[brow]);
      }
    }
  } else if (ptype == MATPRODUCT_AtB) {
    for (i=0, flops = 0; i<A->rmap->n; i++) {
      const PetscInt anzi = a->i[i+1] - a->i[i];
      const PetscInt bnzi = b->i[i+1] - b->i[i];
      flops += (2.*anzi)*bnzi;
    }
  } else { /* TODO */
    flops = 0.;
  }

  mmdata->flops = flops;
  CHKERRQ(PetscLogGpuTimeBegin());

#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  CHKERRCUSPARSE(hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_DEVICE));
  stat = hipsparseCreateCsr(&Cmat->matDescr, Ccsr->num_rows, Ccsr->num_cols, 0,
                          NULL, NULL, NULL,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
  CHKERRCUSPARSE(hipsparseSpGEMM_createDescr(&mmdata->spgemmDesc));
 #if PETSC_PKG_CUDA_VERSION_GE(11,4,0)
 {
  /* cusparseSpGEMMreuse has more reasonable APIs than cusparseSpGEMM, so we prefer to use it.
     We follow the sample code at https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/spgemm_reuse
  */
  void*  dBuffer1 = NULL;
  void*  dBuffer2 = NULL;
  void*  dBuffer3 = NULL;
  /* dBuffer4, dBuffer5 are needed by hipsparseSpGEMMreuse_compute, and therefore are stored in mmdata */
  size_t bufferSize1 = 0;
  size_t bufferSize2 = 0;
  size_t bufferSize3 = 0;
  size_t bufferSize4 = 0;
  size_t bufferSize5 = 0;

  /*----------------------------------------------------------------------*/
  /* ask bufferSize1 bytes for external memory */
  stat = hipsparseSpGEMMreuse_workEstimation(Ccusp->handle, opA, opB, Amat->matDescr, BmatSpDescr, Cmat->matDescr,
                                            HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc,
                                            &bufferSize1, NULL);CHKERRCUSPARSE(stat);
  CHKERRCUDA(hipMalloc((void**) &dBuffer1, bufferSize1));
  /* inspect the matrices A and B to understand the memory requirement for the next step */
  stat = hipsparseSpGEMMreuse_workEstimation(Ccusp->handle, opA, opB, Amat->matDescr, BmatSpDescr, Cmat->matDescr,
                                            HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc,
                                            &bufferSize1, dBuffer1);CHKERRCUSPARSE(stat);

  /*----------------------------------------------------------------------*/
  stat = hipsparseSpGEMMreuse_nnz(Ccusp->handle, opA, opB, Amat->matDescr, BmatSpDescr, Cmat->matDescr,
                                 HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc,
                                 &bufferSize2, NULL, &bufferSize3, NULL, &bufferSize4, NULL);CHKERRCUSPARSE(stat);
  CHKERRCUDA(hipMalloc((void**) &dBuffer2, bufferSize2));
  CHKERRCUDA(hipMalloc((void**) &dBuffer3, bufferSize3));
  CHKERRCUDA(hipMalloc((void**) &mmdata->dBuffer4, bufferSize4));
  stat = hipsparseSpGEMMreuse_nnz(Ccusp->handle, opA, opB, Amat->matDescr, BmatSpDescr, Cmat->matDescr,
                                 HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc,
                                 &bufferSize2, dBuffer2, &bufferSize3, dBuffer3, &bufferSize4, mmdata->dBuffer4);CHKERRCUSPARSE(stat);
  CHKERRCUDA(hipFree(dBuffer1));
  CHKERRCUDA(hipFree(dBuffer2));

  /*----------------------------------------------------------------------*/
  /* get matrix C non-zero entries C_nnz1 */
  CHKERRCUSPARSE(hipsparseSpMatGetSize(Cmat->matDescr, &C_num_rows1, &C_num_cols1, &C_nnz1));
  c->nz = (PetscInt) C_nnz1;
  /* allocate matrix C */
  Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  Ccsr->values         = new THRUSTARRAY(c->nz);CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  /* update matC with the new pointers */
  stat = hipsparseCsrSetPointers(Cmat->matDescr, Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get(),
                                Ccsr->values->data().get());CHKERRCUSPARSE(stat);

  /*----------------------------------------------------------------------*/
  stat = hipsparseSpGEMMreuse_copy(Ccusp->handle, opA, opB, Amat->matDescr, BmatSpDescr, Cmat->matDescr,
                                  HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc,
                                  &bufferSize5, NULL);CHKERRCUSPARSE(stat);
  CHKERRCUDA(hipMalloc((void**) &mmdata->dBuffer5, bufferSize5));
  stat = hipsparseSpGEMMreuse_copy(Ccusp->handle, opA, opB, Amat->matDescr, BmatSpDescr, Cmat->matDescr,
                                  HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc,
                                  &bufferSize5, mmdata->dBuffer5);CHKERRCUSPARSE(stat);
  CHKERRCUDA(hipFree(dBuffer3));
  stat = hipsparseSpGEMMreuse_compute(Ccusp->handle, opA, opB,
                                     Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                     cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                     mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
  CHKERRQ(PetscInfo(C,"Buffer sizes for type %s, result %" PetscInt_FMT " x %" PetscInt_FMT " (k %" PetscInt_FMT ", nzA %" PetscInt_FMT ", nzB %" PetscInt_FMT ", nzC %" PetscInt_FMT ") are: %ldKB %ldKB\n",MatProductTypes[ptype],m,n,k,a->nz,b->nz,c->nz,bufferSize4/1024,bufferSize5/1024));
 }
 #else
  size_t bufSize2;
  /* ask bufferSize bytes for external memory */
  stat = hipsparseSpGEMM_workEstimation(Ccusp->handle, opA, opB,
                                       Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                       cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                       mmdata->spgemmDesc, &bufSize2, NULL);CHKERRCUSPARSE(stat);
  CHKERRCUDA(hipMalloc((void**) &mmdata->mmBuffer2, bufSize2));
  /* inspect the matrices A and B to understand the memory requirement for the next step */
  stat = hipsparseSpGEMM_workEstimation(Ccusp->handle, opA, opB,
                                       Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                       cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                       mmdata->spgemmDesc, &bufSize2, mmdata->mmBuffer2);CHKERRCUSPARSE(stat);
  /* ask bufferSize again bytes for external memory */
  stat = hipsparseSpGEMM_compute(Ccusp->handle, opA, opB,
                                Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                mmdata->spgemmDesc, &mmdata->mmBufferSize, NULL);CHKERRCUSPARSE(stat);
  /* The CUSPARSE documentation is not clear, nor the API
     We need both buffers to perform the operations properly!
     mmdata->mmBuffer2 does not appear anywhere in the compute/copy API
     it only appears for the workEstimation stuff, but it seems it is needed in compute, so probably the address
     is stored in the descriptor! What a messy API... */
  CHKERRCUDA(hipMalloc((void**) &mmdata->mmBuffer, mmdata->mmBufferSize));
  /* compute the intermediate product of A * B */
  stat = hipsparseSpGEMM_compute(Ccusp->handle, opA, opB,
                                Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                                cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT,
                                mmdata->spgemmDesc, &mmdata->mmBufferSize, mmdata->mmBuffer);CHKERRCUSPARSE(stat);
  /* get matrix C non-zero entries C_nnz1 */
  CHKERRCUSPARSE(hipsparseSpMatGetSize(Cmat->matDescr, &C_num_rows1, &C_num_cols1, &C_nnz1));
  c->nz = (PetscInt) C_nnz1;
  CHKERRQ(PetscInfo(C,"Buffer sizes for type %s, result %" PetscInt_FMT " x %" PetscInt_FMT " (k %" PetscInt_FMT ", nzA %" PetscInt_FMT ", nzB %" PetscInt_FMT ", nzC %" PetscInt_FMT ") are: %ldKB %ldKB\n",MatProductTypes[ptype],m,n,k,a->nz,b->nz,c->nz,bufSize2/1024,mmdata->mmBufferSize/1024));
  Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  Ccsr->values = new THRUSTARRAY(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  stat = hipsparseCsrSetPointers(Cmat->matDescr, Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get(),
                                Ccsr->values->data().get());CHKERRCUSPARSE(stat);
  stat = hipsparseSpGEMM_copy(Ccusp->handle, opA, opB,
                             Cmat->alpha_one, Amat->matDescr, BmatSpDescr, Cmat->beta_zero, Cmat->matDescr,
                             cusparse_scalartype, HIPSPARSE_SPGEMM_DEFAULT, mmdata->spgemmDesc);CHKERRCUSPARSE(stat);
 #endif // PETSC_PKG_CUDA_VERSION_GE(11,4,0)
#else
  CHKERRCUSPARSE(hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_HOST));
  stat = hipsparseXcsrgemmNnz(Ccusp->handle, opA, opB,
                             Acsr->num_rows, Bcsr->num_cols, Acsr->num_cols,
                             Amat->descr, Acsr->num_entries, Acsr->row_offsets->data().get(), Acsr->column_indices->data().get(),
                             Bmat->descr, Bcsr->num_entries, Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                             Cmat->descr, Ccsr->row_offsets->data().get(), &cnz);CHKERRCUSPARSE(stat);
  c->nz = cnz;
  Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */
  Ccsr->values = new THRUSTARRAY(c->nz);
  CHKERRCUDA(hipPeekAtLastError()); /* catch out of memory errors */

  CHKERRCUSPARSE(hipsparseSetPointerMode(Ccusp->handle, HIPSPARSE_POINTER_MODE_DEVICE));
  /* with the old gemm interface (removed from 11.0 on) we cannot compute the symbolic factorization only.
     I have tried using the gemm2 interface (alpha * A * B + beta * D), which allows to do symbolic by passing NULL for values, but it seems quite buggy when
     D is NULL, despite the fact that CUSPARSE documentation claims it is supported! */
  stat = cusparse_csr_spgemm(Ccusp->handle, opA, opB,
                             Acsr->num_rows, Bcsr->num_cols, Acsr->num_cols,
                             Amat->descr, Acsr->num_entries, Acsr->values->data().get(), Acsr->row_offsets->data().get(), Acsr->column_indices->data().get(),
                             Bmat->descr, Bcsr->num_entries, Bcsr->values->data().get(), Bcsr->row_offsets->data().get(), Bcsr->column_indices->data().get(),
                             Cmat->descr, Ccsr->values->data().get(), Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get());CHKERRCUSPARSE(stat);
#endif
  CHKERRQ(PetscLogGpuFlops(mmdata->flops));
  CHKERRQ(PetscLogGpuTimeEnd());
finalizesym:
  c->singlemalloc = PETSC_FALSE;
  c->free_a       = PETSC_TRUE;
  c->free_ij      = PETSC_TRUE;
  CHKERRQ(PetscMalloc1(m+1,&c->i));
  CHKERRQ(PetscMalloc1(c->nz,&c->j));
  if (PetscDefined(USE_64BIT_INDICES)) { /* 32 to 64 bit conversion on the GPU and then copy to host (lazy) */
    PetscInt *d_i = c->i;
    THRUSTINTARRAY ii(Ccsr->row_offsets->size());
    THRUSTINTARRAY jj(Ccsr->column_indices->size());
    ii   = *Ccsr->row_offsets;
    jj   = *Ccsr->column_indices;
    if (ciscompressed) d_i = c->compressedrow.i;
    CHKERRCUDA(hipMemcpy(d_i,ii.data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
    CHKERRCUDA(hipMemcpy(c->j,jj.data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
  } else {
    PetscInt *d_i = c->i;
    if (ciscompressed) d_i = c->compressedrow.i;
    CHKERRCUDA(hipMemcpy(d_i,Ccsr->row_offsets->data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
    CHKERRCUDA(hipMemcpy(c->j,Ccsr->column_indices->data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
  }
  if (ciscompressed) { /* need to expand host row offsets */
    PetscInt r = 0;
    c->i[0] = 0;
    for (k = 0; k < c->compressedrow.nrows; k++) {
      const PetscInt next = c->compressedrow.rindex[k];
      const PetscInt old = c->compressedrow.i[k];
      for (; r < next; r++) c->i[r+1] = old;
    }
    for (; r < m; r++) c->i[r+1] = c->compressedrow.i[c->compressedrow.nrows];
  }
  CHKERRQ(PetscLogGpuToCpu((Ccsr->column_indices->size() + Ccsr->row_offsets->size())*sizeof(PetscInt)));
  CHKERRQ(PetscMalloc1(m,&c->ilen));
  CHKERRQ(PetscMalloc1(m,&c->imax));
  c->maxnz = c->nz;
  c->nonzerorowcnt = 0;
  c->rmax = 0;
  for (k = 0; k < m; k++) {
    const PetscInt nn = c->i[k+1] - c->i[k];
    c->ilen[k] = c->imax[k] = nn;
    c->nonzerorowcnt += (PetscInt)!!nn;
    c->rmax = PetscMax(c->rmax,nn);
  }
  CHKERRQ(MatMarkDiagonal_SeqAIJ(C));
  CHKERRQ(PetscMalloc1(c->nz,&c->a));
  Ccsr->num_entries = c->nz;

  C->nonzerostate++;
  CHKERRQ(PetscLayoutSetUp(C->rmap));
  CHKERRQ(PetscLayoutSetUp(C->cmap));
  Ccusp->nonzerostate = C->nonzerostate;
  C->offloadmask   = PETSC_OFFLOAD_UNALLOCATED;
  C->preallocated  = PETSC_TRUE;
  C->assembled     = PETSC_FALSE;
  C->was_assembled = PETSC_FALSE;
  if (product->api_user && A->offloadmask == PETSC_OFFLOAD_BOTH && B->offloadmask == PETSC_OFFLOAD_BOTH) { /* flag the matrix C values as computed, so that the numeric phase will only call MatAssembly */
    mmdata->reusesym = PETSC_TRUE;
    C->offloadmask   = PETSC_OFFLOAD_GPU;
  }
  C->ops->productnumeric = MatProductNumeric_SeqAIJCUSPARSE_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatProductSetFromOptions_SeqAIJ_SeqDense(Mat);

/* handles sparse or dense B */
static PetscErrorCode MatProductSetFromOptions_SeqAIJCUSPARSE(Mat mat)
{
  Mat_Product    *product = mat->product;
  PetscErrorCode ierr;
  PetscBool      isdense = PETSC_FALSE,Biscusp = PETSC_FALSE,Ciscusp = PETSC_TRUE;

  PetscFunctionBegin;
  MatCheckProduct(mat,1);
  CHKERRQ(PetscObjectBaseTypeCompare((PetscObject)product->B,MATSEQDENSE,&isdense));
  if (!product->A->boundtocpu && !product->B->boundtocpu) {
    CHKERRQ(PetscObjectTypeCompare((PetscObject)product->B,MATSEQAIJCUSPARSE,&Biscusp));
  }
  if (product->type == MATPRODUCT_ABC) {
    Ciscusp = PETSC_FALSE;
    if (!product->C->boundtocpu) {
      CHKERRQ(PetscObjectTypeCompare((PetscObject)product->C,MATSEQAIJCUSPARSE,&Ciscusp));
    }
  }
  if (Biscusp && Ciscusp) { /* we can always select the CPU backend */
    PetscBool usecpu = PETSC_FALSE;
    switch (product->type) {
    case MATPRODUCT_AB:
      if (product->api_user) {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatMatMult","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-matmatmult_backend_cpu","Use CPU code","MatMatMult",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      } else {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatProduct_AB","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-mat_product_algorithm_backend_cpu","Use CPU code","MatMatMult",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      }
      break;
    case MATPRODUCT_AtB:
      if (product->api_user) {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatTransposeMatMult","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-mattransposematmult_backend_cpu","Use CPU code","MatTransposeMatMult",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      } else {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatProduct_AtB","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-mat_product_algorithm_backend_cpu","Use CPU code","MatTransposeMatMult",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      }
      break;
    case MATPRODUCT_PtAP:
      if (product->api_user) {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatPtAP","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-matptap_backend_cpu","Use CPU code","MatPtAP",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      } else {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatProduct_PtAP","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-mat_product_algorithm_backend_cpu","Use CPU code","MatPtAP",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      }
      break;
    case MATPRODUCT_RARt:
      if (product->api_user) {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatRARt","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-matrart_backend_cpu","Use CPU code","MatRARt",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      } else {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatProduct_RARt","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-mat_product_algorithm_backend_cpu","Use CPU code","MatRARt",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      }
      break;
    case MATPRODUCT_ABC:
      if (product->api_user) {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatMatMatMult","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-matmatmatmult_backend_cpu","Use CPU code","MatMatMatMult",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      } else {
        ierr = PetscOptionsBegin(PetscObjectComm((PetscObject)mat),((PetscObject)mat)->prefix,"MatProduct_ABC","Mat");CHKERRQ(ierr);
        CHKERRQ(PetscOptionsBool("-mat_product_algorithm_backend_cpu","Use CPU code","MatMatMatMult",usecpu,&usecpu,NULL));
        ierr = PetscOptionsEnd();CHKERRQ(ierr);
      }
      break;
    default:
      break;
    }
    if (usecpu) Biscusp = Ciscusp = PETSC_FALSE;
  }
  /* dispatch */
  if (isdense) {
    switch (product->type) {
    case MATPRODUCT_AB:
    case MATPRODUCT_AtB:
    case MATPRODUCT_ABt:
    case MATPRODUCT_PtAP:
    case MATPRODUCT_RARt:
     if (product->A->boundtocpu) {
        CHKERRQ(MatProductSetFromOptions_SeqAIJ_SeqDense(mat));
      } else {
        mat->ops->productsymbolic = MatProductSymbolic_SeqAIJCUSPARSE_SeqDENSECUDA;
      }
      break;
    case MATPRODUCT_ABC:
      mat->ops->productsymbolic = MatProductSymbolic_ABC_Basic;
      break;
    default:
      break;
    }
  } else if (Biscusp && Ciscusp) {
    switch (product->type) {
    case MATPRODUCT_AB:
    case MATPRODUCT_AtB:
    case MATPRODUCT_ABt:
      mat->ops->productsymbolic = MatProductSymbolic_SeqAIJCUSPARSE_SeqAIJCUSPARSE;
      break;
    case MATPRODUCT_PtAP:
    case MATPRODUCT_RARt:
    case MATPRODUCT_ABC:
      mat->ops->productsymbolic = MatProductSymbolic_ABC_Basic;
      break;
    default:
      break;
    }
  } else { /* fallback for AIJ */
    CHKERRQ(MatProductSetFromOptions_SeqAIJ(mat));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAddKernel_SeqAIJCUSPARSE(A,xx,NULL,yy,PETSC_FALSE,PETSC_FALSE));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy, Vec zz)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAddKernel_SeqAIJCUSPARSE(A,xx,yy,zz,PETSC_FALSE,PETSC_FALSE));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultHermitianTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAddKernel_SeqAIJCUSPARSE(A,xx,NULL,yy,PETSC_TRUE,PETSC_TRUE));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultHermitianTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAddKernel_SeqAIJCUSPARSE(A,xx,yy,zz,PETSC_TRUE,PETSC_TRUE));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAddKernel_SeqAIJCUSPARSE(A,xx,NULL,yy,PETSC_TRUE,PETSC_FALSE));
  PetscFunctionReturn(0);
}

__global__ static void ScatterAdd(PetscInt n, PetscInt *idx,const PetscScalar *x,PetscScalar *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[idx[i]] += x[i];
}

/* z = op(A) x + y. If trans & !herm, op = ^T; if trans & herm, op = ^H; if !trans, op = no-op */
static PetscErrorCode MatMultAddKernel_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz,PetscBool trans,PetscBool herm)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE           *cusparsestruct = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJCUSPARSEMultStruct *matstruct;
  PetscScalar                  *xarray,*zarray,*dptr,*beta,*xptr;
  hipsparseOperation_t          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  PetscBool                    compressed;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
  PetscInt                     nx,ny;
#endif

  PetscFunctionBegin;
  PetscCheckFalse(herm && !trans,PetscObjectComm((PetscObject)A),PETSC_ERR_GPU,"Hermitian and not transpose not supported");
  if (!a->nz) {
    if (!yy) CHKERRQ(VecSet_SeqCUDA(zz,0));
    else CHKERRQ(VecCopy_SeqCUDA(yy,zz));
    PetscFunctionReturn(0);
  }
  /* The line below is necessary due to the operations that modify the matrix on the CPU (axpy, scale, etc) */
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  if (!trans) {
    matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
    PetscCheck(matstruct,PetscObjectComm((PetscObject)A),PETSC_ERR_GPU,"SeqAIJCUSPARSE does not have a 'mat' (need to fix)");
  } else {
    if (herm || !A->form_explicit_transpose) {
      opA = herm ? HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE;
      matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->mat;
    } else {
      if (!cusparsestruct->matTranspose) CHKERRQ(MatSeqAIJCUSPARSEFormExplicitTranspose(A));
      matstruct = (Mat_SeqAIJCUSPARSEMultStruct*)cusparsestruct->matTranspose;
    }
  }
  /* Does the matrix use compressed rows (i.e., drop zero rows)? */
  compressed = matstruct->cprowIndices ? PETSC_TRUE : PETSC_FALSE;

  try {
    CHKERRQ(VecCUDAGetArrayRead(xx,(const PetscScalar**)&xarray));
    if (yy == zz) CHKERRQ(VecCUDAGetArray(zz,&zarray)); /* read & write zz, so need to get uptodate zarray on GPU */
    else CHKERRQ(VecCUDAGetArrayWrite(zz,&zarray)); /* write zz, so no need to init zarray on GPU */

    CHKERRQ(PetscLogGpuTimeBegin());
    if (opA == HIPSPARSE_OPERATION_NON_TRANSPOSE) {
      /* z = A x + beta y.
         If A is compressed (with less rows), then Ax is shorter than the full z, so we need a work vector to store Ax.
         When A is non-compressed, and z = y, we can set beta=1 to compute y = Ax + y in one call.
      */
      xptr = xarray;
      dptr = compressed ? cusparsestruct->workVector->data().get() : zarray;
      beta = (yy == zz && !compressed) ? matstruct->beta_one : matstruct->beta_zero;
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      /* Get length of x, y for y=Ax. ny might be shorter than the work vector's allocated length, since the work vector is
          allocated to accommodate different uses. So we get the length info directly from mat.
       */
      if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
        CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
        nx = mat->num_cols;
        ny = mat->num_rows;
      }
     #endif
    } else {
      /* z = A^T x + beta y
         If A is compressed, then we need a work vector as the shorter version of x to compute A^T x.
         Note A^Tx is of full length, so we set beta to 1.0 if y exists.
       */
      xptr = compressed ? cusparsestruct->workVector->data().get() : xarray;
      dptr = zarray;
      beta = yy ? matstruct->beta_one : matstruct->beta_zero;
      if (compressed) { /* Scatter x to work vector */
        thrust::device_ptr<PetscScalar> xarr = thrust::device_pointer_cast(xarray);
        thrust::for_each(thrust::cuda::par.on(PetscDefaultCudaStream),thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(xarr, matstruct->cprowIndices->begin()))),
                         thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(xarr, matstruct->cprowIndices->begin()))) + matstruct->cprowIndices->size(),
                         VecCUDAEqualsReverse());
      }
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
        CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
        nx = mat->num_rows;
        ny = mat->num_cols;
      }
     #endif
    }

    /* csr_spmv does y = alpha op(A) x + beta y */
    if (cusparsestruct->format == MAT_CUSPARSE_CSR) {
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      PetscCheck(opA >= 0 && opA <= 2,PETSC_COMM_SELF,PETSC_ERR_SUP,"cuSPARSE ABI on hipsparseOperation_t has changed and PETSc has not been updated accordingly");
      if (!matstruct->cuSpMV[opA].initialized) { /* built on demand */
        CHKERRCUSPARSE(hipsparseCreateDnVec(&matstruct->cuSpMV[opA].vecXDescr,nx,xptr,cusparse_scalartype));
        CHKERRCUSPARSE(hipsparseCreateDnVec(&matstruct->cuSpMV[opA].vecYDescr,ny,dptr,cusparse_scalartype));
        CHKERRCUSPARSE(hipsparseSpMV_bufferSize(cusparsestruct->handle, opA, matstruct->alpha_one,
                                               matstruct->matDescr,
                                               matstruct->cuSpMV[opA].vecXDescr, beta,
                                               matstruct->cuSpMV[opA].vecYDescr,
                                               cusparse_scalartype,
                                               cusparsestruct->spmvAlg,
                                               &matstruct->cuSpMV[opA].spmvBufferSize));
        CHKERRCUDA(hipMalloc(&matstruct->cuSpMV[opA].spmvBuffer,matstruct->cuSpMV[opA].spmvBufferSize));

        matstruct->cuSpMV[opA].initialized = PETSC_TRUE;
      } else {
        /* x, y's value pointers might change between calls, but their shape is kept, so we just update pointers */
        CHKERRCUSPARSE(hipsparseDnVecSetValues(matstruct->cuSpMV[opA].vecXDescr,xptr));
        CHKERRCUSPARSE(hipsparseDnVecSetValues(matstruct->cuSpMV[opA].vecYDescr,dptr));
      }

      CHKERRCUSPARSE(hipsparseSpMV(cusparsestruct->handle, opA,
                                  matstruct->alpha_one,
                                  matstruct->matDescr, /* built in MatSeqAIJCUSPARSECopyToGPU() or MatSeqAIJCUSPARSEFormExplicitTranspose() */
                                  matstruct->cuSpMV[opA].vecXDescr,
                                  beta,
                                  matstruct->cuSpMV[opA].vecYDescr,
                                  cusparse_scalartype,
                                  cusparsestruct->spmvAlg,
                                  matstruct->cuSpMV[opA].spmvBuffer));
     #else
      CsrMatrix *mat = (CsrMatrix*)matstruct->mat;
      CHKERRCUSPARSE(cusparse_csr_spmv(cusparsestruct->handle, opA,
                                       mat->num_rows, mat->num_cols,
                                       mat->num_entries, matstruct->alpha_one, matstruct->descr,
                                       mat->values->data().get(), mat->row_offsets->data().get(),
                                       mat->column_indices->data().get(), xptr, beta,
                                       dptr));
     #endif
    } else {
      if (cusparsestruct->nrows) {
       #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
       #else
        hipsparseHybMat_t hybMat = (hipsparseHybMat_t)matstruct->mat;
        CHKERRCUSPARSE(cusparse_hyb_spmv(cusparsestruct->handle, opA,
                                         matstruct->alpha_one, matstruct->descr, hybMat,
                                         xptr, beta,
                                         dptr));
       #endif
      }
    }
    CHKERRQ(PetscLogGpuTimeEnd());

    if (opA == HIPSPARSE_OPERATION_NON_TRANSPOSE) {
      if (yy) { /* MatMultAdd: zz = A*xx + yy */
        if (compressed) { /* A is compressed. We first copy yy to zz, then ScatterAdd the work vector to zz */
          CHKERRQ(VecCopy_SeqCUDA(yy,zz)); /* zz = yy */
        } else if (zz != yy) { /* A is not compressed. zz already contains A*xx, and we just need to add yy */
          CHKERRQ(VecAXPY_SeqCUDA(zz,1.0,yy)); /* zz += yy */
        }
      } else if (compressed) { /* MatMult: zz = A*xx. A is compressed, so we zero zz first, then ScatterAdd the work vector to zz */
        CHKERRQ(VecSet_SeqCUDA(zz,0));
      }

      /* ScatterAdd the result from work vector into the full vector when A is compressed */
      if (compressed) {
        CHKERRQ(PetscLogGpuTimeBegin());
        /* I wanted to make this for_each asynchronous but failed. thrust::async::for_each() returns an event (internally registerred)
           and in the destructor of the scope, it will call hipStreamSynchronize() on this stream. One has to store all events to
           prevent that. So I just add a ScatterAdd kernel.
         */
       #if 0
        thrust::device_ptr<PetscScalar> zptr = thrust::device_pointer_cast(zarray);
        thrust::async::for_each(thrust::cuda::par.on(cusparsestruct->stream),
                         thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstruct->cprowIndices->begin()))),
                         thrust::make_zip_iterator(thrust::make_tuple(cusparsestruct->workVector->begin(), thrust::make_permutation_iterator(zptr, matstruct->cprowIndices->begin()))) + matstruct->cprowIndices->size(),
                         VecCUDAPlusEquals());
       #else
        PetscInt n = matstruct->cprowIndices->size();
        ScatterAdd<<<(n+255)/256,256,0,PetscDefaultCudaStream>>>(n,matstruct->cprowIndices->data().get(),cusparsestruct->workVector->data().get(),zarray);
       #endif
        CHKERRQ(PetscLogGpuTimeEnd());
      }
    } else {
      if (yy && yy != zz) {
        CHKERRQ(VecAXPY_SeqCUDA(zz,1.0,yy)); /* zz += yy */
      }
    }
    CHKERRQ(VecCUDARestoreArrayRead(xx,(const PetscScalar**)&xarray));
    if (yy == zz) CHKERRQ(VecCUDARestoreArray(zz,&zarray));
    else CHKERRQ(VecCUDARestoreArrayWrite(zz,&zarray));
  } catch(char *ex) {
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  if (yy) {
    CHKERRQ(PetscLogGpuFlops(2.0*a->nz));
  } else {
    CHKERRQ(PetscLogGpuFlops(2.0*a->nz-a->nonzerorowcnt));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAddKernel_SeqAIJCUSPARSE(A,xx,yy,zz,PETSC_TRUE,PETSC_FALSE));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatAssemblyEnd_SeqAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscObjectState   onnz = A->nonzerostate;
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatAssemblyEnd_SeqAIJ(A,mode));
  if (onnz != A->nonzerostate && cusp->deviceMat) {

    CHKERRQ(PetscInfo(A,"Destroy device mat since nonzerostate changed\n"));
    CHKERRCUDA(hipFree(cusp->deviceMat));
    cusp->deviceMat = NULL;
  }
  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
/*@
   MatCreateSeqAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format). This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix
   assembly performance the user should preallocate the matrix storage by setting
   the parameter nz (or the array nnz).  By setting these parameters accurately,
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows
         (possibly different for each row) or NULL

   Output Parameter:
.  A - the matrix

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradgm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=NULL for PETSc to control dynamic memory
   allocation.  For large problems you MUST preallocate memory or you
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to
   improve numerical efficiency of matrix-vector products and solves. We
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATSEQAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateSeqAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscFunctionBegin;
  CHKERRQ(MatCreate(comm,A));
  CHKERRQ(MatSetSizes(*A,m,n,m,n));
  CHKERRQ(MatSetType(*A,MATSEQAIJCUSPARSE));
  CHKERRQ(MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDestroy_SeqAIJCUSPARSE(Mat A)
{
  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) {
    CHKERRQ(MatSeqAIJCUSPARSE_Destroy((Mat_SeqAIJCUSPARSE**)&A->spptr));
  } else {
    CHKERRQ(MatSeqAIJCUSPARSETriFactors_Destroy((Mat_SeqAIJCUSPARSETriFactors**)&A->spptr));
  }
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJCopySubArray_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatCUSPARSESetFormat_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatCUSPARSESetUseCPUSolve_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdensecuda_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdense_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqaijcusparse_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatFactorGetSolverType_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatConvert_seqaijcusparse_hypre_C",NULL));
  CHKERRQ(MatDestroy_SeqAIJ(A));
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatConvert_SeqAIJ_SeqAIJCUSPARSE(Mat,MatType,MatReuse,Mat*);
static PetscErrorCode MatBindToCPU_SeqAIJCUSPARSE(Mat,PetscBool);
static PetscErrorCode MatDuplicate_SeqAIJCUSPARSE(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  PetscFunctionBegin;
  CHKERRQ(MatDuplicate_SeqAIJ(A,cpvalues,B));
  CHKERRQ(MatConvert_SeqAIJ_SeqAIJCUSPARSE(*B,MATSEQAIJCUSPARSE,MAT_INPLACE_MATRIX,B));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatAXPY_SeqAIJCUSPARSE(Mat Y,PetscScalar a,Mat X,MatStructure str)
{
  Mat_SeqAIJ         *x = (Mat_SeqAIJ*)X->data,*y = (Mat_SeqAIJ*)Y->data;
  Mat_SeqAIJCUSPARSE *cy;
  Mat_SeqAIJCUSPARSE *cx;
  PetscScalar        *ay;
  const PetscScalar  *ax;
  CsrMatrix          *csry,*csrx;

  PetscFunctionBegin;
  cy = (Mat_SeqAIJCUSPARSE*)Y->spptr;
  cx = (Mat_SeqAIJCUSPARSE*)X->spptr;
  if (X->ops->axpy != Y->ops->axpy) {
    CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(Y,PETSC_FALSE));
    CHKERRQ(MatAXPY_SeqAIJ(Y,a,X,str));
    PetscFunctionReturn(0);
  }
  /* if we are here, it means both matrices are bound to GPU */
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(Y));
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(X));
  PetscCheck(cy->format == MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)Y),PETSC_ERR_GPU,"only MAT_CUSPARSE_CSR supported");
  PetscCheck(cx->format == MAT_CUSPARSE_CSR,PetscObjectComm((PetscObject)X),PETSC_ERR_GPU,"only MAT_CUSPARSE_CSR supported");
  csry = (CsrMatrix*)cy->mat->mat;
  csrx = (CsrMatrix*)cx->mat->mat;
  /* see if we can turn this into a cublas axpy */
  if (str != SAME_NONZERO_PATTERN && x->nz == y->nz && !x->compressedrow.use && !y->compressedrow.use) {
    bool eq = thrust::equal(thrust::device,csry->row_offsets->begin(),csry->row_offsets->end(),csrx->row_offsets->begin());
    if (eq) {
      eq = thrust::equal(thrust::device,csry->column_indices->begin(),csry->column_indices->end(),csrx->column_indices->begin());
    }
    if (eq) str = SAME_NONZERO_PATTERN;
  }
  /* spgeam is buggy with one column */
  if (Y->cmap->n == 1 && str != SAME_NONZERO_PATTERN) str = DIFFERENT_NONZERO_PATTERN;

  if (str == SUBSET_NONZERO_PATTERN) {
    PetscScalar b = 1.0;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    size_t      bufferSize;
    void        *buffer;
#endif

    CHKERRQ(MatSeqAIJCUSPARSEGetArrayRead(X,&ax));
    CHKERRQ(MatSeqAIJCUSPARSEGetArray(Y,&ay));
    CHKERRCUSPARSE(hipsparseSetPointerMode(cy->handle, HIPSPARSE_POINTER_MODE_HOST));
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    CHKERRCUSPARSE(cusparse_csr_spgeam_bufferSize(cy->handle,Y->rmap->n,Y->cmap->n,
                                                  &a,cx->mat->descr,x->nz,ax,csrx->row_offsets->data().get(),csrx->column_indices->data().get(),
                                                  &b,cy->mat->descr,y->nz,ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),
                                                  cy->mat->descr,      ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),&bufferSize));
    CHKERRCUDA(hipMalloc(&buffer,bufferSize));
    CHKERRQ(PetscLogGpuTimeBegin());
    CHKERRCUSPARSE(cusparse_csr_spgeam(cy->handle,Y->rmap->n,Y->cmap->n,
                                       &a,cx->mat->descr,x->nz,ax,csrx->row_offsets->data().get(),csrx->column_indices->data().get(),
                                       &b,cy->mat->descr,y->nz,ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),
                                       cy->mat->descr,      ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),buffer));
    CHKERRQ(PetscLogGpuFlops(x->nz + y->nz));
    CHKERRQ(PetscLogGpuTimeEnd());
    CHKERRCUDA(hipFree(buffer));
#else
    CHKERRQ(PetscLogGpuTimeBegin());
    CHKERRCUSPARSE(cusparse_csr_spgeam(cy->handle,Y->rmap->n,Y->cmap->n,
                                       &a,cx->mat->descr,x->nz,ax,csrx->row_offsets->data().get(),csrx->column_indices->data().get(),
                                       &b,cy->mat->descr,y->nz,ay,csry->row_offsets->data().get(),csry->column_indices->data().get(),
                                       cy->mat->descr,      ay,csry->row_offsets->data().get(),csry->column_indices->data().get()));
    CHKERRQ(PetscLogGpuFlops(x->nz + y->nz));
    CHKERRQ(PetscLogGpuTimeEnd());
#endif
    CHKERRCUSPARSE(hipsparseSetPointerMode(cy->handle, HIPSPARSE_POINTER_MODE_DEVICE));
    CHKERRQ(MatSeqAIJCUSPARSERestoreArrayRead(X,&ax));
    CHKERRQ(MatSeqAIJCUSPARSERestoreArray(Y,&ay));
    CHKERRQ(MatSeqAIJInvalidateDiagonal(Y));
  } else if (str == SAME_NONZERO_PATTERN) {
    hipblasHandle_t cublasv2handle;
    PetscBLASInt   one = 1, bnz = 1;

    CHKERRQ(MatSeqAIJCUSPARSEGetArrayRead(X,&ax));
    CHKERRQ(MatSeqAIJCUSPARSEGetArray(Y,&ay));
    CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
    CHKERRQ(PetscBLASIntCast(x->nz,&bnz));
    CHKERRQ(PetscLogGpuTimeBegin());
    CHKERRCUBLAS(cublasXaxpy(cublasv2handle,bnz,&a,ax,one,ay,one));
    CHKERRQ(PetscLogGpuFlops(2.0*bnz));
    CHKERRQ(PetscLogGpuTimeEnd());
    CHKERRQ(MatSeqAIJCUSPARSERestoreArrayRead(X,&ax));
    CHKERRQ(MatSeqAIJCUSPARSERestoreArray(Y,&ay));
    CHKERRQ(MatSeqAIJInvalidateDiagonal(Y));
  } else {
    CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(Y,PETSC_FALSE));
    CHKERRQ(MatAXPY_SeqAIJ(Y,a,X,str));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatScale_SeqAIJCUSPARSE(Mat Y,PetscScalar a)
{
  Mat_SeqAIJ     *y = (Mat_SeqAIJ*)Y->data;
  PetscScalar    *ay;
  hipblasHandle_t cublasv2handle;
  PetscBLASInt   one = 1, bnz = 1;

  PetscFunctionBegin;
  CHKERRQ(MatSeqAIJCUSPARSEGetArray(Y,&ay));
  CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
  CHKERRQ(PetscBLASIntCast(y->nz,&bnz));
  CHKERRQ(PetscLogGpuTimeBegin());
  CHKERRCUBLAS(cublasXscal(cublasv2handle,bnz,&a,ay,one));
  CHKERRQ(PetscLogGpuFlops(bnz));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatSeqAIJCUSPARSERestoreArray(Y,&ay));
  CHKERRQ(MatSeqAIJInvalidateDiagonal(Y));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatZeroEntries_SeqAIJCUSPARSE(Mat A)
{
  PetscBool      both = PETSC_FALSE;
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) {
    Mat_SeqAIJCUSPARSE *spptr = (Mat_SeqAIJCUSPARSE*)A->spptr;
    if (spptr->mat) {
      CsrMatrix* matrix = (CsrMatrix*)spptr->mat->mat;
      if (matrix->values) {
        both = PETSC_TRUE;
        thrust::fill(thrust::device,matrix->values->begin(),matrix->values->end(),0.);
      }
    }
    if (spptr->matTranspose) {
      CsrMatrix* matrix = (CsrMatrix*)spptr->matTranspose->mat;
      if (matrix->values) {
        thrust::fill(thrust::device,matrix->values->begin(),matrix->values->end(),0.);
      }
    }
  }
  CHKERRQ(PetscArrayzero(a->a,a->i[A->rmap->n]));
  CHKERRQ(MatSeqAIJInvalidateDiagonal(A));
  if (both) A->offloadmask = PETSC_OFFLOAD_BOTH;
  else A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatBindToCPU_SeqAIJCUSPARSE(Mat A,PetscBool flg)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;

  PetscFunctionBegin;
  if (A->factortype != MAT_FACTOR_NONE) {
    A->boundtocpu = flg;
    PetscFunctionReturn(0);
  }
  if (flg) {
    CHKERRQ(MatSeqAIJCUSPARSECopyFromGPU(A));

    A->ops->scale                     = MatScale_SeqAIJ;
    A->ops->axpy                      = MatAXPY_SeqAIJ;
    A->ops->zeroentries               = MatZeroEntries_SeqAIJ;
    A->ops->mult                      = MatMult_SeqAIJ;
    A->ops->multadd                   = MatMultAdd_SeqAIJ;
    A->ops->multtranspose             = MatMultTranspose_SeqAIJ;
    A->ops->multtransposeadd          = MatMultTransposeAdd_SeqAIJ;
    A->ops->multhermitiantranspose    = NULL;
    A->ops->multhermitiantransposeadd = NULL;
    A->ops->productsetfromoptions     = MatProductSetFromOptions_SeqAIJ;
    CHKERRQ(PetscMemzero(a->ops,sizeof(Mat_SeqAIJOps)));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJCopySubArray_C",NULL));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdensecuda_C",NULL));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdense_C",NULL));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",NULL));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",NULL));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJGetArray_C",MatSeqAIJGetArray_SeqAIJ));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqaijcusparse_C",NULL));
  } else {
    A->ops->scale                     = MatScale_SeqAIJCUSPARSE;
    A->ops->axpy                      = MatAXPY_SeqAIJCUSPARSE;
    A->ops->zeroentries               = MatZeroEntries_SeqAIJCUSPARSE;
    A->ops->mult                      = MatMult_SeqAIJCUSPARSE;
    A->ops->multadd                   = MatMultAdd_SeqAIJCUSPARSE;
    A->ops->multtranspose             = MatMultTranspose_SeqAIJCUSPARSE;
    A->ops->multtransposeadd          = MatMultTransposeAdd_SeqAIJCUSPARSE;
    A->ops->multhermitiantranspose    = MatMultHermitianTranspose_SeqAIJCUSPARSE;
    A->ops->multhermitiantransposeadd = MatMultHermitianTransposeAdd_SeqAIJCUSPARSE;
    A->ops->productsetfromoptions     = MatProductSetFromOptions_SeqAIJCUSPARSE;
    a->ops->getarray                  = MatSeqAIJGetArray_SeqAIJCUSPARSE;
    a->ops->restorearray              = MatSeqAIJRestoreArray_SeqAIJCUSPARSE;
    a->ops->getarrayread              = MatSeqAIJGetArrayRead_SeqAIJCUSPARSE;
    a->ops->restorearrayread          = MatSeqAIJRestoreArrayRead_SeqAIJCUSPARSE;
    a->ops->getarraywrite             = MatSeqAIJGetArrayWrite_SeqAIJCUSPARSE;
    a->ops->restorearraywrite         = MatSeqAIJRestoreArrayWrite_SeqAIJCUSPARSE;
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSeqAIJCopySubArray_C",MatSeqAIJCopySubArray_SeqAIJCUSPARSE));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdensecuda_C",MatProductSetFromOptions_SeqAIJCUSPARSE));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqdense_C",MatProductSetFromOptions_SeqAIJCUSPARSE));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSetPreallocationCOO_C",MatSetPreallocationCOO_SeqAIJCUSPARSE));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatSetValuesCOO_C",MatSetValuesCOO_SeqAIJCUSPARSE));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatProductSetFromOptions_seqaijcusparse_seqaijcusparse_C",MatProductSetFromOptions_SeqAIJCUSPARSE));
  }
  A->boundtocpu = flg;
  if (flg && a->inode.size) {
    a->inode.use = PETSC_TRUE;
  } else {
    a->inode.use = PETSC_FALSE;
  }
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatConvert_SeqAIJ_SeqAIJCUSPARSE(Mat A, MatType mtype, MatReuse reuse, Mat* newmat)
{
  Mat              B;

  PetscFunctionBegin;
  CHKERRQ(PetscDeviceInitialize(PETSC_DEVICE_CUDA)); /* first use of CUSPARSE may be via MatConvert */
  if (reuse == MAT_INITIAL_MATRIX) {
    CHKERRQ(MatDuplicate(A,MAT_COPY_VALUES,newmat));
  } else if (reuse == MAT_REUSE_MATRIX) {
    CHKERRQ(MatCopy(A,*newmat,SAME_NONZERO_PATTERN));
  }
  B = *newmat;

  CHKERRQ(PetscFree(B->defaultvectype));
  CHKERRQ(PetscStrallocpy(VECCUDA,&B->defaultvectype));

  if (reuse != MAT_REUSE_MATRIX && !B->spptr) {
    if (B->factortype == MAT_FACTOR_NONE) {
      Mat_SeqAIJCUSPARSE *spptr;
      CHKERRQ(PetscNew(&spptr));
      CHKERRCUSPARSE(hipsparseCreate(&spptr->handle));
      CHKERRCUSPARSE(hipsparseSetStream(spptr->handle,PetscDefaultCudaStream));
      spptr->format     = MAT_CUSPARSE_CSR;
     #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
     #if PETSC_PKG_CUDA_VERSION_GE(11,2,0)
      spptr->spmvAlg    = HIPSPARSE_SPMV_CSR_ALG1; /* default, since we only support csr */
     #else
      spptr->spmvAlg    = HIPSPARSE_CSRMV_ALG1;    /* default, since we only support csr */
     #endif
      spptr->spmmAlg    = HIPSPARSE_SPMM_CSR_ALG1; /* default, only support column-major dense matrix B */
      spptr->csr2cscAlg = HIPSPARSE_CSR2CSC_ALG1;
     #endif
      B->spptr = spptr;
    } else {
      Mat_SeqAIJCUSPARSETriFactors *spptr;

      CHKERRQ(PetscNew(&spptr));
      CHKERRCUSPARSE(hipsparseCreate(&spptr->handle));
      CHKERRCUSPARSE(hipsparseSetStream(spptr->handle,PetscDefaultCudaStream));
      B->spptr = spptr;
    }
    B->offloadmask = PETSC_OFFLOAD_UNALLOCATED;
  }
  B->ops->assemblyend    = MatAssemblyEnd_SeqAIJCUSPARSE;
  B->ops->destroy        = MatDestroy_SeqAIJCUSPARSE;
  B->ops->setoption      = MatSetOption_SeqAIJCUSPARSE;
  B->ops->setfromoptions = MatSetFromOptions_SeqAIJCUSPARSE;
  B->ops->bindtocpu      = MatBindToCPU_SeqAIJCUSPARSE;
  B->ops->duplicate      = MatDuplicate_SeqAIJCUSPARSE;

  CHKERRQ(MatBindToCPU_SeqAIJCUSPARSE(B,PETSC_FALSE));
  CHKERRQ(PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSPARSE));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatCUSPARSESetFormat_C",MatCUSPARSESetFormat_SeqAIJCUSPARSE));
#if defined(PETSC_HAVE_HYPRE)
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqaijcusparse_hypre_C",MatConvert_AIJ_HYPRE));
#endif
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatCUSPARSESetUseCPUSolve_C",MatCUSPARSESetUseCPUSolve_SeqAIJCUSPARSE));
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatCreate_SeqAIJCUSPARSE(Mat B)
{
  PetscFunctionBegin;
  CHKERRQ(MatCreate_SeqAIJ(B));
  CHKERRQ(MatConvert_SeqAIJ_SeqAIJCUSPARSE(B,MATSEQAIJCUSPARSE,MAT_INPLACE_MATRIX,&B));
  PetscFunctionReturn(0);
}

/*MC
   MATSEQAIJCUSPARSE - MATAIJCUSPARSE = "(seq)aijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on Nvidia GPUs using the CUSPARSE library.

   Options Database Keys:
+  -mat_type aijcusparse - sets the matrix type to "seqaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr - sets the storage format of matrices (for MatMult and factors in MatSolve) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
-  -mat_cusparse_mult_storage_format csr - sets the storage format of matrices (for MatMult) during a call to MatSetFromOptions(). Other options include ell (ellpack) or hyb (hybrid).
+  -mat_cusparse_use_cpu_solve - Do MatSolve on CPU

  Level: beginner

.seealso: MatCreateSeqAIJCUSPARSE(), MATAIJCUSPARSE, MatCreateAIJCUSPARSE(), MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/

PETSC_EXTERN PetscErrorCode MatGetFactor_seqaijcusparse_cusparse_band(Mat,MatFactorType,Mat*);

PETSC_EXTERN PetscErrorCode MatSolverTypeRegister_CUSPARSE(void)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolverTypeRegister(MATSOLVERCUSPARSEBAND,MATSEQAIJ,MAT_FACTOR_LU,MatGetFactor_seqaijcusparse_cusparse_band));
  CHKERRQ(MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_LU,MatGetFactor_seqaijcusparse_cusparse));
  CHKERRQ(MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_CHOLESKY,MatGetFactor_seqaijcusparse_cusparse));
  CHKERRQ(MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ILU,MatGetFactor_seqaijcusparse_cusparse));
  CHKERRQ(MatSolverTypeRegister(MATSOLVERCUSPARSE,MATSEQAIJCUSPARSE,MAT_FACTOR_ICC,MatGetFactor_seqaijcusparse_cusparse));

  PetscFunctionReturn(0);
}

static PetscErrorCode MatResetPreallocationCOO_SeqAIJCUSPARSE(Mat mat)
{
  Mat_SeqAIJCUSPARSE* cusp = (Mat_SeqAIJCUSPARSE*)mat->spptr;
  hipError_t         cerr;

  PetscFunctionBegin;
  if (!cusp) PetscFunctionReturn(0);
  delete cusp->cooPerm;
  delete cusp->cooPerm_a;
  cusp->cooPerm = NULL;
  cusp->cooPerm_a = NULL;
  if (cusp->use_extended_coo) {
    CHKERRCUDA(hipFree(cusp->jmap_d));
    CHKERRCUDA(hipFree(cusp->perm_d));
  }
  cusp->use_extended_coo = PETSC_FALSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSE_Destroy(Mat_SeqAIJCUSPARSE **cusparsestruct)
{
  PetscFunctionBegin;
  if (*cusparsestruct) {
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->mat,(*cusparsestruct)->format));
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&(*cusparsestruct)->matTranspose,(*cusparsestruct)->format));
    delete (*cusparsestruct)->workVector;
    delete (*cusparsestruct)->rowoffsets_gpu;
    delete (*cusparsestruct)->cooPerm;
    delete (*cusparsestruct)->cooPerm_a;
    delete (*cusparsestruct)->csr2csc_i;
    if ((*cusparsestruct)->handle) CHKERRCUSPARSE(hipsparseDestroy((*cusparsestruct)->handle));
    if ((*cusparsestruct)->jmap_d) CHKERRCUDA(hipFree((*cusparsestruct)->jmap_d));
    if ((*cusparsestruct)->perm_d) CHKERRCUDA(hipFree((*cusparsestruct)->perm_d));
    CHKERRQ(PetscFree(*cusparsestruct));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode CsrMatrix_Destroy(CsrMatrix **mat)
{
  PetscFunctionBegin;
  if (*mat) {
    delete (*mat)->values;
    delete (*mat)->column_indices;
    delete (*mat)->row_offsets;
    delete *mat;
    *mat = 0;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSETriFactorStruct **trifactor)
{
  PetscFunctionBegin;
  if (*trifactor) {
    if ((*trifactor)->descr) CHKERRCUSPARSE(hipsparseDestroyMatDescr((*trifactor)->descr));
    if ((*trifactor)->solveInfo) CHKERRCUSPARSE(cusparse_destroy_analysis_info((*trifactor)->solveInfo));
    CHKERRQ(CsrMatrix_Destroy(&(*trifactor)->csrMat));
    if ((*trifactor)->solveBuffer)   CHKERRCUDA(hipFree((*trifactor)->solveBuffer));
    if ((*trifactor)->AA_h)   CHKERRCUDA(hipHostFree((*trifactor)->AA_h));
   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    if ((*trifactor)->csr2cscBuffer) CHKERRCUDA(hipFree((*trifactor)->csr2cscBuffer));
   #endif
    CHKERRQ(PetscFree(*trifactor));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSEMultStruct_Destroy(Mat_SeqAIJCUSPARSEMultStruct **matstruct,MatCUSPARSEStorageFormat format)
{
  CsrMatrix        *mat;

  PetscFunctionBegin;
  if (*matstruct) {
    if ((*matstruct)->mat) {
      if (format==MAT_CUSPARSE_ELL || format==MAT_CUSPARSE_HYB) {
       #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_CUSPARSE_ELL and MAT_CUSPARSE_HYB are not supported since CUDA-11.0");
       #else
        hipsparseHybMat_t hybMat = (hipsparseHybMat_t)(*matstruct)->mat;
        CHKERRCUSPARSE(hipsparseDestroyHybMat(hybMat));
       #endif
      } else {
        mat = (CsrMatrix*)(*matstruct)->mat;
        CsrMatrix_Destroy(&mat);
      }
    }
    if ((*matstruct)->descr) CHKERRCUSPARSE(hipsparseDestroyMatDescr((*matstruct)->descr));
    delete (*matstruct)->cprowIndices;
    if ((*matstruct)->alpha_one) CHKERRCUDA(hipFree((*matstruct)->alpha_one));
    if ((*matstruct)->beta_zero) CHKERRCUDA(hipFree((*matstruct)->beta_zero));
    if ((*matstruct)->beta_one)  CHKERRCUDA(hipFree((*matstruct)->beta_one));

   #if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
    Mat_SeqAIJCUSPARSEMultStruct *mdata = *matstruct;
    if (mdata->matDescr) CHKERRCUSPARSE(hipsparseDestroySpMat(mdata->matDescr));
    for (int i=0; i<3; i++) {
      if (mdata->cuSpMV[i].initialized) {
        CHKERRCUDA(hipFree(mdata->cuSpMV[i].spmvBuffer));
        CHKERRCUSPARSE(hipsparseDestroyDnVec(mdata->cuSpMV[i].vecXDescr));
        CHKERRCUSPARSE(hipsparseDestroyDnVec(mdata->cuSpMV[i].vecYDescr));
      }
    }
   #endif
    delete *matstruct;
    *matstruct = NULL;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSETriFactors_Reset(Mat_SeqAIJCUSPARSETriFactors_p* trifactors)
{
  PetscFunctionBegin;
  if (*trifactors) {
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->loTriFactorPtr));
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->upTriFactorPtr));
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->loTriFactorPtrTranspose));
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&(*trifactors)->upTriFactorPtrTranspose));
    delete (*trifactors)->rpermIndices;
    delete (*trifactors)->cpermIndices;
    delete (*trifactors)->workVector;
    (*trifactors)->rpermIndices = NULL;
    (*trifactors)->cpermIndices = NULL;
    (*trifactors)->workVector = NULL;
    if ((*trifactors)->a_band_d)   CHKERRCUDA(hipFree((*trifactors)->a_band_d));
    if ((*trifactors)->i_band_d)   CHKERRCUDA(hipFree((*trifactors)->i_band_d));
    (*trifactors)->init_dev_prop = PETSC_FALSE;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCUSPARSETriFactors_Destroy(Mat_SeqAIJCUSPARSETriFactors** trifactors)
{
  hipsparseHandle_t handle;

  PetscFunctionBegin;
  if (*trifactors) {
    CHKERRQ(MatSeqAIJCUSPARSETriFactors_Reset(trifactors));
    if (handle = (*trifactors)->handle) {
      CHKERRCUSPARSE(hipsparseDestroy(handle));
    }
    CHKERRQ(PetscFree(*trifactors));
  }
  PetscFunctionReturn(0);
}

struct IJCompare
{
  __host__ __device__
  inline bool operator() (const thrust::tuple<PetscInt, PetscInt> &t1, const thrust::tuple<PetscInt, PetscInt> &t2)
  {
    if (t1.get<0>() < t2.get<0>()) return true;
    if (t1.get<0>() == t2.get<0>()) return t1.get<1>() < t2.get<1>();
    return false;
  }
};

struct IJEqual
{
  __host__ __device__
  inline bool operator() (const thrust::tuple<PetscInt, PetscInt> &t1, const thrust::tuple<PetscInt, PetscInt> &t2)
  {
    if (t1.get<0>() != t2.get<0>() || t1.get<1>() != t2.get<1>()) return false;
    return true;
  }
};

struct IJDiff
{
  __host__ __device__
  inline PetscInt operator() (const PetscInt &t1, const PetscInt &t2)
  {
    return t1 == t2 ? 0 : 1;
  }
};

struct IJSum
{
  __host__ __device__
  inline PetscInt operator() (const PetscInt &t1, const PetscInt &t2)
  {
    return t1||t2;
  }
};

#include <thrust/iterator/discard_iterator.h>
/* Associated with MatSetPreallocationCOO_SeqAIJCUSPARSE_Basic() */
PetscErrorCode MatSetValuesCOO_SeqAIJCUSPARSE_Basic(Mat A, const PetscScalar v[], InsertMode imode)
{
  Mat_SeqAIJCUSPARSE                    *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ                            *a = (Mat_SeqAIJ*)A->data;
  THRUSTARRAY                           *cooPerm_v = NULL;
  thrust::device_ptr<const PetscScalar> d_v;
  CsrMatrix                             *matrix;
  PetscInt                              n;

  PetscFunctionBegin;
  PetscCheck(cusp,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUSPARSE struct");
  PetscCheck(cusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUSPARSE CsrMatrix");
  if (!cusp->cooPerm) {
    CHKERRQ(MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY));
    CHKERRQ(MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY));
    PetscFunctionReturn(0);
  }
  matrix = (CsrMatrix*)cusp->mat->mat;
  PetscCheck(matrix->values,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  if (!v) {
    if (imode == INSERT_VALUES) thrust::fill(thrust::device,matrix->values->begin(),matrix->values->end(),0.);
    goto finalize;
  }
  n = cusp->cooPerm->size();
  if (isCudaMem(v)) {
    d_v = thrust::device_pointer_cast(v);
  } else {
    cooPerm_v = new THRUSTARRAY(n);
    cooPerm_v->assign(v,v+n);
    d_v = cooPerm_v->data();
    CHKERRQ(PetscLogCpuToGpu(n*sizeof(PetscScalar)));
  }
  CHKERRQ(PetscLogGpuTimeBegin());
  if (imode == ADD_VALUES) { /* ADD VALUES means add to existing ones */
    if (cusp->cooPerm_a) { /* there are repeated entries in d_v[], and we need to add these them */
      THRUSTARRAY *cooPerm_w = new THRUSTARRAY(matrix->values->size());
      auto vbit = thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin());
      /* thrust::reduce_by_key(keys_first,keys_last,values_first,keys_output,values_output)
        cooPerm_a = [0,0,1,2,3,4]. The length is n, number of nonozeros in d_v[].
        cooPerm_a is ordered. d_v[i] is the cooPerm_a[i]-th unique nonzero.
      */
      thrust::reduce_by_key(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),vbit,thrust::make_discard_iterator(),cooPerm_w->begin(),thrust::equal_to<PetscInt>(),thrust::plus<PetscScalar>());
      thrust::transform(cooPerm_w->begin(),cooPerm_w->end(),matrix->values->begin(),matrix->values->begin(),thrust::plus<PetscScalar>());
      delete cooPerm_w;
    } else {
      /* all nonzeros in d_v[] are unique entries */
      auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin()),
                                                                matrix->values->begin()));
      auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->end()),
                                                                matrix->values->end()));
      thrust::for_each(zibit,zieit,VecCUDAPlusEquals()); /* values[i] += d_v[cooPerm[i]]  */
    }
  } else {
    if (cusp->cooPerm_a) { /* repeated entries in COO, with INSERT_VALUES -> reduce */
      auto vbit = thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin());
      thrust::reduce_by_key(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),vbit,thrust::make_discard_iterator(),matrix->values->begin(),thrust::equal_to<PetscInt>(),thrust::plus<PetscScalar>());
    } else {
      auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->begin()),
                                                                matrix->values->begin()));
      auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v,cusp->cooPerm->end()),
                                                                matrix->values->end()));
      thrust::for_each(zibit,zieit,VecCUDAEquals());
    }
  }
  CHKERRQ(PetscLogGpuTimeEnd());
finalize:
  delete cooPerm_v;
  A->offloadmask = PETSC_OFFLOAD_GPU;
  CHKERRQ(PetscObjectStateIncrease((PetscObject)A));
  /* shorter version of MatAssemblyEnd_SeqAIJ */
  CHKERRQ(PetscInfo(A,"Matrix size: %" PetscInt_FMT " X %" PetscInt_FMT "; storage space: 0 unneeded,%" PetscInt_FMT " used\n",A->rmap->n,A->cmap->n,a->nz));
  CHKERRQ(PetscInfo(A,"Number of mallocs during MatSetValues() is 0\n"));
  CHKERRQ(PetscInfo(A,"Maximum nonzeros in any row is %" PetscInt_FMT "\n",a->rmax));
  a->reallocs         = 0;
  A->info.mallocs    += 0;
  A->info.nz_unneeded = 0;
  A->assembled = A->was_assembled = PETSC_TRUE;
  A->num_ass++;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqAIJCUSPARSEInvalidateTranspose(Mat A, PetscBool destroy)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  if (!cusp) PetscFunctionReturn(0);
  if (destroy) {
    CHKERRQ(MatSeqAIJCUSPARSEMultStruct_Destroy(&cusp->matTranspose,cusp->format));
    delete cusp->csr2csc_i;
    cusp->csr2csc_i = NULL;
  }
  A->transupdated = PETSC_FALSE;
  PetscFunctionReturn(0);
}

#include <thrust/binary_search.h>
/* 'Basic' means it only works when coo_i[] and coo_j[] do not contain negative indices */
PetscErrorCode MatSetPreallocationCOO_SeqAIJCUSPARSE_Basic(Mat A, PetscCount n, const PetscInt coo_i[], const PetscInt coo_j[])
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  PetscInt           cooPerm_n, nzr = 0;

  PetscFunctionBegin;
  CHKERRQ(PetscLayoutSetUp(A->rmap));
  CHKERRQ(PetscLayoutSetUp(A->cmap));
  cooPerm_n = cusp->cooPerm ? cusp->cooPerm->size() : 0;
  if (n != cooPerm_n) {
    delete cusp->cooPerm;
    delete cusp->cooPerm_a;
    cusp->cooPerm = NULL;
    cusp->cooPerm_a = NULL;
  }
  if (n) {
    THRUSTINTARRAY d_i(n);
    THRUSTINTARRAY d_j(n);
    THRUSTINTARRAY ii(A->rmap->n);

    if (!cusp->cooPerm)   { cusp->cooPerm   = new THRUSTINTARRAY(n); }
    if (!cusp->cooPerm_a) { cusp->cooPerm_a = new THRUSTINTARRAY(n); }

    CHKERRQ(PetscLogCpuToGpu(2.*n*sizeof(PetscInt)));
    d_i.assign(coo_i,coo_i+n);
    d_j.assign(coo_j,coo_j+n);

    /* Ex.
      n = 6
      coo_i = [3,3,1,4,1,4]
      coo_j = [3,2,2,5,2,6]
    */
    auto fkey = thrust::make_zip_iterator(thrust::make_tuple(d_i.begin(),d_j.begin()));
    auto ekey = thrust::make_zip_iterator(thrust::make_tuple(d_i.end(),d_j.end()));

    CHKERRQ(PetscLogGpuTimeBegin());
    thrust::sequence(thrust::device, cusp->cooPerm->begin(), cusp->cooPerm->end(), 0);
    thrust::sort_by_key(fkey, ekey, cusp->cooPerm->begin(), IJCompare()); /* sort by row, then by col */
    *cusp->cooPerm_a = d_i; /* copy the sorted array */
    THRUSTINTARRAY w = d_j;

    /*
      d_i     = [1,1,3,3,4,4]
      d_j     = [2,2,2,3,5,6]
      cooPerm = [2,4,1,0,3,5]
    */
    auto nekey = thrust::unique(fkey, ekey, IJEqual()); /* unique (d_i, d_j) */

    /*
      d_i     = [1,3,3,4,4,x]
                            ^ekey
      d_j     = [2,2,3,5,6,x]
                           ^nekye
    */
    if (nekey == ekey) { /* all entries are unique */
      delete cusp->cooPerm_a;
      cusp->cooPerm_a = NULL;
    } else { /* Stefano: I couldn't come up with a more elegant algorithm */
      /* idea: any change in i or j in the (i,j) sequence implies a new nonzero */
      adjacent_difference(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),cusp->cooPerm_a->begin(),IJDiff()); /* cooPerm_a: [1,1,3,3,4,4] => [1,0,1,0,1,0]*/
      adjacent_difference(w.begin(),w.end(),w.begin(),IJDiff());                                              /* w:         [2,2,2,3,5,6] => [2,0,0,1,1,1]*/
      (*cusp->cooPerm_a)[0] = 0; /* clear the first entry, though accessing an entry on device implies a hipMemcpy */
      w[0] = 0;
      thrust::transform(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),w.begin(),cusp->cooPerm_a->begin(),IJSum()); /* cooPerm_a =          [0,0,1,1,1,1]*/
      thrust::inclusive_scan(cusp->cooPerm_a->begin(),cusp->cooPerm_a->end(),cusp->cooPerm_a->begin(),thrust::plus<PetscInt>()); /*cooPerm_a=[0,0,1,2,3,4]*/
    }
    thrust::counting_iterator<PetscInt> search_begin(0);
    thrust::upper_bound(d_i.begin(), nekey.get_iterator_tuple().get<0>(), /* binary search entries of [0,1,2,3,4,5,6) in ordered array d_i = [1,3,3,4,4], supposing A->rmap->n = 6. */
                        search_begin, search_begin + A->rmap->n,  /* return in ii[] the index of last position in d_i[] where value could be inserted without violating the ordering */
                        ii.begin()); /* ii = [0,1,1,3,5,5]. A leading 0 will be added later */
    CHKERRQ(PetscLogGpuTimeEnd());

    CHKERRQ(MatSeqXAIJFreeAIJ(A,&a->a,&a->j,&a->i));
    a->singlemalloc = PETSC_FALSE;
    a->free_a       = PETSC_TRUE;
    a->free_ij      = PETSC_TRUE;
    CHKERRQ(PetscMalloc1(A->rmap->n+1,&a->i));
    a->i[0] = 0; /* a->i = [0,0,1,1,3,5,5] */
    CHKERRCUDA(hipMemcpy(a->i+1,ii.data().get(),A->rmap->n*sizeof(PetscInt),hipMemcpyDeviceToHost));
    a->nz = a->maxnz = a->i[A->rmap->n];
    a->rmax = 0;
    CHKERRQ(PetscMalloc1(a->nz,&a->a));
    CHKERRQ(PetscMalloc1(a->nz,&a->j));
    CHKERRCUDA(hipMemcpy(a->j,d_j.data().get(),a->nz*sizeof(PetscInt),hipMemcpyDeviceToHost));
    if (!a->ilen) CHKERRQ(PetscMalloc1(A->rmap->n,&a->ilen));
    if (!a->imax) CHKERRQ(PetscMalloc1(A->rmap->n,&a->imax));
    for (PetscInt i = 0; i < A->rmap->n; i++) {
      const PetscInt nnzr = a->i[i+1] - a->i[i];
      nzr += (PetscInt)!!(nnzr);
      a->ilen[i] = a->imax[i] = nnzr;
      a->rmax = PetscMax(a->rmax,nnzr);
    }
    a->nonzerorowcnt = nzr;
    A->preallocated = PETSC_TRUE;
    CHKERRQ(PetscLogGpuToCpu((A->rmap->n+a->nz)*sizeof(PetscInt)));
    CHKERRQ(MatMarkDiagonal_SeqAIJ(A));
  } else {
    CHKERRQ(MatSeqAIJSetPreallocation(A,0,NULL));
  }
  CHKERRQ(MatSetOption(A,MAT_NEW_NONZERO_ALLOCATION_ERR,PETSC_TRUE));

  /* We want to allocate the CUSPARSE struct for matvec now.
     The code is so convoluted now that I prefer to copy zeros */
  CHKERRQ(PetscArrayzero(a->a,a->nz));
  CHKERRQ(MatCheckCompressedRow(A,nzr,&a->compressedrow,a->i,A->rmap->n,0.6));
  A->offloadmask = PETSC_OFFLOAD_CPU;
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_TRUE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatSetPreallocationCOO_SeqAIJCUSPARSE(Mat mat, PetscCount coo_n, const PetscInt coo_i[], const PetscInt coo_j[])
{
  Mat_SeqAIJ         *seq;
  Mat_SeqAIJCUSPARSE *dev;
  PetscBool          coo_basic = PETSC_TRUE;
  PetscMemType       mtype = PETSC_MEMTYPE_DEVICE;

  PetscFunctionBegin;
  CHKERRQ(MatResetPreallocationCOO_SeqAIJ(mat));
  CHKERRQ(MatResetPreallocationCOO_SeqAIJCUSPARSE(mat));
  if (coo_i) {
    CHKERRQ(PetscGetMemType(coo_i,&mtype));
    if (PetscMemTypeHost(mtype)) {
      for (PetscCount k=0; k<coo_n; k++) {
        if (coo_i[k] < 0 || coo_j[k] < 0) {coo_basic = PETSC_FALSE; break;}
      }
    }
  }

  if (coo_basic) { /* i,j are on device or do not contain negative indices */
    CHKERRQ(MatSetPreallocationCOO_SeqAIJCUSPARSE_Basic(mat,coo_n,coo_i,coo_j));
  } else {
    CHKERRQ(MatSetPreallocationCOO_SeqAIJ(mat,coo_n,coo_i,coo_j));
    mat->offloadmask = PETSC_OFFLOAD_CPU;
    CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(mat));
    seq  = static_cast<Mat_SeqAIJ*>(mat->data);
    dev  = static_cast<Mat_SeqAIJCUSPARSE*>(mat->spptr);
    CHKERRCUDA(hipMalloc((void**)&dev->jmap_d,(seq->nz+1)*sizeof(PetscCount)));
    CHKERRCUDA(hipMemcpy(dev->jmap_d,seq->jmap,(seq->nz+1)*sizeof(PetscCount),hipMemcpyHostToDevice));
    CHKERRCUDA(hipMalloc((void**)&dev->perm_d,seq->Atot*sizeof(PetscCount)));
    CHKERRCUDA(hipMemcpy(dev->perm_d,seq->perm,seq->Atot*sizeof(PetscCount),hipMemcpyHostToDevice));
    dev->use_extended_coo = PETSC_TRUE;
  }
  PetscFunctionReturn(0);
}

__global__ void MatAddCOOValues(const PetscScalar kv[],PetscCount nnz,const PetscCount jmap[],const PetscCount perm[],InsertMode imode,PetscScalar a[])
{
  PetscCount        i = blockIdx.x*blockDim.x + threadIdx.x;
  const PetscCount  grid_size = gridDim.x * blockDim.x;
  for (; i<nnz; i+= grid_size) {
    PetscScalar sum = 0.0;
    for (PetscCount k=jmap[i]; k<jmap[i+1]; k++) sum += kv[perm[k]];
    a[i] = (imode == INSERT_VALUES? 0.0 : a[i]) + sum;
  }
}

PetscErrorCode MatSetValuesCOO_SeqAIJCUSPARSE(Mat A, const PetscScalar v[], InsertMode imode)
{
  Mat_SeqAIJ          *seq = (Mat_SeqAIJ*)A->data;
  Mat_SeqAIJCUSPARSE  *dev = (Mat_SeqAIJCUSPARSE*)A->spptr;
  PetscCount          Annz = seq->nz;
  PetscMemType        memtype;
  const PetscScalar   *v1 = v;
  PetscScalar         *Aa;

  PetscFunctionBegin;
  if (dev->use_extended_coo) {
    CHKERRQ(PetscGetMemType(v,&memtype));
    if (PetscMemTypeHost(memtype)) { /* If user gave v[] in host, we might need to copy it to device if any */
      CHKERRCUDA(hipMalloc((void**)&v1,seq->coo_n*sizeof(PetscScalar)));
      CHKERRCUDA(hipMemcpy((void*)v1,v,seq->coo_n*sizeof(PetscScalar),hipMemcpyHostToDevice));
    }

    if (imode == INSERT_VALUES) CHKERRQ(MatSeqAIJCUSPARSEGetArrayWrite(A,&Aa));
    else CHKERRQ(MatSeqAIJCUSPARSEGetArray(A,&Aa));

    if (Annz) {
      MatAddCOOValues<<<(Annz+255)/256,256>>>(v1,Annz,dev->jmap_d,dev->perm_d,imode,Aa);
      CHKERRCUDA(hipPeekAtLastError());
    }

    if (imode == INSERT_VALUES) CHKERRQ(MatSeqAIJCUSPARSERestoreArrayWrite(A,&Aa));
    else CHKERRQ(MatSeqAIJCUSPARSERestoreArray(A,&Aa));

    if (PetscMemTypeHost(memtype)) CHKERRCUDA(hipFree((void*)v1));
  } else {
    CHKERRQ(MatSetValuesCOO_SeqAIJCUSPARSE_Basic(A,v,imode));
  }
  PetscFunctionReturn(0);
}

/*@C
    MatSeqAIJCUSPARSEGetIJ - returns the device row storage i and j indices for MATSEQAIJCUSPARSE matrices.

   Not collective

    Input Parameters:
+   A - the matrix
-   compressed - PETSC_TRUE or PETSC_FALSE indicating the matrix data structure should be always returned in compressed form

    Output Parameters:
+   ia - the CSR row pointers
-   ja - the CSR column indices

    Level: developer

    Notes:
      When compressed is true, the CSR structure does not contain empty rows

.seealso: MatSeqAIJCUSPARSERestoreIJ(), MatSeqAIJCUSPARSEGetArrayRead()
@*/
PetscErrorCode MatSeqAIJCUSPARSEGetIJ(Mat A, PetscBool compressed, const int** i, const int **j)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  if (!i || !j) PetscFunctionReturn(0);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  PetscCheckFalse(cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  PetscCheck(cusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  if (i) {
    if (!compressed && a->compressedrow.use) { /* need full row offset */
      if (!cusp->rowoffsets_gpu) {
        cusp->rowoffsets_gpu  = new THRUSTINTARRAY32(A->rmap->n + 1);
        cusp->rowoffsets_gpu->assign(a->i,a->i + A->rmap->n + 1);
        CHKERRQ(PetscLogCpuToGpu((A->rmap->n + 1)*sizeof(PetscInt)));
      }
      *i = cusp->rowoffsets_gpu->data().get();
    } else *i = csr->row_offsets->data().get();
  }
  if (j) *j = csr->column_indices->data().get();
  PetscFunctionReturn(0);
}

/*@C
    MatSeqAIJCUSPARSERestoreIJ - restore the device row storage i and j indices obtained with MatSeqAIJCUSPARSEGetIJ()

   Not collective

    Input Parameters:
+   A - the matrix
-   compressed - PETSC_TRUE or PETSC_FALSE indicating the matrix data structure should be always returned in compressed form

    Output Parameters:
+   ia - the CSR row pointers
-   ja - the CSR column indices

    Level: developer

.seealso: MatSeqAIJCUSPARSEGetIJ()
@*/
PetscErrorCode MatSeqAIJCUSPARSERestoreIJ(Mat A, PetscBool compressed, const int** i, const int **j)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  if (i) *i = NULL;
  if (j) *j = NULL;
  PetscFunctionReturn(0);
}

/*@C
   MatSeqAIJCUSPARSEGetArrayRead - gives read-only access to the array where the device data for a MATSEQAIJCUSPARSE matrix is stored

   Not Collective

   Input Parameter:
.   A - a MATSEQAIJCUSPARSE matrix

   Output Parameter:
.   a - pointer to the device data

   Level: developer

   Notes: may trigger host-device copies if up-to-date matrix data is on host

.seealso: MatSeqAIJCUSPARSEGetArray(), MatSeqAIJCUSPARSEGetArrayWrite(), MatSeqAIJCUSPARSERestoreArrayRead()
@*/
PetscErrorCode MatSeqAIJCUSPARSEGetArrayRead(Mat A, const PetscScalar** a)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  PetscCheckFalse(cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  PetscCheck(cusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  PetscCheck(csr->values,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  *a = csr->values->data().get();
  PetscFunctionReturn(0);
}

/*@C
   MatSeqAIJCUSPARSERestoreArrayRead - restore the read-only access array obtained from MatSeqAIJCUSPARSEGetArrayRead()

   Not Collective

   Input Parameter:
.   A - a MATSEQAIJCUSPARSE matrix

   Output Parameter:
.   a - pointer to the device data

   Level: developer

.seealso: MatSeqAIJCUSPARSEGetArrayRead()
@*/
PetscErrorCode MatSeqAIJCUSPARSERestoreArrayRead(Mat A, const PetscScalar** a)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  *a = NULL;
  PetscFunctionReturn(0);
}

/*@C
   MatSeqAIJCUSPARSEGetArray - gives read-write access to the array where the device data for a MATSEQAIJCUSPARSE matrix is stored

   Not Collective

   Input Parameter:
.   A - a MATSEQAIJCUSPARSE matrix

   Output Parameter:
.   a - pointer to the device data

   Level: developer

   Notes: may trigger host-device copies if up-to-date matrix data is on host

.seealso: MatSeqAIJCUSPARSEGetArrayRead(), MatSeqAIJCUSPARSEGetArrayWrite(), MatSeqAIJCUSPARSERestoreArray()
@*/
PetscErrorCode MatSeqAIJCUSPARSEGetArray(Mat A, PetscScalar** a)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  PetscCheckFalse(cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
  PetscCheck(cusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  PetscCheck(csr->values,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  *a = csr->values->data().get();
  A->offloadmask = PETSC_OFFLOAD_GPU;
  CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_FALSE));
  PetscFunctionReturn(0);
}
/*@C
   MatSeqAIJCUSPARSERestoreArray - restore the read-write access array obtained from MatSeqAIJCUSPARSEGetArray()

   Not Collective

   Input Parameter:
.   A - a MATSEQAIJCUSPARSE matrix

   Output Parameter:
.   a - pointer to the device data

   Level: developer

.seealso: MatSeqAIJCUSPARSEGetArray()
@*/
PetscErrorCode MatSeqAIJCUSPARSERestoreArray(Mat A, PetscScalar** a)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  CHKERRQ(MatSeqAIJInvalidateDiagonal(A));
  CHKERRQ(PetscObjectStateIncrease((PetscObject)A));
  *a = NULL;
  PetscFunctionReturn(0);
}

/*@C
   MatSeqAIJCUSPARSEGetArrayWrite - gives write access to the array where the device data for a MATSEQAIJCUSPARSE matrix is stored

   Not Collective

   Input Parameter:
.   A - a MATSEQAIJCUSPARSE matrix

   Output Parameter:
.   a - pointer to the device data

   Level: developer

   Notes: does not trigger host-device copies and flags data validity on the GPU

.seealso: MatSeqAIJCUSPARSEGetArray(), MatSeqAIJCUSPARSEGetArrayRead(), MatSeqAIJCUSPARSERestoreArrayWrite()
@*/
PetscErrorCode MatSeqAIJCUSPARSEGetArrayWrite(Mat A, PetscScalar** a)
{
  Mat_SeqAIJCUSPARSE *cusp = (Mat_SeqAIJCUSPARSE*)A->spptr;
  CsrMatrix          *csr;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  PetscCheckFalse(cusp->format == MAT_CUSPARSE_ELL || cusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  PetscCheck(cusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
  csr = (CsrMatrix*)cusp->mat->mat;
  PetscCheck(csr->values,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing CUDA memory");
  *a = csr->values->data().get();
  A->offloadmask = PETSC_OFFLOAD_GPU;
  CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(A,PETSC_FALSE));
  PetscFunctionReturn(0);
}

/*@C
   MatSeqAIJCUSPARSERestoreArrayWrite - restore the write-only access array obtained from MatSeqAIJCUSPARSEGetArrayWrite()

   Not Collective

   Input Parameter:
.   A - a MATSEQAIJCUSPARSE matrix

   Output Parameter:
.   a - pointer to the device data

   Level: developer

.seealso: MatSeqAIJCUSPARSEGetArrayWrite()
@*/
PetscErrorCode MatSeqAIJCUSPARSERestoreArrayWrite(Mat A, PetscScalar** a)
{
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidPointer(a,2);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  CHKERRQ(MatSeqAIJInvalidateDiagonal(A));
  CHKERRQ(PetscObjectStateIncrease((PetscObject)A));
  *a = NULL;
  PetscFunctionReturn(0);
}

struct IJCompare4
{
  __host__ __device__
  inline bool operator() (const thrust::tuple<int, int, PetscScalar, int> &t1, const thrust::tuple<int, int, PetscScalar, int> &t2)
  {
    if (t1.get<0>() < t2.get<0>()) return true;
    if (t1.get<0>() == t2.get<0>()) return t1.get<1>() < t2.get<1>();
    return false;
  }
};

struct Shift
{
  int _shift;

  Shift(int shift) : _shift(shift) {}
  __host__ __device__
  inline int operator() (const int &c)
  {
    return c + _shift;
  }
};

/* merges two SeqAIJCUSPARSE matrices A, B by concatenating their rows. [A';B']' operation in matlab notation */
PetscErrorCode MatSeqAIJCUSPARSEMergeMats(Mat A,Mat B,MatReuse reuse,Mat* C)
{
  Mat_SeqAIJ                   *a = (Mat_SeqAIJ*)A->data, *b = (Mat_SeqAIJ*)B->data, *c;
  Mat_SeqAIJCUSPARSE           *Acusp = (Mat_SeqAIJCUSPARSE*)A->spptr, *Bcusp = (Mat_SeqAIJCUSPARSE*)B->spptr, *Ccusp;
  Mat_SeqAIJCUSPARSEMultStruct *Cmat;
  CsrMatrix                    *Acsr,*Bcsr,*Ccsr;
  PetscInt                     Annz,Bnnz;
  hipsparseStatus_t             stat;
  PetscInt                     i,m,n,zero = 0;

  PetscFunctionBegin;
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  PetscValidHeaderSpecific(B,MAT_CLASSID,2);
  PetscValidPointer(C,4);
  PetscCheckTypeName(A,MATSEQAIJCUSPARSE);
  PetscCheckTypeName(B,MATSEQAIJCUSPARSE);
  PetscCheck(A->rmap->n == B->rmap->n,PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Invalid number or rows %" PetscInt_FMT " != %" PetscInt_FMT,A->rmap->n,B->rmap->n);
  PetscCheckFalse(reuse == MAT_INPLACE_MATRIX,PETSC_COMM_SELF,PETSC_ERR_SUP,"MAT_INPLACE_MATRIX not supported");
  PetscCheckFalse(Acusp->format == MAT_CUSPARSE_ELL || Acusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  PetscCheckFalse(Bcusp->format == MAT_CUSPARSE_ELL || Bcusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
  if (reuse == MAT_INITIAL_MATRIX) {
    m     = A->rmap->n;
    n     = A->cmap->n + B->cmap->n;
    CHKERRQ(MatCreate(PETSC_COMM_SELF,C));
    CHKERRQ(MatSetSizes(*C,m,n,m,n));
    CHKERRQ(MatSetType(*C,MATSEQAIJCUSPARSE));
    c     = (Mat_SeqAIJ*)(*C)->data;
    Ccusp = (Mat_SeqAIJCUSPARSE*)(*C)->spptr;
    Cmat  = new Mat_SeqAIJCUSPARSEMultStruct;
    Ccsr  = new CsrMatrix;
    Cmat->cprowIndices      = NULL;
    c->compressedrow.use    = PETSC_FALSE;
    c->compressedrow.nrows  = 0;
    c->compressedrow.i      = NULL;
    c->compressedrow.rindex = NULL;
    Ccusp->workVector       = NULL;
    Ccusp->nrows    = m;
    Ccusp->mat      = Cmat;
    Ccusp->mat->mat = Ccsr;
    Ccsr->num_rows  = m;
    Ccsr->num_cols  = n;
    CHKERRCUSPARSE(hipsparseCreateMatDescr(&Cmat->descr));
    CHKERRCUSPARSE(hipsparseSetMatIndexBase(Cmat->descr, HIPSPARSE_INDEX_BASE_ZERO));
    CHKERRCUSPARSE(hipsparseSetMatType(Cmat->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHKERRCUDA(hipMalloc((void **)&(Cmat->alpha_one),sizeof(PetscScalar)));
    CHKERRCUDA(hipMalloc((void **)&(Cmat->beta_zero),sizeof(PetscScalar)));
    CHKERRCUDA(hipMalloc((void **)&(Cmat->beta_one), sizeof(PetscScalar)));
    CHKERRCUDA(hipMemcpy(Cmat->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
    CHKERRCUDA(hipMemcpy(Cmat->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice));
    CHKERRCUDA(hipMemcpy(Cmat->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
    CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
    CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(B));
    PetscCheck(Acusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
    PetscCheck(Bcusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");

    Acsr = (CsrMatrix*)Acusp->mat->mat;
    Bcsr = (CsrMatrix*)Bcusp->mat->mat;
    Annz = (PetscInt)Acsr->column_indices->size();
    Bnnz = (PetscInt)Bcsr->column_indices->size();
    c->nz = Annz + Bnnz;
    Ccsr->row_offsets = new THRUSTINTARRAY32(m+1);
    Ccsr->column_indices = new THRUSTINTARRAY32(c->nz);
    Ccsr->values = new THRUSTARRAY(c->nz);
    Ccsr->num_entries = c->nz;
    Ccusp->cooPerm = new THRUSTINTARRAY(c->nz);
    if (c->nz) {
      auto Acoo = new THRUSTINTARRAY32(Annz);
      auto Bcoo = new THRUSTINTARRAY32(Bnnz);
      auto Ccoo = new THRUSTINTARRAY32(c->nz);
      THRUSTINTARRAY32 *Aroff,*Broff;

      if (a->compressedrow.use) { /* need full row offset */
        if (!Acusp->rowoffsets_gpu) {
          Acusp->rowoffsets_gpu  = new THRUSTINTARRAY32(A->rmap->n + 1);
          Acusp->rowoffsets_gpu->assign(a->i,a->i + A->rmap->n + 1);
          CHKERRQ(PetscLogCpuToGpu((A->rmap->n + 1)*sizeof(PetscInt)));
        }
        Aroff = Acusp->rowoffsets_gpu;
      } else Aroff = Acsr->row_offsets;
      if (b->compressedrow.use) { /* need full row offset */
        if (!Bcusp->rowoffsets_gpu) {
          Bcusp->rowoffsets_gpu  = new THRUSTINTARRAY32(B->rmap->n + 1);
          Bcusp->rowoffsets_gpu->assign(b->i,b->i + B->rmap->n + 1);
          CHKERRQ(PetscLogCpuToGpu((B->rmap->n + 1)*sizeof(PetscInt)));
        }
        Broff = Bcusp->rowoffsets_gpu;
      } else Broff = Bcsr->row_offsets;
      CHKERRQ(PetscLogGpuTimeBegin());
      stat = hipsparseXcsr2coo(Acusp->handle,
                              Aroff->data().get(),
                              Annz,
                              m,
                              Acoo->data().get(),
                              HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
      stat = hipsparseXcsr2coo(Bcusp->handle,
                              Broff->data().get(),
                              Bnnz,
                              m,
                              Bcoo->data().get(),
                              HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
      /* Issues when using bool with large matrices on SUMMIT 10.2.89 */
      auto Aperm = thrust::make_constant_iterator(1);
      auto Bperm = thrust::make_constant_iterator(0);
#if PETSC_PKG_CUDA_VERSION_GE(10,0,0)
      auto Bcib = thrust::make_transform_iterator(Bcsr->column_indices->begin(),Shift(A->cmap->n));
      auto Bcie = thrust::make_transform_iterator(Bcsr->column_indices->end(),Shift(A->cmap->n));
#else
      /* there are issues instantiating the merge operation using a transform iterator for the columns of B */
      auto Bcib = Bcsr->column_indices->begin();
      auto Bcie = Bcsr->column_indices->end();
      thrust::transform(Bcib,Bcie,Bcib,Shift(A->cmap->n));
#endif
      auto wPerm = new THRUSTINTARRAY32(Annz+Bnnz);
      auto Azb = thrust::make_zip_iterator(thrust::make_tuple(Acoo->begin(),Acsr->column_indices->begin(),Acsr->values->begin(),Aperm));
      auto Aze = thrust::make_zip_iterator(thrust::make_tuple(Acoo->end(),Acsr->column_indices->end(),Acsr->values->end(),Aperm));
      auto Bzb = thrust::make_zip_iterator(thrust::make_tuple(Bcoo->begin(),Bcib,Bcsr->values->begin(),Bperm));
      auto Bze = thrust::make_zip_iterator(thrust::make_tuple(Bcoo->end(),Bcie,Bcsr->values->end(),Bperm));
      auto Czb = thrust::make_zip_iterator(thrust::make_tuple(Ccoo->begin(),Ccsr->column_indices->begin(),Ccsr->values->begin(),wPerm->begin()));
      auto p1 = Ccusp->cooPerm->begin();
      auto p2 = Ccusp->cooPerm->begin();
      thrust::advance(p2,Annz);
      PetscStackCallThrust(thrust::merge(thrust::device,Azb,Aze,Bzb,Bze,Czb,IJCompare4()));
#if PETSC_PKG_CUDA_VERSION_LT(10,0,0)
      thrust::transform(Bcib,Bcie,Bcib,Shift(-A->cmap->n));
#endif
      auto cci = thrust::make_counting_iterator(zero);
      auto cce = thrust::make_counting_iterator(c->nz);
#if 0 //Errors on SUMMIT cuda 11.1.0
      PetscStackCallThrust(thrust::partition_copy(thrust::device,cci,cce,wPerm->begin(),p1,p2,thrust::identity<int>()));
#else
      auto pred = thrust::identity<int>();
      PetscStackCallThrust(thrust::copy_if(thrust::device,cci,cce,wPerm->begin(),p1,pred));
      PetscStackCallThrust(thrust::remove_copy_if(thrust::device,cci,cce,wPerm->begin(),p2,pred));
#endif
      stat = hipsparseXcoo2csr(Ccusp->handle,
                              Ccoo->data().get(),
                              c->nz,
                              m,
                              Ccsr->row_offsets->data().get(),
                              HIPSPARSE_INDEX_BASE_ZERO);CHKERRCUSPARSE(stat);
      CHKERRQ(PetscLogGpuTimeEnd());
      delete wPerm;
      delete Acoo;
      delete Bcoo;
      delete Ccoo;
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
      stat = hipsparseCreateCsr(&Cmat->matDescr, Ccsr->num_rows, Ccsr->num_cols, Ccsr->num_entries,
                               Ccsr->row_offsets->data().get(), Ccsr->column_indices->data().get(), Ccsr->values->data().get(),
                               HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                               HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
#endif
      if (A->form_explicit_transpose && B->form_explicit_transpose) { /* if A and B have the transpose, generate C transpose too */
        CHKERRQ(MatSeqAIJCUSPARSEFormExplicitTranspose(A));
        CHKERRQ(MatSeqAIJCUSPARSEFormExplicitTranspose(B));
        PetscBool AT = Acusp->matTranspose ? PETSC_TRUE : PETSC_FALSE, BT = Bcusp->matTranspose ? PETSC_TRUE : PETSC_FALSE;
        Mat_SeqAIJCUSPARSEMultStruct *CmatT = new Mat_SeqAIJCUSPARSEMultStruct;
        CsrMatrix *CcsrT = new CsrMatrix;
        CsrMatrix *AcsrT = AT ? (CsrMatrix*)Acusp->matTranspose->mat : NULL;
        CsrMatrix *BcsrT = BT ? (CsrMatrix*)Bcusp->matTranspose->mat : NULL;

        (*C)->form_explicit_transpose = PETSC_TRUE;
        (*C)->transupdated = PETSC_TRUE;
        Ccusp->rowoffsets_gpu = NULL;
        CmatT->cprowIndices = NULL;
        CmatT->mat = CcsrT;
        CcsrT->num_rows = n;
        CcsrT->num_cols = m;
        CcsrT->num_entries = c->nz;

        CcsrT->row_offsets = new THRUSTINTARRAY32(n+1);
        CcsrT->column_indices = new THRUSTINTARRAY32(c->nz);
        CcsrT->values = new THRUSTARRAY(c->nz);

        CHKERRQ(PetscLogGpuTimeBegin());
        auto rT = CcsrT->row_offsets->begin();
        if (AT) {
          rT = thrust::copy(AcsrT->row_offsets->begin(),AcsrT->row_offsets->end(),rT);
          thrust::advance(rT,-1);
        }
        if (BT) {
          auto titb = thrust::make_transform_iterator(BcsrT->row_offsets->begin(),Shift(a->nz));
          auto tite = thrust::make_transform_iterator(BcsrT->row_offsets->end(),Shift(a->nz));
          thrust::copy(titb,tite,rT);
        }
        auto cT = CcsrT->column_indices->begin();
        if (AT) cT = thrust::copy(AcsrT->column_indices->begin(),AcsrT->column_indices->end(),cT);
        if (BT) thrust::copy(BcsrT->column_indices->begin(),BcsrT->column_indices->end(),cT);
        auto vT = CcsrT->values->begin();
        if (AT) vT = thrust::copy(AcsrT->values->begin(),AcsrT->values->end(),vT);
        if (BT) thrust::copy(BcsrT->values->begin(),BcsrT->values->end(),vT);
        CHKERRQ(PetscLogGpuTimeEnd());

        CHKERRCUSPARSE(hipsparseCreateMatDescr(&CmatT->descr));
        CHKERRCUSPARSE(hipsparseSetMatIndexBase(CmatT->descr, HIPSPARSE_INDEX_BASE_ZERO));
        CHKERRCUSPARSE(hipsparseSetMatType(CmatT->descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        CHKERRCUDA(hipMalloc((void **)&(CmatT->alpha_one),sizeof(PetscScalar)));
        CHKERRCUDA(hipMalloc((void **)&(CmatT->beta_zero),sizeof(PetscScalar)));
        CHKERRCUDA(hipMalloc((void **)&(CmatT->beta_one), sizeof(PetscScalar)));
        CHKERRCUDA(hipMemcpy(CmatT->alpha_one,&PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
        CHKERRCUDA(hipMemcpy(CmatT->beta_zero,&PETSC_CUSPARSE_ZERO,sizeof(PetscScalar),hipMemcpyHostToDevice));
        CHKERRCUDA(hipMemcpy(CmatT->beta_one, &PETSC_CUSPARSE_ONE, sizeof(PetscScalar),hipMemcpyHostToDevice));
#if PETSC_PKG_CUDA_VERSION_GE(11,0,0)
        stat = hipsparseCreateCsr(&CmatT->matDescr, CcsrT->num_rows, CcsrT->num_cols, CcsrT->num_entries,
                                 CcsrT->row_offsets->data().get(), CcsrT->column_indices->data().get(), CcsrT->values->data().get(),
                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                 HIPSPARSE_INDEX_BASE_ZERO, cusparse_scalartype);CHKERRCUSPARSE(stat);
#endif
        Ccusp->matTranspose = CmatT;
      }
    }

    c->singlemalloc = PETSC_FALSE;
    c->free_a       = PETSC_TRUE;
    c->free_ij      = PETSC_TRUE;
    CHKERRQ(PetscMalloc1(m+1,&c->i));
    CHKERRQ(PetscMalloc1(c->nz,&c->j));
    if (PetscDefined(USE_64BIT_INDICES)) { /* 32 to 64 bit conversion on the GPU and then copy to host (lazy) */
      THRUSTINTARRAY ii(Ccsr->row_offsets->size());
      THRUSTINTARRAY jj(Ccsr->column_indices->size());
      ii   = *Ccsr->row_offsets;
      jj   = *Ccsr->column_indices;
      CHKERRCUDA(hipMemcpy(c->i,ii.data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
      CHKERRCUDA(hipMemcpy(c->j,jj.data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
    } else {
      CHKERRCUDA(hipMemcpy(c->i,Ccsr->row_offsets->data().get(),Ccsr->row_offsets->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
      CHKERRCUDA(hipMemcpy(c->j,Ccsr->column_indices->data().get(),Ccsr->column_indices->size()*sizeof(PetscInt),hipMemcpyDeviceToHost));
    }
    CHKERRQ(PetscLogGpuToCpu((Ccsr->column_indices->size() + Ccsr->row_offsets->size())*sizeof(PetscInt)));
    CHKERRQ(PetscMalloc1(m,&c->ilen));
    CHKERRQ(PetscMalloc1(m,&c->imax));
    c->maxnz = c->nz;
    c->nonzerorowcnt = 0;
    c->rmax = 0;
    for (i = 0; i < m; i++) {
      const PetscInt nn = c->i[i+1] - c->i[i];
      c->ilen[i] = c->imax[i] = nn;
      c->nonzerorowcnt += (PetscInt)!!nn;
      c->rmax = PetscMax(c->rmax,nn);
    }
    CHKERRQ(MatMarkDiagonal_SeqAIJ(*C));
    CHKERRQ(PetscMalloc1(c->nz,&c->a));
    (*C)->nonzerostate++;
    CHKERRQ(PetscLayoutSetUp((*C)->rmap));
    CHKERRQ(PetscLayoutSetUp((*C)->cmap));
    Ccusp->nonzerostate = (*C)->nonzerostate;
    (*C)->preallocated  = PETSC_TRUE;
  } else {
    PetscCheckFalse((*C)->rmap->n != B->rmap->n,PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Invalid number or rows %" PetscInt_FMT " != %" PetscInt_FMT,(*C)->rmap->n,B->rmap->n);
    c = (Mat_SeqAIJ*)(*C)->data;
    if (c->nz) {
      Ccusp = (Mat_SeqAIJCUSPARSE*)(*C)->spptr;
      PetscCheck(Ccusp->cooPerm,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing cooPerm");
      PetscCheckFalse(Ccusp->format == MAT_CUSPARSE_ELL || Ccusp->format == MAT_CUSPARSE_HYB,PETSC_COMM_SELF,PETSC_ERR_SUP,"Not implemented");
      PetscCheckFalse(Ccusp->nonzerostate != (*C)->nonzerostate,PETSC_COMM_SELF,PETSC_ERR_COR,"Wrong nonzerostate");
      CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(A));
      CHKERRQ(MatSeqAIJCUSPARSECopyToGPU(B));
      PetscCheck(Acusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
      PetscCheck(Bcusp->mat,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing Mat_SeqAIJCUSPARSEMultStruct");
      Acsr = (CsrMatrix*)Acusp->mat->mat;
      Bcsr = (CsrMatrix*)Bcusp->mat->mat;
      Ccsr = (CsrMatrix*)Ccusp->mat->mat;
      PetscCheckFalse(Acsr->num_entries != (PetscInt)Acsr->values->size(),PETSC_COMM_SELF,PETSC_ERR_COR,"A nnz %" PetscInt_FMT " != %" PetscInt_FMT,Acsr->num_entries,(PetscInt)Acsr->values->size());
      PetscCheckFalse(Bcsr->num_entries != (PetscInt)Bcsr->values->size(),PETSC_COMM_SELF,PETSC_ERR_COR,"B nnz %" PetscInt_FMT " != %" PetscInt_FMT,Bcsr->num_entries,(PetscInt)Bcsr->values->size());
      PetscCheckFalse(Ccsr->num_entries != (PetscInt)Ccsr->values->size(),PETSC_COMM_SELF,PETSC_ERR_COR,"C nnz %" PetscInt_FMT " != %" PetscInt_FMT,Ccsr->num_entries,(PetscInt)Ccsr->values->size());
      PetscCheckFalse(Ccsr->num_entries != Acsr->num_entries + Bcsr->num_entries,PETSC_COMM_SELF,PETSC_ERR_COR,"C nnz %" PetscInt_FMT " != %" PetscInt_FMT " + %" PetscInt_FMT,Ccsr->num_entries,Acsr->num_entries,Bcsr->num_entries);
      PetscCheck(Ccusp->cooPerm->size() == Ccsr->values->size(),PETSC_COMM_SELF,PETSC_ERR_COR,"permSize %" PetscInt_FMT " != %" PetscInt_FMT,(PetscInt)Ccusp->cooPerm->size(),(PetscInt)Ccsr->values->size());
      auto pmid = Ccusp->cooPerm->begin();
      thrust::advance(pmid,Acsr->num_entries);
      CHKERRQ(PetscLogGpuTimeBegin());
      auto zibait = thrust::make_zip_iterator(thrust::make_tuple(Acsr->values->begin(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),Ccusp->cooPerm->begin())));
      auto zieait = thrust::make_zip_iterator(thrust::make_tuple(Acsr->values->end(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),pmid)));
      thrust::for_each(zibait,zieait,VecCUDAEquals());
      auto zibbit = thrust::make_zip_iterator(thrust::make_tuple(Bcsr->values->begin(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),pmid)));
      auto ziebit = thrust::make_zip_iterator(thrust::make_tuple(Bcsr->values->end(),
                                                                 thrust::make_permutation_iterator(Ccsr->values->begin(),Ccusp->cooPerm->end())));
      thrust::for_each(zibbit,ziebit,VecCUDAEquals());
      CHKERRQ(MatSeqAIJCUSPARSEInvalidateTranspose(*C,PETSC_FALSE));
      if (A->form_explicit_transpose && B->form_explicit_transpose && (*C)->form_explicit_transpose) {
        PetscCheck(Ccusp->matTranspose,PETSC_COMM_SELF,PETSC_ERR_COR,"Missing transpose Mat_SeqAIJCUSPARSEMultStruct");
        PetscBool AT = Acusp->matTranspose ? PETSC_TRUE : PETSC_FALSE, BT = Bcusp->matTranspose ? PETSC_TRUE : PETSC_FALSE;
        CsrMatrix *AcsrT = AT ? (CsrMatrix*)Acusp->matTranspose->mat : NULL;
        CsrMatrix *BcsrT = BT ? (CsrMatrix*)Bcusp->matTranspose->mat : NULL;
        CsrMatrix *CcsrT = (CsrMatrix*)Ccusp->matTranspose->mat;
        auto vT = CcsrT->values->begin();
        if (AT) vT = thrust::copy(AcsrT->values->begin(),AcsrT->values->end(),vT);
        if (BT) thrust::copy(BcsrT->values->begin(),BcsrT->values->end(),vT);
        (*C)->transupdated = PETSC_TRUE;
      }
      CHKERRQ(PetscLogGpuTimeEnd());
    }
  }
  CHKERRQ(PetscObjectStateIncrease((PetscObject)*C));
  (*C)->assembled     = PETSC_TRUE;
  (*C)->was_assembled = PETSC_FALSE;
  (*C)->offloadmask   = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSeqAIJCopySubArray_SeqAIJCUSPARSE(Mat A, PetscInt n, const PetscInt idx[], PetscScalar v[])
{
  bool              dmem;
  const PetscScalar *av;

  PetscFunctionBegin;
  dmem = isCudaMem(v);
  CHKERRQ(MatSeqAIJCUSPARSEGetArrayRead(A,&av));
  if (n && idx) {
    THRUSTINTARRAY widx(n);
    widx.assign(idx,idx+n);
    CHKERRQ(PetscLogCpuToGpu(n*sizeof(PetscInt)));

    THRUSTARRAY *w = NULL;
    thrust::device_ptr<PetscScalar> dv;
    if (dmem) {
      dv = thrust::device_pointer_cast(v);
    } else {
      w = new THRUSTARRAY(n);
      dv = w->data();
    }
    thrust::device_ptr<const PetscScalar> dav = thrust::device_pointer_cast(av);

    auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(dav,widx.begin()),dv));
    auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(dav,widx.end()),dv+n));
    thrust::for_each(zibit,zieit,VecCUDAEquals());
    if (w) {
      CHKERRCUDA(hipMemcpy(v,w->data().get(),n*sizeof(PetscScalar),hipMemcpyDeviceToHost));
    }
    delete w;
  } else {
    CHKERRCUDA(hipMemcpy(v,av,n*sizeof(PetscScalar),dmem ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost));
  }
  if (!dmem) CHKERRQ(PetscLogCpuToGpu(n*sizeof(PetscScalar)));
  CHKERRQ(MatSeqAIJCUSPARSERestoreArrayRead(A,&av));
  PetscFunctionReturn(0);
}
