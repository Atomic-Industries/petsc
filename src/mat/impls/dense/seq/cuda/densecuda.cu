#include "hip/hip_runtime.h"
/*
     Defines the matrix operations for sequential dense with CUDA
*/
#include <petscpkg_version.h>
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1
#include <../src/mat/impls/dense/seq/dense.h> /*I "petscmat.h" I*/
#include <petsc/private/cudavecimpl.h> /* cublas definitions are here */

#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnCpotrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnCpotrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnCpotrs((a),(b),(c),(d),(hipComplex*)(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnCpotri((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnCpotri_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnCsytrf((a),(b),(c),(hipComplex*)(d),(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnCsytrf_bufferSize((a),(b),(hipComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnCgetrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnCgetrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnCgetrs((a),(b),(c),(d),(hipComplex*)(e),(f),(g),(hipComplex*)(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnCgeqrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnCgeqrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnCunmqr_bufferSize((a),(b),(c),(d),(e),(f),(hipComplex*)(g),(h),(hipComplex*)(i),(hipComplex*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnCunmqr((a),(b),(c),(d),(e),(f),(hipComplex*)(g),(h),(hipComplex*)(i),(hipComplex*)(j),(k),(hipComplex*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasCtrsm((a),(b),(c),(d),(e),(f),(g),(hipComplex*)(h),(hipComplex*)(i),(j),(hipComplex*)(k),(l))
#else /* complex double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnZpotrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnZpotrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnZpotrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnZpotri((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnZpotri_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnZsytrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnZsytrf_bufferSize((a),(b),(hipDoubleComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnZgetrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnZgetrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnZgetrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(g),(hipDoubleComplex*)(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnZgeqrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnZgeqrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnZunmqr_bufferSize((a),(b),(c),(d),(e),(f),(hipDoubleComplex*)(g),(h),(hipDoubleComplex*)(i),(hipDoubleComplex*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnZunmqr((a),(b),(c),(d),(e),(f),(hipDoubleComplex*)(g),(h),(hipDoubleComplex*)(i),(hipDoubleComplex*)(j),(k),(hipDoubleComplex*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasZtrsm((a),(b),(c),(d),(e),(f),(g),(hipDoubleComplex*)(h),(hipDoubleComplex*)(i),(j),(hipDoubleComplex*)(k),(l))
#endif
#else /* real single */
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnSpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnSpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnSpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnSpotri((a),(b),(c),(d),(e),(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnSpotri_bufferSize((a),(b),(c),(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnSsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnSsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnSgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnSgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnSgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnSgeqrf_bufferSize((a),(b),(c),(float*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnSgeqrf((a),(b),(c),(float*)(d),(e),(float*)(f),(float*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnSormqr_bufferSize((a),(b),(c),(d),(e),(f),(float*)(g),(h),(float*)(i),(float*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnSormqr((a),(b),(c),(d),(e),(f),(float*)(g),(h),(float*)(i),(float*)(j),(k),(float*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasStrsm((a),(b),(c),(d),(e),(f),(g),(float*)(h),(float*)(i),(j),(float*)(k),(l))
#else /* real double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnDpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnDpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnDpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXpotri(a,b,c,d,e,f,g,h)        hipsolverDnDpotri((a),(b),(c),(d),(e),(f),(g),(h))
#define cusolverDnXpotri_bufferSize(a,b,c,d,e,f) hipsolverDnDpotri_bufferSize((a),(b),(c),(d),(e),(f))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnDsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnDsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnDgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnDgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnDgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#define hipsolverDnXgeqrf_bufferSize(a,b,c,d,e,f) hipsolverDnDgeqrf_bufferSize((a),(b),(c),(double*)(d),(e),(f))
#define hipsolverDnXgeqrf(a,b,c,d,e,f,g,h,i)      hipsolverDnDgeqrf((a),(b),(c),(double*)(d),(e),(double*)(f),(double*)(g),(h),(i))
#define cusolverDnXormqr_bufferSize(a,b,c,d,e,f,g,h,i,j,k,l) hipsolverDnDormqr_bufferSize((a),(b),(c),(d),(e),(f),(double*)(g),(h),(double*)(i),(double*)(j),(k),(l))
#define cusolverDnXormqr(a,b,c,d,e,f,g,h,i,j,k,l,m,n) hipsolverDnDormqr((a),(b),(c),(d),(e),(f),(double*)(g),(h),(double*)(i),(double*)(j),(k),(double*)(l),(m),(n))
#define cublasXtrsm(a,b,c,d,e,f,g,h,i,j,k,l)     hipblasDtrsm((a),(b),(c),(d),(e),(f),(g),(double*)(h),(double*)(i),(j),(double*)(k),(l))
#endif
#endif

typedef struct {
  PetscScalar *d_v; /* pointer to the matrix on the GPU */
  PetscBool   user_alloc;
  PetscScalar *unplacedarray; /* if one called MatCUDADensePlaceArray(), this is where it stashed the original */
  PetscBool   unplaced_user_alloc;
  /* factorization support */
  PetscCuBLASInt *d_fact_ipiv; /* device pivots */
  PetscScalar *d_fact_tau;  /* device QR tau vector */
  PetscScalar *d_fact_work; /* device workspace */
  PetscCuBLASInt fact_lwork;
  PetscCuBLASInt *d_fact_info; /* device info */
  /* workspace */
  Vec         workvec;
} Mat_SeqDenseCUDA;

PetscErrorCode MatSeqDenseCUDASetPreallocation(Mat A, PetscScalar *d_data)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        iscuda;

  PetscFunctionBegin;
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&iscuda));
  if (!iscuda) PetscFunctionReturn(0);
  CHKERRQ(PetscLayoutSetUp(A->rmap));
  CHKERRQ(PetscLayoutSetUp(A->cmap));
  /* it may happen CPU preallocation has not been performed */
  if (cA->lda <= 0) cA->lda = A->rmap->n;
  if (!dA->user_alloc) CHKERRCUDA(hipFree(dA->d_v));
  if (!d_data) { /* petsc-allocated storage */
    size_t sz;

    CHKERRQ(PetscIntMultError(cA->lda,A->cmap->n,NULL));
    sz   = cA->lda*A->cmap->n*sizeof(PetscScalar);
    CHKERRCUDA(hipMalloc((void**)&dA->d_v,sz));
    CHKERRCUDA(hipMemset(dA->d_v,0,sz));
    dA->user_alloc = PETSC_FALSE;
  } else { /* user-allocated storage */
    dA->d_v        = d_data;
    dA->user_alloc = PETSC_TRUE;
  }
  A->offloadmask  = PETSC_OFFLOAD_GPU;
  A->preallocated = PETSC_TRUE;
  A->assembled    = PETSC_TRUE;
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyFromGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQDENSECUDA);
  CHKERRQ(PetscInfo(A,"%s matrix %d x %d\n",A->offloadmask == PETSC_OFFLOAD_GPU ? "Copy" : "Reusing",A->rmap->n,A->cmap->n));
  if (A->offloadmask == PETSC_OFFLOAD_GPU) {
    if (!cA->v) { /* MatCreateSeqDenseCUDA may not allocate CPU memory. Allocate if needed */
      CHKERRQ(MatSeqDenseSetPreallocation(A,NULL));
    }
    CHKERRQ(PetscLogEventBegin(MAT_DenseCopyFromGPU,A,0,0,0));
    if (cA->lda > A->rmap->n) {
      CHKERRCUDA(hipMemcpy2D(cA->v,cA->lda*sizeof(PetscScalar),dA->d_v,cA->lda*sizeof(PetscScalar),A->rmap->n*sizeof(PetscScalar),A->cmap->n,hipMemcpyDeviceToHost));
    } else {
      CHKERRCUDA(hipMemcpy(cA->v,dA->d_v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyDeviceToHost));
    }
    CHKERRQ(PetscLogGpuToCpu(cA->lda*sizeof(PetscScalar)*A->cmap->n));
    CHKERRQ(PetscLogEventEnd(MAT_DenseCopyFromGPU,A,0,0,0));

    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyToGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        copy;

  PetscFunctionBegin;
  PetscCheckTypeName(A,MATSEQDENSECUDA);
  if (A->boundtocpu) PetscFunctionReturn(0);
  copy = (PetscBool)(A->offloadmask == PETSC_OFFLOAD_CPU || A->offloadmask == PETSC_OFFLOAD_UNALLOCATED);
  CHKERRQ(PetscInfo(A,"%s matrix %d x %d\n",copy ? "Copy" : "Reusing",A->rmap->n,A->cmap->n));
  if (copy) {
    if (!dA->d_v) { /* Allocate GPU memory if not present */
      CHKERRQ(MatSeqDenseCUDASetPreallocation(A,NULL));
    }
    CHKERRQ(PetscLogEventBegin(MAT_DenseCopyToGPU,A,0,0,0));
    if (cA->lda > A->rmap->n) {
      CHKERRCUDA(hipMemcpy2D(dA->d_v,cA->lda*sizeof(PetscScalar),cA->v,cA->lda*sizeof(PetscScalar),A->rmap->n*sizeof(PetscScalar),A->cmap->n,hipMemcpyHostToDevice));
    } else {
      CHKERRCUDA(hipMemcpy(dA->d_v,cA->v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyHostToDevice));
    }
    CHKERRQ(PetscLogCpuToGpu(cA->lda*sizeof(PetscScalar)*A->cmap->n));
    CHKERRQ(PetscLogEventEnd(MAT_DenseCopyToGPU,A,0,0,0));

    A->offloadmask = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCopy_SeqDenseCUDA(Mat A,Mat B,MatStructure str)
{
  PetscErrorCode    ierr;
  const PetscScalar *va;
  PetscScalar       *vb;
  PetscInt          lda1,lda2,m=A->rmap->n,n=A->cmap->n;
  hipError_t       cerr;

  PetscFunctionBegin;
  /* If the two matrices don't have the same copy implementation, they aren't compatible for fast copy. */
  if (A->ops->copy != B->ops->copy) {
    CHKERRQ(MatCopy_Basic(A,B,str));
    PetscFunctionReturn(0);
  }
  PetscCheckFalse(m != B->rmap->n || n != B->cmap->n,PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"size(B) != size(A)");
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&va));
  CHKERRQ(MatDenseCUDAGetArrayWrite(B,&vb));
  CHKERRQ(MatDenseGetLDA(A,&lda1));
  CHKERRQ(MatDenseGetLDA(B,&lda2));
  CHKERRQ(PetscLogGpuTimeBegin());
  if (lda1>m || lda2>m) {
    CHKERRCUDA(hipMemcpy2D(vb,lda2*sizeof(PetscScalar),va,lda1*sizeof(PetscScalar),m*sizeof(PetscScalar),n,hipMemcpyDeviceToDevice));
  } else {
    CHKERRCUDA(hipMemcpy(vb,va,m*(n*sizeof(PetscScalar)),hipMemcpyDeviceToDevice));
  }
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArrayWrite(B,&vb));
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&va));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatZeroEntries_SeqDenseCUDA(Mat A)
{
  PetscErrorCode ierr;
  PetscScalar    *va;
  PetscInt       lda,m = A->rmap->n,n = A->cmap->n;
  hipError_t    cerr;

  PetscFunctionBegin;
  CHKERRQ(MatDenseCUDAGetArrayWrite(A,&va));
  CHKERRQ(MatDenseGetLDA(A,&lda));
  CHKERRQ(PetscLogGpuTimeBegin());
  if (lda>m) {
    CHKERRCUDA(hipMemset2D(va,lda*sizeof(PetscScalar),0,m*sizeof(PetscScalar),n));
  } else {
    CHKERRCUDA(hipMemset(va,0,m*(n*sizeof(PetscScalar))));
  }
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArrayWrite(A,&va));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAPlaceArray_SeqDenseCUDA(Mat A, const PetscScalar *a)
{
  Mat_SeqDense     *aa = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  PetscCheck(!aa->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!aa->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  PetscCheck(!dA->unplacedarray,PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
  if (aa->v) CHKERRQ(MatSeqDenseCUDACopyToGPU(A));
  dA->unplacedarray = dA->d_v;
  dA->unplaced_user_alloc = dA->user_alloc;
  dA->d_v = (PetscScalar*)a;
  dA->user_alloc = PETSC_TRUE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAResetArray_SeqDenseCUDA(Mat A)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  PetscCheck(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (a->v) CHKERRQ(MatSeqDenseCUDACopyToGPU(A));
  dA->d_v = dA->unplacedarray;
  dA->user_alloc = dA->unplaced_user_alloc;
  dA->unplacedarray = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAReplaceArray_SeqDenseCUDA(Mat A, const PetscScalar *a)
{
  Mat_SeqDense     *aa = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  PetscCheck(!aa->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!aa->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  PetscCheck(!dA->unplacedarray,PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
  if (!dA->user_alloc) CHKERRCUDA(hipFree(dA->d_v));
  dA->d_v = (PetscScalar*)a;
  dA->user_alloc = PETSC_FALSE;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArrayWrite_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  if (!dA->d_v) {
    CHKERRQ(MatSeqDenseCUDASetPreallocation(A,NULL));
  }
  *a = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArrayWrite_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  if (a) *a = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArrayRead_SeqDenseCUDA(Mat A, const PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqDenseCUDACopyToGPU(A));
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArrayRead_SeqDenseCUDA(Mat A, const PetscScalar **a)
{
  PetscFunctionBegin;
  if (a) *a = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDAGetArray_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  CHKERRQ(MatSeqDenseCUDACopyToGPU(A));
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseCUDARestoreArray_SeqDenseCUDA(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  if (a) *a = NULL;
  PetscFunctionReturn(0);
}

PETSC_EXTERN PetscErrorCode MatSeqDenseCUDAInvertFactors_Private(Mat A)
{
#if PETSC_PKG_CUDA_VERSION_GE(10,1,0)
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  hipsolverHandle_t handle;
  PetscCuBLASInt     n,lda;
#if defined(PETSC_USE_DEBUG)
  PetscCuBLASInt     info;
#endif

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&n));
  CHKERRQ(PetscCuBLASIntCast(a->lda,&lda));
  PetscCheckFalse(A->factortype == MAT_FACTOR_LU,PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDngetri not implemented");
  if (A->factortype == MAT_FACTOR_CHOLESKY) {
    if (!dA->d_fact_ipiv) { /* spd */
      PetscCuBLASInt il;

      CHKERRQ(MatDenseCUDAGetArray(A,&da));
      CHKERRCUSOLVER(cusolverDnXpotri_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&il));
      if (il > dA->fact_lwork) {
        dA->fact_lwork = il;

        CHKERRCUDA(hipFree(dA->d_fact_work));
        CHKERRCUDA(hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work)));
      }
      CHKERRQ(PetscLogGpuTimeBegin());
      CHKERRCUSOLVER(cusolverDnXpotri(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info));
      CHKERRQ(PetscLogGpuTimeEnd());
      CHKERRQ(MatDenseCUDARestoreArray(A,&da));
      /* TODO (write cuda kernel) */
      CHKERRQ(MatSeqDenseSymmetrize_Private(A,PETSC_TRUE));
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytri not implemented");
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Not implemented");
#if defined(PETSC_USE_DEBUG)
  CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
  PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: leading minor of order %d is zero",info);
  PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  CHKERRQ(PetscLogGpuFlops(1.0*n*n*n/3.0));
  A->ops->solve          = NULL;
  A->ops->solvetranspose = NULL;
  A->ops->matsolve       = NULL;
  A->factortype          = MAT_FACTOR_NONE;

  CHKERRQ(PetscFree(A->solvertype));
  PetscFunctionReturn(0);
#else
  SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Upgrade to CUDA version 10.1.0 or higher");
#endif
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal(Mat A, Vec xx, Vec yy, PetscBool transpose,
                                                     PetscErrorCode (*matsolve)(Mat,PetscScalar*,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscBool))
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar      *y;
  PetscCuBLASInt   m=0, k=0;
  PetscBool        xiscuda, yiscuda, aiscuda;

  PetscFunctionBegin;
  PetscCheckFalse(A->factortype == MAT_FACTOR_NONE,PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&k));
  CHKERRQ(PetscObjectTypeCompare((PetscObject)xx,VECSEQCUDA,&xiscuda));
  CHKERRQ(PetscObjectTypeCompare((PetscObject)yy,VECSEQCUDA,&yiscuda));
  {
    const PetscScalar *x;
    PetscBool xishost = PETSC_TRUE;

    /* The logic here is to try to minimize the amount of memory copying:
       if we call VecCUDAGetArrayRead(X,&x) every time xiscuda and the
       data is not offloaded to the GPU yet, then the data is copied to the
       GPU.  But we are only trying to get the data in order to copy it into the y
       array.  So the array x will be wherever the data already is so that
       only one memcpy is performed */
    if (xiscuda && xx->offloadmask & PETSC_OFFLOAD_GPU) {
      CHKERRQ(VecCUDAGetArrayRead(xx, &x));
      xishost =  PETSC_FALSE;
    } else {
      CHKERRQ(VecGetArrayRead(xx, &x));
    }
    if (k < m || !yiscuda) {
      if (!dA->workvec) {
        CHKERRQ(VecCreateSeqCUDA(PetscObjectComm((PetscObject)A), m, &(dA->workvec)));
      }
      CHKERRQ(VecCUDAGetArrayWrite(dA->workvec, &y));
    } else {
      CHKERRQ(VecCUDAGetArrayWrite(yy,&y));
    }
    CHKERRCUDA(hipMemcpy(y,x,m*sizeof(PetscScalar),xishost ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
  }
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&aiscuda));
  if (!aiscuda) {
    CHKERRQ(MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A));
  }
  CHKERRQ((*matsolve) (A, y, m, m, 1, k, transpose));
  if (!aiscuda) {
    CHKERRQ(MatConvert(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A));
  }
  if (k < m || !yiscuda) {
    PetscScalar *yv;

    /* The logic here is that the data is not yet in either yy's GPU array or its
       CPU array.  There is nothing in the interface to say where the user would like
       it to end up.  So we choose the GPU, because it is the faster option */
    if (yiscuda) {
      CHKERRQ(VecCUDAGetArrayWrite(yy,&yv));
    } else {
      CHKERRQ(VecGetArray(yy,&yv));
    }
    CHKERRCUDA(hipMemcpy(yv,y,k*sizeof(PetscScalar),yiscuda ? hipMemcpyDeviceToDevice: hipMemcpyDeviceToHost));
    if (yiscuda) {
      CHKERRQ(VecCUDARestoreArrayWrite(yy,&yv));
    } else {
      CHKERRQ(VecRestoreArray(yy,&yv));
    }
    CHKERRQ(VecCUDARestoreArrayWrite(dA->workvec, &y));
  } else {
    CHKERRQ(VecCUDARestoreArrayWrite(yy,&y));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_Internal(Mat A, Mat B, Mat X, PetscBool transpose,
                                                        PetscErrorCode (*matsolve)(Mat,PetscScalar*,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscCuBLASInt,PetscBool))
{
  PetscScalar       *y;
  PetscInt          n, _ldb, _ldx;
  PetscBool         biscuda, xiscuda, aiscuda;
  PetscCuBLASInt    nrhs=0,m=0,k=0,ldb=0,ldx=0,ldy=0;

  PetscFunctionBegin;
  PetscCheckFalse(A->factortype == MAT_FACTOR_NONE,PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&k));
  CHKERRQ(MatGetSize(B,NULL,&n));
  CHKERRQ(PetscCuBLASIntCast(n,&nrhs));
  CHKERRQ(MatDenseGetLDA(B,&_ldb));
  CHKERRQ(PetscCuBLASIntCast(_ldb, &ldb));
  CHKERRQ(MatDenseGetLDA(X,&_ldx));
  CHKERRQ(PetscCuBLASIntCast(_ldx, &ldx));

  CHKERRQ(PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&biscuda));
  CHKERRQ(PetscObjectTypeCompare((PetscObject)X,MATSEQDENSECUDA,&xiscuda));
  {
    /* The logic here is to try to minimize the amount of memory copying:
       if we call MatDenseCUDAGetArrayRead(B,&b) every time biscuda and the
       data is not offloaded to the GPU yet, then the data is copied to the
       GPU.  But we are only trying to get the data in order to copy it into the y
       array.  So the array b will be wherever the data already is so that
       only one memcpy is performed */
    const PetscScalar *b;

    /* some copying from B will be involved */
    PetscBool bishost = PETSC_TRUE;

    if (biscuda && B->offloadmask & PETSC_OFFLOAD_GPU) {
      CHKERRQ(MatDenseCUDAGetArrayRead(B,&b));
      bishost = PETSC_FALSE;
    } else {
      CHKERRQ(MatDenseGetArrayRead(B,&b));
    }
    if (ldx < m || !xiscuda) {
      /* X's array cannot serve as the array (too small or not on device), B's
       * array cannot serve as the array (const), so allocate a new array  */
      ldy = m;
      CHKERRCUDA(hipMalloc((void**)&y,nrhs*m*sizeof(PetscScalar)));
    } else {
      /* X's array should serve as the array */
      ldy = ldx;
      CHKERRQ(MatDenseCUDAGetArrayWrite(X,&y));
    }
    CHKERRCUDA(hipMemcpy2D(y,ldy*sizeof(PetscScalar),b,ldb*sizeof(PetscScalar),m*sizeof(PetscScalar),nrhs,bishost ? hipMemcpyHostToDevice: hipMemcpyDeviceToDevice));
    if (bishost) {
      CHKERRQ(MatDenseRestoreArrayRead(B,&b));
    } else {
      CHKERRQ(MatDenseCUDARestoreArrayRead(B,&b));
    }
  }
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&aiscuda));
  if (!aiscuda) {
    CHKERRQ(MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A));
  }
  CHKERRQ((*matsolve) (A, y, ldy, m, nrhs, k, transpose));
  if (!aiscuda) {
    CHKERRQ(MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A));
  }
  if (ldx < m || !xiscuda) {
    PetscScalar *x;

    /* The logic here is that the data is not yet in either X's GPU array or its
       CPU array.  There is nothing in the interface to say where the user would like
       it to end up.  So we choose the GPU, because it is the faster option */
    if (xiscuda) {
      CHKERRQ(MatDenseCUDAGetArrayWrite(X,&x));
    } else {
      CHKERRQ(MatDenseGetArray(X,&x));
    }
    CHKERRCUDA(hipMemcpy2D(x,ldx*sizeof(PetscScalar),y,ldy*sizeof(PetscScalar),k*sizeof(PetscScalar),nrhs,xiscuda ? hipMemcpyDeviceToDevice: hipMemcpyDeviceToHost));
    if (xiscuda) {
      CHKERRQ(MatDenseCUDARestoreArrayWrite(X,&x));
    } else {
      CHKERRQ(MatDenseRestoreArray(X,&x));
    }
    CHKERRCUDA(hipFree(y));
  } else {
    CHKERRQ(MatDenseCUDARestoreArrayWrite(X,&y));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal_LU(Mat A, PetscScalar *x, PetscCuBLASInt ldx, PetscCuBLASInt m, PetscCuBLASInt nrhs, PetscCuBLASInt k, PetscBool T)
{
  Mat_SeqDense       *mat = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscCuBLASInt     lda;
  hipsolverHandle_t handle;
  int                info;

  PetscFunctionBegin;
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&da));
  CHKERRQ(PetscCuBLASIntCast(mat->lda,&lda));
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(PetscLogGpuTimeBegin());
  CHKERRQ(PetscInfo(A,"LU solve %d x %d on backend\n",m,k));
  CHKERRCUSOLVER(hipsolverDnXgetrs(handle,T ? HIPBLAS_OP_T : HIPBLAS_OP_N,m,nrhs,da,lda,dA->d_fact_ipiv,x,ldx,dA->d_fact_info));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&da));
  if (PetscDefined(USE_DEBUG)) {
    CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
    PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  }
  CHKERRQ(PetscLogGpuFlops(nrhs*(2.0*m*m - m)));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal_Cholesky(Mat A, PetscScalar *x, PetscCuBLASInt ldx, PetscCuBLASInt m, PetscCuBLASInt nrhs, PetscCuBLASInt k, PetscBool T)
{
  Mat_SeqDense       *mat = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscCuBLASInt     lda;
  hipsolverHandle_t handle;
  int                info;

  PetscFunctionBegin;
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&da));
  CHKERRQ(PetscCuBLASIntCast(mat->lda,&lda));
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(PetscLogGpuTimeBegin());
  CHKERRQ(PetscInfo(A,"Cholesky solve %d x %d on backend\n",m,k));
  if (!dA->d_fact_ipiv) { /* spd */
    /* ========= Program hit hipErrorNotReady (error 34) due to "device not ready" on CUDA API call to hipEventQuery. */
    CHKERRCUSOLVER(hipsolverDnXpotrs(handle,HIPBLAS_FILL_MODE_LOWER,m,nrhs,da,lda,x,ldx,dA->d_fact_info));
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytrs not implemented");
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&da));
  if (PetscDefined(USE_DEBUG)) {
    CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
    PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  }
  CHKERRQ(PetscLogGpuFlops(nrhs*(2.0*m*m - m)));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Internal_QR(Mat A, PetscScalar *x, PetscCuBLASInt ldx, PetscCuBLASInt m, PetscCuBLASInt nrhs, PetscCuBLASInt k, PetscBool T)
{
  Mat_SeqDense       *mat = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscCuBLASInt     lda, rank;
  hipsolverHandle_t handle;
  hipblasHandle_t     bhandle;
  hipsolverStatus_t   csrr;
  hipblasStatus_t     cbrr;
  int                info;
  hipblasOperation_t  trans;
  PetscScalar        one = 1.;

  PetscFunctionBegin;
  CHKERRQ(PetscCuBLASIntCast(mat->rank,&rank));
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&da));
  CHKERRQ(PetscCuBLASIntCast(mat->lda,&lda));
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(PetscCUBLASGetHandle(&bhandle));
  CHKERRQ(PetscLogGpuTimeBegin());
  CHKERRQ(PetscInfo(A,"QR solve %d x %d on backend\n",m,k));
  if (!T) {
    if (PetscDefined(USE_COMPLEX)) {
      trans = HIPBLAS_OP_C;
    } else {
      trans = HIPBLAS_OP_T;
    }
    csrr = cusolverDnXormqr(handle, HIPBLAS_SIDE_LEFT, trans, m, nrhs, rank, da, lda, dA->d_fact_tau, x, ldx, dA->d_fact_work, dA->fact_lwork, dA->d_fact_info);CHKERRCUSOLVER(csrr);
    if (PetscDefined(USE_DEBUG)) {
      CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
      PetscCheck(info == 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
    }
    cbrr = cublasXtrsm(bhandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, rank, nrhs, &one, da, lda, x, ldx);CHKERRCUBLAS(cbrr);
  } else {
    cbrr = cublasXtrsm(bhandle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, rank, nrhs, &one, da, lda, x, ldx);CHKERRCUBLAS(cbrr);
    csrr = cusolverDnXormqr(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, m, nrhs, rank, da, lda, dA->d_fact_tau, x, ldx, dA->d_fact_work, dA->fact_lwork, dA->d_fact_info);CHKERRCUSOLVER(csrr);
    if (PetscDefined(USE_DEBUG)) {
      CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
      PetscCheck(info == 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
    }
  }
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&da));
  CHKERRQ(PetscLogFlops(nrhs*(4.0*m*mat->rank - PetscSqr(mat->rank))));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_LU(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_LU));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA_LU(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_LU));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Cholesky(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_Cholesky));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA_Cholesky(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_Cholesky));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_QR(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_QR));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA_QR(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatSolve_SeqDenseCUDA_Internal(A, xx, yy, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_QR));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_LU(Mat A,Mat B,Mat X)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_LU));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolveTranspose_SeqDenseCUDA_LU(Mat A,Mat B,Mat X)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_LU));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_Cholesky(Mat A,Mat B,Mat X)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_Cholesky));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolveTranspose_SeqDenseCUDA_Cholesky(Mat A,Mat B,Mat X)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_Cholesky));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA_QR(Mat A,Mat B,Mat X)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_FALSE, MatSolve_SeqDenseCUDA_Internal_QR));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolveTranspose_SeqDenseCUDA_QR(Mat A,Mat B,Mat X)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatSolve_SeqDenseCUDA_Internal(A, B, X, PETSC_TRUE, MatSolve_SeqDenseCUDA_Internal_QR));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactor_SeqDenseCUDA(Mat A,IS rperm,IS cperm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscCuBLASInt     m,n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverHandle_t handle;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(MatDenseCUDAGetArray(A,&da));
  CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&n));
  CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(a->lda,&lda));
  CHKERRQ(PetscInfo(A,"LU factor %d x %d on backend\n",m,n));
  if (!dA->d_fact_ipiv) {
    CHKERRCUDA(hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv)));
  }
  if (!dA->fact_lwork) {
    CHKERRCUSOLVER(hipsolverDnXgetrf_bufferSize(handle,m,n,da,lda,&dA->fact_lwork));
    CHKERRCUDA(hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work)));
  }
  if (!dA->d_fact_info) {
    CHKERRCUDA(hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info)));
  }
  CHKERRQ(PetscLogGpuTimeBegin());
  CHKERRCUSOLVER(hipsolverDnXgetrf(handle,m,n,da,lda,dA->d_fact_work,dA->d_fact_ipiv,dA->d_fact_info));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArray(A,&da));
#if defined(PETSC_USE_DEBUG)
  CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
  PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_LU_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  A->factortype = MAT_FACTOR_LU;
  CHKERRQ(PetscLogGpuFlops(2.0*n*n*m/3.0));

  A->ops->solve             = MatSolve_SeqDenseCUDA_LU;
  A->ops->solvetranspose    = MatSolveTranspose_SeqDenseCUDA_LU;
  A->ops->matsolve          = MatMatSolve_SeqDenseCUDA_LU;
  A->ops->matsolvetranspose = MatMatSolveTranspose_SeqDenseCUDA_LU;

  CHKERRQ(PetscFree(A->solvertype));
  CHKERRQ(PetscStrallocpy(MATSOLVERCUDA,&A->solvertype));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactor_SeqDenseCUDA(Mat A,IS perm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscCuBLASInt     n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverHandle_t handle;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&n));
  CHKERRQ(PetscInfo(A,"Cholesky factor %d x %d on backend\n",n,n));
  if (A->spd) {
    CHKERRQ(MatDenseCUDAGetArray(A,&da));
    CHKERRQ(PetscCuBLASIntCast(a->lda,&lda));
    if (!dA->fact_lwork) {
      CHKERRCUSOLVER(hipsolverDnXpotrf_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&dA->fact_lwork));
      CHKERRCUDA(hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work)));
    }
    if (!dA->d_fact_info) {
      CHKERRCUDA(hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info)));
    }
    CHKERRQ(PetscLogGpuTimeBegin());
    CHKERRCUSOLVER(hipsolverDnXpotrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info));
    CHKERRQ(PetscLogGpuTimeEnd());

    CHKERRQ(MatDenseCUDARestoreArray(A,&da));
#if defined(PETSC_USE_DEBUG)
    CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
    PetscCheckFalse(info > 0,PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
    A->factortype = MAT_FACTOR_CHOLESKY;
    CHKERRQ(PetscLogGpuFlops(1.0*n*n*n/3.0));
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"cusolverDnsytrs unavailable. Use MAT_FACTOR_LU");
#if 0
    /* at the time of writing this interface (cuda 10.0), cusolverDn does not implement *sytrs and *hetr* routines
       The code below should work, and it can be activated when *sytrs routines will be available */
    if (!dA->d_fact_ipiv) {
      CHKERRCUDA(hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv)));
    }
    if (!dA->fact_lwork) {
      CHKERRCUSOLVER(cusolverDnXsytrf_bufferSize(handle,n,da,lda,&dA->fact_lwork));
      CHKERRCUDA(hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work)));
    }
    if (!dA->d_fact_info) {
      CHKERRCUDA(hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info)));
    }
    CHKERRQ(PetscLogGpuTimeBegin());
    CHKERRCUSOLVER(cusolverDnXsytrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_ipiv,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info));
    CHKERRQ(PetscLogGpuTimeEnd());
#endif

  A->ops->solve             = MatSolve_SeqDenseCUDA_Cholesky;
  A->ops->solvetranspose    = MatSolveTranspose_SeqDenseCUDA_Cholesky;
  A->ops->matsolve          = MatMatSolve_SeqDenseCUDA_Cholesky;
  A->ops->matsolvetranspose = MatMatSolveTranspose_SeqDenseCUDA_Cholesky;
  CHKERRQ(PetscFree(A->solvertype));
  CHKERRQ(PetscStrallocpy(MATSOLVERCUDA,&A->solvertype));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatQRFactor_SeqDenseCUDA(Mat A,IS col,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscCuBLASInt     m,min,max,n,lda;
#if defined(PETSC_USE_DEBUG)
  int                info;
#endif
  hipsolverHandle_t handle;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  CHKERRQ(PetscCUSOLVERDnGetHandle(&handle));
  CHKERRQ(MatDenseCUDAGetArray(A,&da));
  CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&n));
  CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(a->lda,&lda));
  CHKERRQ(PetscInfo(A,"QR factor %d x %d on backend\n",m,n));
  max = PetscMax(m,n);
  min = PetscMin(m,n);
  if (!dA->d_fact_tau) CHKERRCUDA(hipMalloc((void**)&dA->d_fact_tau,min*sizeof(*dA->d_fact_tau)));
  if (!dA->d_fact_ipiv) CHKERRCUDA(hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv)));
  if (!dA->fact_lwork) {
    CHKERRCUSOLVER(hipsolverDnXgeqrf_bufferSize(handle,m,n,da,lda,&dA->fact_lwork));
    CHKERRCUDA(hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work)));
  }
  if (!dA->d_fact_info) CHKERRCUDA(hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info)));
  if (!dA->workvec) CHKERRQ(VecCreateSeqCUDA(PetscObjectComm((PetscObject)A), m, &(dA->workvec)));
  CHKERRQ(PetscLogGpuTimeBegin());
  CHKERRCUSOLVER(hipsolverDnXgeqrf(handle,m,n,da,lda,dA->d_fact_tau,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(MatDenseCUDARestoreArray(A,&da));
#if defined(PETSC_USE_DEBUG)
  CHKERRCUDA(hipMemcpy(&info, dA->d_fact_info, sizeof(PetscCuBLASInt), hipMemcpyDeviceToHost));
  PetscCheckFalse(info < 0,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
#endif
  A->factortype = MAT_FACTOR_QR;
  a->rank = min;
  CHKERRQ(PetscLogGpuFlops(2.0*min*min*(max-min/3.0)));

  A->ops->solve             = MatSolve_SeqDenseCUDA_QR;
  A->ops->solvetranspose    = MatSolveTranspose_SeqDenseCUDA_QR;
  A->ops->matsolve          = MatMatSolve_SeqDenseCUDA_QR;
  A->ops->matsolvetranspose = MatMatSolveTranspose_SeqDenseCUDA_QR;

  CHKERRQ(PetscFree(A->solvertype));
  CHKERRQ(PetscStrallocpy(MATSOLVERCUDA,&A->solvertype));
  PetscFunctionReturn(0);
}

/* GEMM kernel: C = op(A)*op(B), tA, tB flag transposition */
PETSC_INTERN PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(Mat A,Mat B,Mat C,PetscBool tA,PetscBool tB)
{
  const PetscScalar *da,*db;
  PetscScalar       *dc;
  PetscScalar       one=1.0,zero=0.0;
  PetscCuBLASInt    m,n,k;
  PetscInt          alda,blda,clda;
  hipblasHandle_t    cublasv2handle;
  PetscBool         Aiscuda,Biscuda;
  hipblasStatus_t    berr;

  PetscFunctionBegin;
  /* we may end up with SEQDENSE as one of the arguments */
  CHKERRQ(PetscObjectTypeCompare((PetscObject)A,MATSEQDENSECUDA,&Aiscuda));
  CHKERRQ(PetscObjectTypeCompare((PetscObject)B,MATSEQDENSECUDA,&Biscuda));
  if (!Aiscuda) CHKERRQ(MatConvert(A,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&A));
  if (!Biscuda) CHKERRQ(MatConvert(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B));
  CHKERRQ(PetscCuBLASIntCast(C->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(C->cmap->n,&n));
  if (tA) CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&k));
  else    CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&k));
  if (!m || !n || !k) PetscFunctionReturn(0);
  CHKERRQ(PetscInfo(C,"Matrix-Matrix product %d x %d x %d on backend\n",m,k,n));
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&da));
  CHKERRQ(MatDenseCUDAGetArrayRead(B,&db));
  CHKERRQ(MatDenseCUDAGetArrayWrite(C,&dc));
  CHKERRQ(MatDenseGetLDA(A,&alda));
  CHKERRQ(MatDenseGetLDA(B,&blda));
  CHKERRQ(MatDenseGetLDA(C,&clda));
  CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
  CHKERRQ(PetscLogGpuTimeBegin());
  berr = cublasXgemm(cublasv2handle,tA ? HIPBLAS_OP_T : HIPBLAS_OP_N,tB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,k,&one,da,alda,db,blda,&zero,dc,clda);CHKERRCUBLAS(berr);
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(1.0*m*n*k + 1.0*m*n*(k-1)));
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&da));
  CHKERRQ(MatDenseCUDARestoreArrayRead(B,&db));
  CHKERRQ(MatDenseCUDARestoreArrayWrite(C,&dc));
  if (!Aiscuda) CHKERRQ(MatConvert(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A));
  if (!Biscuda) CHKERRQ(MatConvert(B,MATSEQDENSE,MAT_INPLACE_MATRIX,&B));
  PetscFunctionReturn(0);
}

PetscErrorCode MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_TRUE,PETSC_FALSE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_FALSE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscFunctionBegin;
  CHKERRQ(MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_TRUE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatProductSetFromOptions_SeqDenseCUDA(Mat C)
{
  PetscFunctionBegin;
  CHKERRQ(MatProductSetFromOptions_SeqDense(C));
  PetscFunctionReturn(0);
}

/* zz = op(A)*xx + yy
   if yy == NULL, only MatMult */
static PetscErrorCode MatMultAdd_SeqDenseCUDA_Private(Mat A,Vec xx,Vec yy,Vec zz,PetscBool trans)
{
  Mat_SeqDense      *mat = (Mat_SeqDense*)A->data;
  const PetscScalar *xarray,*da;
  PetscScalar       *zarray;
  PetscScalar       one=1.0,zero=0.0;
  PetscCuBLASInt    m, n, lda;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;

  PetscFunctionBegin;
   /* mult add */
  if (yy && yy != zz) CHKERRQ(VecCopy_SeqCUDA(yy,zz));
  if (!A->rmap->n || !A->cmap->n) {
    /* mult only */
    if (!yy) CHKERRQ(VecSet_SeqCUDA(zz,0.0));
    PetscFunctionReturn(0);
  }
  CHKERRQ(PetscInfo(A,"Matrix-vector product %d x %d on backend\n",A->rmap->n,A->cmap->n));
  CHKERRQ(PetscCuBLASIntCast(A->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(A->cmap->n,&n));
  CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&da));
  CHKERRQ(PetscCuBLASIntCast(mat->lda,&lda));
  CHKERRQ(VecCUDAGetArrayRead(xx,&xarray));
  CHKERRQ(VecCUDAGetArray(zz,&zarray));
  CHKERRQ(PetscLogGpuTimeBegin());
  berr = cublasXgemv(cublasv2handle,trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,&one,da,lda,xarray,1,(yy ? &one : &zero),zarray,1);CHKERRCUBLAS(berr);
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(2.0*A->rmap->n*A->cmap->n - (yy ? 0 : A->rmap->n)));
  CHKERRQ(VecCUDARestoreArrayRead(xx,&xarray));
  CHKERRQ(VecCUDARestoreArray(zz,&zarray));
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&da));
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_FALSE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTransposeAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_TRUE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatMult_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_FALSE));
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTranspose_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscFunctionBegin;
  CHKERRQ(MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_TRUE));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArrayRead_SeqDenseCUDA(Mat A,const PetscScalar **array)
{
  Mat_SeqDense *mat = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  CHKERRQ(MatSeqDenseCUDACopyFromGPU(A));
  *array = mat->v;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArrayWrite_SeqDenseCUDA(Mat A,PetscScalar **array)
{
  Mat_SeqDense *mat = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  /* MatCreateSeqDenseCUDA may not allocate CPU memory. Allocate if needed */
  if (!mat->v) CHKERRQ(MatSeqDenseSetPreallocation(A,NULL));
  *array = mat->v;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetArray_SeqDenseCUDA(Mat A,PetscScalar **array)
{
  Mat_SeqDense *mat = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  CHKERRQ(MatSeqDenseCUDACopyFromGPU(A));
  *array = mat->v;
  A->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatScale_SeqDenseCUDA(Mat Y,PetscScalar alpha)
{
  Mat_SeqDense   *y = (Mat_SeqDense*)Y->data;
  PetscScalar    *dy;
  PetscCuBLASInt j,N,m,lday,one = 1;
  hipblasHandle_t cublasv2handle;

  PetscFunctionBegin;
  CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
  CHKERRQ(MatDenseCUDAGetArray(Y,&dy));
  CHKERRQ(PetscCuBLASIntCast(Y->rmap->n*Y->cmap->n,&N));
  CHKERRQ(PetscCuBLASIntCast(Y->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(y->lda,&lday));
  CHKERRQ(PetscInfo(Y,"Performing Scale %d x %d on backend\n",Y->rmap->n,Y->cmap->n));
  CHKERRQ(PetscLogGpuTimeBegin());
  if (lday>m) {
    for (j=0; j<Y->cmap->n; j++) CHKERRCUBLAS(cublasXscal(cublasv2handle,m,&alpha,dy+lday*j,one));
  } else CHKERRCUBLAS(cublasXscal(cublasv2handle,N,&alpha,dy,one));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(N));
  CHKERRQ(MatDenseCUDARestoreArray(Y,&dy));
  PetscFunctionReturn(0);
}

PetscErrorCode MatAXPY_SeqDenseCUDA(Mat Y,PetscScalar alpha,Mat X,MatStructure str)
{
  Mat_SeqDense      *x = (Mat_SeqDense*)X->data;
  Mat_SeqDense      *y = (Mat_SeqDense*)Y->data;
  const PetscScalar *dx;
  PetscScalar       *dy;
  PetscCuBLASInt    j,N,m,ldax,lday,one = 1;
  hipblasHandle_t    cublasv2handle;

  PetscFunctionBegin;
  if (!X->rmap->n || !X->cmap->n) PetscFunctionReturn(0);
  CHKERRQ(PetscCUBLASGetHandle(&cublasv2handle));
  CHKERRQ(MatDenseCUDAGetArrayRead(X,&dx));
  if (alpha == 0.0) CHKERRQ(MatDenseCUDAGetArrayWrite(Y,&dy));
  else              CHKERRQ(MatDenseCUDAGetArray(Y,&dy));
  CHKERRQ(PetscCuBLASIntCast(X->rmap->n*X->cmap->n,&N));
  CHKERRQ(PetscCuBLASIntCast(X->rmap->n,&m));
  CHKERRQ(PetscCuBLASIntCast(x->lda,&ldax));
  CHKERRQ(PetscCuBLASIntCast(y->lda,&lday));
  CHKERRQ(PetscInfo(Y,"Performing AXPY %d x %d on backend\n",Y->rmap->n,Y->cmap->n));
  CHKERRQ(PetscLogGpuTimeBegin());
  if (ldax>m || lday>m) {
    for (j=0; j<X->cmap->n; j++) {
      CHKERRCUBLAS(cublasXaxpy(cublasv2handle,m,&alpha,dx+j*ldax,one,dy+j*lday,one));
    }
  } else CHKERRCUBLAS(cublasXaxpy(cublasv2handle,N,&alpha,dx,one,dy,one));
  CHKERRQ(PetscLogGpuTimeEnd());
  CHKERRQ(PetscLogGpuFlops(PetscMax(2.*N-1,0)));
  CHKERRQ(MatDenseCUDARestoreArrayRead(X,&dx));
  if (alpha == 0.0) CHKERRQ(MatDenseCUDARestoreArrayWrite(Y,&dy));
  else              CHKERRQ(MatDenseCUDARestoreArray(Y,&dy));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatReset_SeqDenseCUDA(Mat A)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  if (dA) {
    PetscCheck(!dA->unplacedarray,PETSC_COMM_SELF,PETSC_ERR_ORDER,"MatDenseCUDAResetArray() must be called first");
    if (!dA->user_alloc) CHKERRCUDA(hipFree(dA->d_v));
    CHKERRCUDA(hipFree(dA->d_fact_tau));
    CHKERRCUDA(hipFree(dA->d_fact_ipiv));
    CHKERRCUDA(hipFree(dA->d_fact_info));
    CHKERRCUDA(hipFree(dA->d_fact_work));
    CHKERRQ(VecDestroy(&dA->workvec));
  }
  CHKERRQ(PetscFree(A->spptr));
  PetscFunctionReturn(0);
}

PetscErrorCode MatDestroy_SeqDenseCUDA(Mat A)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  /* prevent to copy back data if we own the data pointer */
  if (!a->user_alloc) A->offloadmask = PETSC_OFFLOAD_CPU;
  CHKERRQ(MatConvert_SeqDenseCUDA_SeqDense(A,MATSEQDENSE,MAT_INPLACE_MATRIX,&A));
  CHKERRQ(MatDestroy_SeqDense(A));
  PetscFunctionReturn(0);
}

PetscErrorCode MatDuplicate_SeqDenseCUDA(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  MatDuplicateOption hcpvalues = (cpvalues == MAT_COPY_VALUES && A->offloadmask != PETSC_OFFLOAD_CPU) ? MAT_DO_NOT_COPY_VALUES : cpvalues;

  PetscFunctionBegin;
  CHKERRQ(MatCreate(PetscObjectComm((PetscObject)A),B));
  CHKERRQ(MatSetSizes(*B,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n));
  CHKERRQ(MatSetType(*B,((PetscObject)A)->type_name));
  CHKERRQ(MatDuplicateNoCreate_SeqDense(*B,A,hcpvalues));
  if (cpvalues == MAT_COPY_VALUES && hcpvalues != MAT_COPY_VALUES) {
    CHKERRQ(MatCopy_SeqDenseCUDA(A,*B,SAME_NONZERO_PATTERN));
  }
  if (cpvalues != MAT_COPY_VALUES) { /* allocate memory if needed */
    Mat_SeqDenseCUDA *dB = (Mat_SeqDenseCUDA*)(*B)->spptr;
    if (!dB->d_v) {
      CHKERRQ(MatSeqDenseCUDASetPreallocation(*B,NULL));
    }
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatGetColumnVector_SeqDenseCUDA(Mat A,Vec v,PetscInt col)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar      *x;
  PetscBool        viscuda;

  PetscFunctionBegin;
  CHKERRQ(PetscObjectTypeCompareAny((PetscObject)v,&viscuda,VECSEQCUDA,VECMPICUDA,VECCUDA,""));
  if (viscuda && !v->boundtocpu) { /* update device data */
    CHKERRQ(VecCUDAGetArrayWrite(v,&x));
    if (A->offloadmask & PETSC_OFFLOAD_GPU) {
      CHKERRCUDA(hipMemcpy(x,dA->d_v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyHostToHost));
    } else {
      CHKERRCUDA(hipMemcpy(x,a->v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyHostToDevice));
    }
    CHKERRQ(VecCUDARestoreArrayWrite(v,&x));
  } else { /* update host data */
    CHKERRQ(VecGetArrayWrite(v,&x));
    if (A->offloadmask == PETSC_OFFLOAD_UNALLOCATED || A->offloadmask & PETSC_OFFLOAD_CPU) {
      CHKERRQ(PetscArraycpy(x,a->v+col*a->lda,A->rmap->n));
    } else if (A->offloadmask & PETSC_OFFLOAD_GPU) {
      CHKERRCUDA(hipMemcpy(x,dA->d_v + col*a->lda,A->rmap->n*sizeof(PetscScalar),hipMemcpyDeviceToHost));
    }
    CHKERRQ(VecRestoreArrayWrite(v,&x));
  }
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatGetFactor_seqdense_cuda(Mat A,MatFactorType ftype,Mat *fact)
{
  PetscFunctionBegin;
  CHKERRQ(MatCreate(PetscObjectComm((PetscObject)A),fact));
  CHKERRQ(MatSetSizes(*fact,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n));
  CHKERRQ(MatSetType(*fact,MATSEQDENSECUDA));
  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU) {
    (*fact)->ops->lufactorsymbolic = MatLUFactorSymbolic_SeqDense;
    (*fact)->ops->ilufactorsymbolic = MatLUFactorSymbolic_SeqDense;
  } else if (ftype == MAT_FACTOR_CHOLESKY || ftype == MAT_FACTOR_ICC) {
    (*fact)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqDense;
  } else if (ftype == MAT_FACTOR_QR) {
    CHKERRQ(PetscObjectComposeFunction((PetscObject)(*fact),"MatQRFactor_C",MatQRFactor_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)(*fact),"MatQRFactorSymbolic_C",MatQRFactorSymbolic_SeqDense));
  }
  (*fact)->factortype = ftype;
  CHKERRQ(PetscFree((*fact)->solvertype));
  CHKERRQ(PetscStrallocpy(MATSOLVERCUDA,&(*fact)->solvertype));
  CHKERRQ(PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_LU]));
  CHKERRQ(PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_ILU]));
  CHKERRQ(PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_CHOLESKY]));
  CHKERRQ(PetscStrallocpy(MATORDERINGEXTERNAL,(char**)&(*fact)->preferredordering[MAT_FACTOR_ICC]));
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVec_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  CHKERRQ(MatDenseCUDAGetArray(A,(PetscScalar**)&a->ptrinuse));
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    CHKERRQ(VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec));
    CHKERRQ(PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec));
  }
  a->vecinuse = col + 1;
  CHKERRQ(VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda));
  *v   = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVec_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  PetscCheck(a->cvec,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  CHKERRQ(VecCUDAResetArray(a->cvec));
  CHKERRQ(MatDenseCUDARestoreArray(A,(PetscScalar**)&a->ptrinuse));
  if (v) *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVecRead_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  CHKERRQ(MatDenseCUDAGetArrayRead(A,&a->ptrinuse));
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    CHKERRQ(VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec));
    CHKERRQ(PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec));
  }
  a->vecinuse = col + 1;
  CHKERRQ(VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda));
  CHKERRQ(VecLockReadPush(a->cvec));
  *v = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVecRead_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  PetscCheck(a->cvec,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  CHKERRQ(VecLockReadPop(a->cvec));
  CHKERRQ(VecCUDAResetArray(a->cvec));
  CHKERRQ(MatDenseCUDARestoreArrayRead(A,&a->ptrinuse));
  if (v) *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumnVecWrite_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  CHKERRQ(MatDenseCUDAGetArrayWrite(A,(PetscScalar**)&a->ptrinuse));
  if (!a->cvec) { /* we pass the data of A, to prevent allocating needless GPU memory the first time VecCUDAPlaceArray is called */
    CHKERRQ(VecCreateSeqCUDAWithArray(PetscObjectComm((PetscObject)A),A->rmap->bs,A->rmap->n,a->ptrinuse,&a->cvec));
    CHKERRQ(PetscLogObjectParent((PetscObject)A,(PetscObject)a->cvec));
  }
  a->vecinuse = col + 1;
  CHKERRQ(VecCUDAPlaceArray(a->cvec,a->ptrinuse + (size_t)col * (size_t)a->lda));
  *v = a->cvec;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumnVecWrite_SeqDenseCUDA(Mat A,PetscInt col,Vec *v)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetColumnVec() first");
  PetscCheck(a->cvec,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column vector");
  a->vecinuse = 0;
  CHKERRQ(VecCUDAResetArray(a->cvec));
  CHKERRQ(MatDenseCUDARestoreArrayWrite(A,(PetscScalar**)&a->ptrinuse));
  if (v) *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetSubMatrix_SeqDenseCUDA(Mat A,PetscInt cbegin,PetscInt cend,Mat *v)
{
  Mat_SeqDense     *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  PetscCheck(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  if (a->cmat && cend-cbegin != a->cmat->cmap->N) {
    CHKERRQ(MatDestroy(&a->cmat));
  }
  CHKERRQ(MatSeqDenseCUDACopyToGPU(A));
  if (!a->cmat) {
    CHKERRQ(MatCreateDenseCUDA(PetscObjectComm((PetscObject)A),A->rmap->n,PETSC_DECIDE,A->rmap->N,cend-cbegin,dA->d_v+(size_t)cbegin*a->lda,&a->cmat));
    CHKERRQ(PetscLogObjectParent((PetscObject)A,(PetscObject)a->cmat));
  } else {
    CHKERRQ(MatDenseCUDAPlaceArray(a->cmat,dA->d_v+(size_t)cbegin*a->lda));
  }
  CHKERRQ(MatDenseSetLDA(a->cmat,a->lda));
  if (a->v) CHKERRQ(MatDensePlaceArray(a->cmat,a->v+(size_t)cbegin*a->lda));
  a->cmat->offloadmask = A->offloadmask;
  a->matinuse = cbegin + 1;
  *v = a->cmat;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreSubMatrix_SeqDenseCUDA(Mat A,Mat *v)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseGetSubMatrix() first");
  PetscCheck(a->cmat,PETSC_COMM_SELF,PETSC_ERR_PLIB,"Missing internal column matrix");
  PetscCheck(*v == a->cmat,PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Not the matrix obtained from MatDenseGetSubMatrix()");
  a->matinuse = 0;
  A->offloadmask = (a->cmat->offloadmask == PETSC_OFFLOAD_CPU) ? PETSC_OFFLOAD_CPU : PETSC_OFFLOAD_GPU;
  CHKERRQ(MatDenseCUDAResetArray(a->cmat));
  if (a->unplacedarray) CHKERRQ(MatDenseResetArray(a->cmat));
  a->cmat->offloadmask = PETSC_OFFLOAD_UNALLOCATED;
  *v = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode  MatDenseSetLDA_SeqDenseCUDA(Mat A,PetscInt lda)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscBool        data;

  PetscFunctionBegin;
  data = (PetscBool)((A->rmap->n > 0 && A->cmap->n > 0) ? (dA->d_v ? PETSC_TRUE : PETSC_FALSE) : PETSC_FALSE);
  PetscCheckFalse(!dA->user_alloc && data && cA->lda!=lda,PETSC_COMM_SELF,PETSC_ERR_ORDER,"LDA cannot be changed after allocation of internal storage");
  PetscCheck(lda >= A->rmap->n,PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"LDA %" PetscInt_FMT " must be at least matrix dimension %" PetscInt_FMT,lda,A->rmap->n);
  cA->lda = lda;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSetUp_SeqDenseCUDA(Mat A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  CHKERRQ(PetscLayoutSetUp(A->rmap));
  CHKERRQ(PetscLayoutSetUp(A->cmap));
  if (!A->preallocated) {
    CHKERRQ(MatSeqDenseCUDASetPreallocation(A,NULL));
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatBindToCPU_SeqDenseCUDA(Mat A,PetscBool flg)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  PetscCheck(!a->vecinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreColumnVec() first");
  PetscCheck(!a->matinuse,PETSC_COMM_SELF,PETSC_ERR_ORDER,"Need to call MatDenseRestoreSubMatrix() first");
  A->boundtocpu = flg;
  if (!flg) {
    PetscBool iscuda;

    CHKERRQ(PetscObjectTypeCompare((PetscObject)a->cvec,VECSEQCUDA,&iscuda));
    if (!iscuda) {
      CHKERRQ(VecDestroy(&a->cvec));
    }
    CHKERRQ(PetscObjectTypeCompare((PetscObject)a->cmat,MATSEQDENSECUDA,&iscuda));
    if (!iscuda) {
      CHKERRQ(MatDestroy(&a->cmat));
    }
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",MatDenseGetArray_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",MatDenseGetArrayRead_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayWrite_C",MatDenseGetArrayWrite_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVec_C",MatDenseGetColumnVec_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVec_C",MatDenseRestoreColumnVec_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecRead_C",MatDenseGetColumnVecRead_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecRead_C",MatDenseRestoreColumnVecRead_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecWrite_C",MatDenseGetColumnVecWrite_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecWrite_C",MatDenseRestoreColumnVecWrite_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetSubMatrix_C",MatDenseGetSubMatrix_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreSubMatrix_C",MatDenseRestoreSubMatrix_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseSetLDA_C",MatDenseSetLDA_SeqDenseCUDA));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatQRFactor_C",MatQRFactor_SeqDenseCUDA));

    A->ops->duplicate               = MatDuplicate_SeqDenseCUDA;
    A->ops->mult                    = MatMult_SeqDenseCUDA;
    A->ops->multadd                 = MatMultAdd_SeqDenseCUDA;
    A->ops->multtranspose           = MatMultTranspose_SeqDenseCUDA;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDenseCUDA;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->axpy                    = MatAXPY_SeqDenseCUDA;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDenseCUDA;
    A->ops->lufactor                = MatLUFactor_SeqDenseCUDA;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDenseCUDA;
    A->ops->getcolumnvector         = MatGetColumnVector_SeqDenseCUDA;
    A->ops->scale                   = MatScale_SeqDenseCUDA;
    A->ops->copy                    = MatCopy_SeqDenseCUDA;
    A->ops->zeroentries             = MatZeroEntries_SeqDenseCUDA;
    A->ops->setup                   = MatSetUp_SeqDenseCUDA;
  } else {
    /* make sure we have an up-to-date copy on the CPU */
    CHKERRQ(MatSeqDenseCUDACopyFromGPU(A));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",MatDenseGetArray_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",MatDenseGetArray_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayWrite_C",MatDenseGetArray_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVec_C",MatDenseGetColumnVec_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVec_C",MatDenseRestoreColumnVec_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecRead_C",MatDenseGetColumnVecRead_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecRead_C",MatDenseRestoreColumnVecRead_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetColumnVecWrite_C",MatDenseGetColumnVecWrite_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreColumnVecWrite_C",MatDenseRestoreColumnVecWrite_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseGetSubMatrix_C",MatDenseGetSubMatrix_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreSubMatrix_C",MatDenseRestoreSubMatrix_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatDenseSetLDA_C",MatDenseSetLDA_SeqDense));
    CHKERRQ(PetscObjectComposeFunction((PetscObject)A,"MatQRFactor_C",MatQRFactor_SeqDense));

    A->ops->duplicate               = MatDuplicate_SeqDense;
    A->ops->mult                    = MatMult_SeqDense;
    A->ops->multadd                 = MatMultAdd_SeqDense;
    A->ops->multtranspose           = MatMultTranspose_SeqDense;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDense;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDense;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDense_SeqDense;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDense_SeqDense;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDense_SeqDense;
    A->ops->axpy                    = MatAXPY_SeqDense;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDense;
    A->ops->lufactor                = MatLUFactor_SeqDense;
    A->ops->productsetfromoptions   = MatProductSetFromOptions_SeqDense;
    A->ops->getcolumnvector         = MatGetColumnVector_SeqDense;
    A->ops->scale                   = MatScale_SeqDense;
    A->ops->copy                    = MatCopy_SeqDense;
    A->ops->zeroentries             = MatZeroEntries_SeqDense;
    A->ops->setup                   = MatSetUp_SeqDense;
  }
  if (a->cmat) {
    CHKERRQ(MatBindToCPU(a->cmat,flg));
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDenseCUDA_SeqDense(Mat M,MatType type,MatReuse reuse,Mat *newmat)
{
  Mat              B;
  Mat_SeqDense     *a;

  PetscFunctionBegin;
  if (reuse == MAT_REUSE_MATRIX || reuse == MAT_INITIAL_MATRIX) {
    /* TODO these cases should be optimized */
    CHKERRQ(MatConvert_Basic(M,type,reuse,newmat));
    PetscFunctionReturn(0);
  }

  B    = *newmat;
  CHKERRQ(MatBindToCPU_SeqDenseCUDA(B,PETSC_TRUE));
  CHKERRQ(MatReset_SeqDenseCUDA(B));
  CHKERRQ(PetscFree(B->defaultvectype));
  CHKERRQ(PetscStrallocpy(VECSTANDARD,&B->defaultvectype));
  CHKERRQ(PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSE));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqdensecuda_seqdense_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArray_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayRead_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayWrite_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArray_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayRead_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayWrite_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAPlaceArray_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAResetArray_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAReplaceArray_C",NULL));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatProductSetFromOptions_seqaij_seqdensecuda_C",NULL));
  a    = (Mat_SeqDense*)B->data;
  CHKERRQ(VecDestroy(&a->cvec)); /* cvec might be VECSEQCUDA. Destroy it and rebuild a VECSEQ when needed */
  B->ops->bindtocpu = NULL;
  B->ops->destroy = MatDestroy_SeqDense;
  B->offloadmask = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDense_SeqDenseCUDA(Mat M,MatType type,MatReuse reuse,Mat *newmat)
{
  Mat_SeqDenseCUDA *dB;
  Mat              B;
  Mat_SeqDense     *a;

  PetscFunctionBegin;
  CHKERRQ(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
  if (reuse == MAT_REUSE_MATRIX || reuse == MAT_INITIAL_MATRIX) {
    /* TODO these cases should be optimized */
    CHKERRQ(MatConvert_Basic(M,type,reuse,newmat));
    PetscFunctionReturn(0);
  }

  B    = *newmat;
  CHKERRQ(PetscFree(B->defaultvectype));
  CHKERRQ(PetscStrallocpy(VECCUDA,&B->defaultvectype));
  CHKERRQ(PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSECUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatConvert_seqdensecuda_seqdense_C",            MatConvert_SeqDenseCUDA_SeqDense));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArray_C",                        MatDenseCUDAGetArray_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayRead_C",                    MatDenseCUDAGetArrayRead_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAGetArrayWrite_C",                   MatDenseCUDAGetArrayWrite_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArray_C",                    MatDenseCUDARestoreArray_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayRead_C",                MatDenseCUDARestoreArrayRead_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDARestoreArrayWrite_C",               MatDenseCUDARestoreArrayWrite_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAPlaceArray_C",                      MatDenseCUDAPlaceArray_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAResetArray_C",                      MatDenseCUDAResetArray_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatDenseCUDAReplaceArray_C",                    MatDenseCUDAReplaceArray_SeqDenseCUDA));
  CHKERRQ(PetscObjectComposeFunction((PetscObject)B,"MatProductSetFromOptions_seqaij_seqdensecuda_C",MatProductSetFromOptions_SeqAIJ_SeqDense));
  a    = (Mat_SeqDense*)B->data;
  CHKERRQ(VecDestroy(&a->cvec)); /* cvec might be VECSEQ. Destroy it and rebuild a VECSEQCUDA when needed */
  CHKERRQ(PetscNewLog(B,&dB));

  B->spptr = dB;
  B->offloadmask = PETSC_OFFLOAD_UNALLOCATED;

  CHKERRQ(MatBindToCPU_SeqDenseCUDA(B,PETSC_FALSE));
  B->ops->bindtocpu = MatBindToCPU_SeqDenseCUDA;
  B->ops->destroy  = MatDestroy_SeqDenseCUDA;
  PetscFunctionReturn(0);
}

/*@C
   MatCreateSeqDenseCUDA - Creates a sequential matrix in dense format using CUDA.

   Collective

   Input Parameters:
+  comm - MPI communicator
.  m - number of rows
.  n - number of columns
-  data - optional location of GPU matrix data.  Set data=NULL for PETSc
   to control matrix memory allocation.

   Output Parameter:
.  A - the matrix

   Notes:

   Level: intermediate

.seealso: MatCreate(), MatCreateSeqDense()
@*/
PetscErrorCode  MatCreateSeqDenseCUDA(MPI_Comm comm,PetscInt m,PetscInt n,PetscScalar *data,Mat *A)
{
  PetscMPIInt size;

  PetscFunctionBegin;
  CHKERRMPI(MPI_Comm_size(comm,&size));
  PetscCheck(size <= 1,comm,PETSC_ERR_ARG_WRONG,"Invalid communicator size %d",size);
  CHKERRQ(MatCreate(comm,A));
  CHKERRQ(MatSetSizes(*A,m,n,m,n));
  CHKERRQ(MatSetType(*A,MATSEQDENSECUDA));
  CHKERRQ(MatSeqDenseCUDASetPreallocation(*A,data));
  PetscFunctionReturn(0);
}

/*MC
   MATSEQDENSECUDA - MATSEQDENSECUDA = "seqdensecuda" - A matrix type to be used for sequential dense matrices on GPUs.

   Options Database Keys:
. -mat_type seqdensecuda - sets the matrix type to "seqdensecuda" during a call to MatSetFromOptions()

  Level: beginner
M*/
PETSC_EXTERN PetscErrorCode MatCreate_SeqDenseCUDA(Mat B)
{
  PetscFunctionBegin;
  CHKERRQ(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
  CHKERRQ(MatCreate_SeqDense(B));
  CHKERRQ(MatConvert_SeqDense_SeqDenseCUDA(B,MATSEQDENSECUDA,MAT_INPLACE_MATRIX,&B));
  PetscFunctionReturn(0);
}
