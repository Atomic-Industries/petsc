
/*
     Defines the matrix operations for sequential dense with CUDA
*/

#include <../src/mat/impls/dense/seq/dense.h> /*I "petscmat.h" I*/
#include <../src/mat/impls/aij/seq/aij.h>
#include <../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h>

/* TODO: Move to a different include file */
#include <hipsolver.h>
#if defined(PETSC_USE_COMPLEX)
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnCpotrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnCpotrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnCpotrs((a),(b),(c),(d),(hipComplex*)(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnCsytrf((a),(b),(c),(hipComplex*)(d),(e),(f),(hipComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnCsytrf_bufferSize((a),(b),(hipComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnCgetrf((a),(b),(c),(hipComplex*)(d),(e),(hipComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnCgetrf_bufferSize((a),(b),(c),(hipComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnCgetrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(g),(hipDoubleComplex*)(h),(i),(j))
#else /* complex double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnZpotrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnZpotrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnZpotrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnZsytrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(f),(hipDoubleComplex*)(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnZsytrf_bufferSize((a),(b),(hipDoubleComplex*)(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnZgetrf((a),(b),(c),(hipDoubleComplex*)(d),(e),(hipDoubleComplex*)(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnZgetrf_bufferSize((a),(b),(c),(hipDoubleComplex*)(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnZgetrs((a),(b),(c),(d),(hipDoubleComplex*)(e),(f),(g),(hipDoubleComplex*)(h),(i),(j))
#endif
#else /* real single */
#if defined(PETSC_USE_REAL_SINGLE)
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnSpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnSpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnSpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnSsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnSsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnSgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnSgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnSgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#else /* real double */
#define hipsolverDnXpotrf(a,b,c,d,e,f,g,h)        hipsolverDnDpotrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXpotrf_bufferSize(a,b,c,d,e,f) hipsolverDnDpotrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXpotrs(a,b,c,d,e,f,g,h,i)      hipsolverDnDpotrs((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf(a,b,c,d,e,f,g,h,i)      hipsolverDnDsytrf((a),(b),(c),(d),(e),(f),(g),(h),(i))
#define cusolverDnXsytrf_bufferSize(a,b,c,d,e)   hipsolverDnDsytrf_bufferSize((a),(b),(c),(d),(e))
#define hipsolverDnXgetrf(a,b,c,d,e,f,g,h)        hipsolverDnDgetrf((a),(b),(c),(d),(e),(f),(g),(h))
#define hipsolverDnXgetrf_bufferSize(a,b,c,d,e,f) hipsolverDnDgetrf_bufferSize((a),(b),(c),(d),(e),(f))
#define hipsolverDnXgetrs(a,b,c,d,e,f,g,h,i,j)    hipsolverDnDgetrs((a),(b),(c),(d),(e),(f),(g),(h),(i),(j))
#endif
#endif

/* copy and pasted from the CUBLAS implementation */
/* Where to place the stream ? */
/*
hipStream_t stream;
ccer = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);CHKERRCUDA(ccer);
cerr = hipsolverSetStream(handle,stream);CHKERRCUSOLVER(cerr);
ccer = hipStreamDestroy(stream);CHKERRCUDA(ccer);
*/
#define CHKERRCUSOLVER(err) do {if (PetscUnlikely(err)) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSOLVER error %d",err);} while(0)
static PetscErrorCode PetscCUSOLVERDnDestroyHandle();
static PetscErrorCode PetscCUSOLVERDnGetHandle_Private(hipsolverHandle_t **handle)
{
  static hipsolverHandle_t cusolverdnhandle = NULL;
  hipsolverStatus_t          cerr;
  PetscErrorCode            ierr;

  PetscFunctionBegin;
  if (!cusolverdnhandle) {
    cerr = hipsolverDnCreate(&cusolverdnhandle);CHKERRCUSOLVER(cerr);
    ierr = PetscRegisterFinalize(PetscCUSOLVERDnDestroyHandle);CHKERRQ(ierr);
  }
  *handle = &cusolverdnhandle;
  PetscFunctionReturn(0);
}

PetscErrorCode PetscCUSOLVERDnInitializeHandle(void)
{
  hipsolverHandle_t *p_cusolverdnhandle;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = PetscCUSOLVERDnGetHandle_Private(&p_cusolverdnhandle);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode PetscCUSOLVERDnGetHandle(hipsolverHandle_t *handle)
{
  hipsolverHandle_t *p_cusolverdnhandle;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr    = PetscCUSOLVERDnGetHandle_Private(&p_cusolverdnhandle);CHKERRQ(ierr);
  *handle = *p_cusolverdnhandle;
  PetscFunctionReturn(0);
}

PetscErrorCode PetscCUSOLVERDnDestroyHandle()
{
  hipsolverHandle_t *p_cusolverdnhandle;
  hipsolverStatus_t cerr;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscCUSOLVERDnGetHandle_Private(&p_cusolverdnhandle);CHKERRQ(ierr);
  cerr = hipsolverDnDestroy(*p_cusolverdnhandle);CHKERRCUSOLVER(cerr);
  *p_cusolverdnhandle = NULL;  /* Ensures proper reinitialization */
  PetscFunctionReturn(0);
}

//TODO
//static PetscErrorCode MatSeqDenseSymmetrize_Private(Mat A, PetscBool hermitian)
//{
//  Mat_SeqDense *mat = (Mat_SeqDense*)A->data;
//  PetscInt      j, k, n = A->rmap->n;
//
//  PetscFunctionBegin;
//  if (A->rmap->n != A->cmap->n) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_SUP,"Cannot symmetrize a rectangular matrix");
//  if (!hermitian) {
//    for (k=0;k<n;k++) {
//      for (j=k;j<n;j++) {
//        mat->v[j*mat->lda + k] = mat->v[k*mat->lda + j];
//      }
//    }
//  } else {
//    for (k=0;k<n;k++) {
//      for (j=k;j<n;j++) {
//        mat->v[j*mat->lda + k] = PetscConj(mat->v[k*mat->lda + j]);
//      }
//    }
//  }
//  PetscFunctionReturn(0);
//}
//
//PETSC_EXTERN PetscErrorCode MatSeqDenseInvertFactors_Private(Mat A)
//{
//#if defined(PETSC_MISSING_LAPACK_POTRF)
//  PetscFunctionBegin;
//  SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"POTRF - Lapack routine is unavailable.");
//#else
//  Mat_SeqDense   *mat = (Mat_SeqDense*)A->data;
//  PetscErrorCode ierr;
//  PetscBLASInt   info,n;
//
//  PetscFunctionBegin;
//  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
//  ierr = PetscBLASIntCast(A->cmap->n,&n);CHKERRQ(ierr);
//  if (A->factortype == MAT_FACTOR_LU) {
//    if (!mat->pivots) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Pivots not present");
//    if (!mat->fwork) {
//      mat->lfwork = n;
//      ierr = PetscMalloc1(mat->lfwork,&mat->fwork);CHKERRQ(ierr);
//      ierr = PetscLogObjectMemory((PetscObject)A,mat->lfwork*sizeof(PetscBLASInt));CHKERRQ(ierr);
//    }
//    ierr = PetscFPTrapPush(PETSC_FP_TRAP_OFF);CHKERRQ(ierr);
//    PetscStackCallBLAS("LAPACKgetri",LAPACKgetri_(&n,mat->v,&mat->lda,mat->pivots,mat->fwork,&mat->lfwork,&info));
//    ierr = PetscFPTrapPop();CHKERRQ(ierr);
//    ierr = PetscLogFlops((1.0*A->cmap->n*A->cmap->n*A->cmap->n)/3.0);CHKERRQ(ierr); /* TODO CHECK FLOPS */
//  } else if (A->factortype == MAT_FACTOR_CHOLESKY) {
//    if (A->spd) {
//      ierr = PetscFPTrapPush(PETSC_FP_TRAP_OFF);CHKERRQ(ierr);
//      PetscStackCallBLAS("LAPACKpotri",LAPACKpotri_("L",&n,mat->v,&mat->lda,&info));
//      ierr = PetscFPTrapPop();CHKERRQ(ierr);
//      ierr = MatSeqDenseSymmetrize_Private(A,PETSC_TRUE);CHKERRQ(ierr);
//#if defined(PETSC_USE_COMPLEX)
//    } else if (A->hermitian) {
//      if (!mat->pivots) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Pivots not present");
//      if (!mat->fwork) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Fwork not present");
//      ierr = PetscFPTrapPush(PETSC_FP_TRAP_OFF);CHKERRQ(ierr);
//      PetscStackCallBLAS("LAPACKhetri",LAPACKhetri_("L",&n,mat->v,&mat->lda,mat->pivots,mat->fwork,&info));
//      ierr = PetscFPTrapPop();CHKERRQ(ierr);
//      ierr = MatSeqDenseSymmetrize_Private(A,PETSC_TRUE);CHKERRQ(ierr);
//#endif
//    } else { /* symmetric case */
//      if (!mat->pivots) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Pivots not present");
//      if (!mat->fwork) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Fwork not present");
//      ierr = PetscFPTrapPush(PETSC_FP_TRAP_OFF);CHKERRQ(ierr);
//      PetscStackCallBLAS("LAPACKsytri",LAPACKsytri_("L",&n,mat->v,&mat->lda,mat->pivots,mat->fwork,&info));
//      ierr = PetscFPTrapPop();CHKERRQ(ierr);
//      ierr = MatSeqDenseSymmetrize_Private(A,PETSC_FALSE);CHKERRQ(ierr);
//    }
//    if (info) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad Inversion: zero pivot in row %D",(PetscInt)info-1);
//    ierr = PetscLogFlops((1.0*A->cmap->n*A->cmap->n*A->cmap->n)/3.0);CHKERRQ(ierr); /* TODO CHECK FLOPS */
//  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
//#endif
//
//  A->ops->solve             = NULL;
//  A->ops->matsolve          = NULL;
//  A->ops->solvetranspose    = NULL;
//  A->ops->matsolvetranspose = NULL;
//  A->ops->solveadd          = NULL;
//  A->ops->solvetransposeadd = NULL;
//  A->factortype             = MAT_FACTOR_NONE;
//  ierr                      = PetscFree(A->solvertype);CHKERRQ(ierr);
//  PetscFunctionReturn(0);
//}


#if 0
/* ------------------------------------------------------------------*/
static PetscErrorCode MatSOR_SeqDense(Mat A,Vec bb,PetscReal omega,MatSORType flag,PetscReal shift,PetscInt its,PetscInt lits,Vec xx)
{
  Mat_SeqDense      *mat = (Mat_SeqDense*)A->data;
  PetscScalar       *x,*v = mat->v,zero = 0.0,xt;
  const PetscScalar *b;
  PetscErrorCode    ierr;
  PetscInt          m = A->rmap->n,i;
  PetscBLASInt      o = 1,bm;

  PetscFunctionBegin;
  if (shift == -1) shift = 0.0; /* negative shift indicates do not error on zero diagonal; this code never zeros on zero diagonal */
  ierr = PetscBLASIntCast(m,&bm);CHKERRQ(ierr);
  if (flag & SOR_ZERO_INITIAL_GUESS) {
    /* this is a hack fix, should have another version without the second BLASdotu */
    ierr = VecSet(xx,zero);CHKERRQ(ierr);
  }
  ierr = VecGetArray(xx,&x);CHKERRQ(ierr);
  ierr = VecGetArrayRead(bb,&b);CHKERRQ(ierr);
  its  = its*lits;
  if (its <= 0) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Relaxation requires global its %D and local its %D both positive",its,lits);
  while (its--) {
    if (flag & SOR_FORWARD_SWEEP || flag & SOR_LOCAL_FORWARD_SWEEP) {
      for (i=0; i<m; i++) {
        PetscStackCallBLAS("BLASdotu",xt   = b[i] - BLASdotu_(&bm,v+i,&bm,x,&o));
        x[i] = (1. - omega)*x[i] + omega*(xt+v[i + i*m]*x[i])/(v[i + i*m]+shift);
      }
    }
    if (flag & SOR_BACKWARD_SWEEP || flag & SOR_LOCAL_BACKWARD_SWEEP) {
      for (i=m-1; i>=0; i--) {
        PetscStackCallBLAS("BLASdotu",xt   = b[i] - BLASdotu_(&bm,v+i,&bm,x,&o));
        x[i] = (1. - omega)*x[i] + omega*(xt+v[i + i*m]*x[i])/(v[i + i*m]+shift);
      }
    }
  }
  ierr = VecRestoreArrayRead(bb,&b);CHKERRQ(ierr);
  ierr = VecRestoreArray(xx,&x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDensePlaceArray_SeqDense(Mat A,const PetscScalar array[])
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  a->unplacedarray       = a->v;
  a->unplaced_user_alloc = a->user_alloc;
  a->v                   = (PetscScalar*) array;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseResetArray_SeqDense(Mat A)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  a->v             = a->unplacedarray;
  a->user_alloc    = a->unplaced_user_alloc;
  a->unplacedarray = NULL;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatGetRowMax_SeqDense(Mat A,Vec v,PetscInt idx[])
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;
  PetscInt       i,j,m = A->rmap->n,n = A->cmap->n,p;
  PetscScalar    *x;
  MatScalar      *aa = a->v;

  PetscFunctionBegin;
  if (A->factortype) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Not for factored matrix");

  ierr = VecSet(v,0.0);CHKERRQ(ierr);
  ierr = VecGetArray(v,&x);CHKERRQ(ierr);
  ierr = VecGetLocalSize(v,&p);CHKERRQ(ierr);
  if (p != A->rmap->n) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Nonconforming matrix and vector");
  for (i=0; i<m; i++) {
    x[i] = aa[i]; if (idx) idx[i] = 0;
    for (j=1; j<n; j++) {
      if (PetscRealPart(x[i]) < PetscRealPart(aa[i+m*j])) {x[i] = aa[i + m*j]; if (idx) idx[i] = j;}
    }
  }
  ierr = VecRestoreArray(v,&x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatGetRowMaxAbs_SeqDense(Mat A,Vec v,PetscInt idx[])
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;
  PetscInt       i,j,m = A->rmap->n,n = A->cmap->n,p;
  PetscScalar    *x;
  PetscReal      atmp;
  MatScalar      *aa = a->v;

  PetscFunctionBegin;
  if (A->factortype) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Not for factored matrix");

  ierr = VecSet(v,0.0);CHKERRQ(ierr);
  ierr = VecGetArray(v,&x);CHKERRQ(ierr);
  ierr = VecGetLocalSize(v,&p);CHKERRQ(ierr);
  if (p != A->rmap->n) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Nonconforming matrix and vector");
  for (i=0; i<m; i++) {
    x[i] = PetscAbsScalar(aa[i]);
    for (j=1; j<n; j++) {
      atmp = PetscAbsScalar(aa[i+m*j]);
      if (PetscAbsScalar(x[i]) < atmp) {x[i] = atmp; if (idx) idx[i] = j;}
    }
  }
  ierr = VecRestoreArray(v,&x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatGetRowMin_SeqDense(Mat A,Vec v,PetscInt idx[])
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;
  PetscInt       i,j,m = A->rmap->n,n = A->cmap->n,p;
  PetscScalar    *x;
  MatScalar      *aa = a->v;

  PetscFunctionBegin;
  if (A->factortype) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Not for factored matrix");

  ierr = VecSet(v,0.0);CHKERRQ(ierr);
  ierr = VecGetArray(v,&x);CHKERRQ(ierr);
  ierr = VecGetLocalSize(v,&p);CHKERRQ(ierr);
  if (p != A->rmap->n) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_SIZ,"Nonconforming matrix and vector");
  for (i=0; i<m; i++) {
    x[i] = aa[i]; if (idx) idx[i] = 0;
    for (j=1; j<n; j++) {
      if (PetscRealPart(x[i]) > PetscRealPart(aa[i+m*j])) {x[i] = aa[i + m*j]; if (idx) idx[i] = j;}
    }
  }
  ierr = VecRestoreArray(v,&x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatGetColumnVector_SeqDense(Mat A,Vec v,PetscInt col)
{
  Mat_SeqDense   *a = (Mat_SeqDense*)A->data;
  PetscErrorCode ierr;
  PetscScalar    *x;

  PetscFunctionBegin;
  if (A->factortype) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Not for factored matrix");

  ierr = VecGetArray(v,&x);CHKERRQ(ierr);
  ierr = PetscArraycpy(x,a->v+col*a->lda,A->rmap->n);CHKERRQ(ierr);
  ierr = VecRestoreArray(v,&x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatGetColumnNorms_SeqDense(Mat A,NormType type,PetscReal *norms)
{
  PetscErrorCode    ierr;
  PetscInt          i,j,m,n;
  const PetscScalar *a;

  PetscFunctionBegin;
  ierr = MatGetSize(A,&m,&n);CHKERRQ(ierr);
  ierr = PetscArrayzero(norms,n);CHKERRQ(ierr);
  ierr = MatDenseGetArrayRead(A,&a);CHKERRQ(ierr);
  if (type == NORM_2) {
    for (i=0; i<n; i++) {
      for (j=0; j<m; j++) {
        norms[i] += PetscAbsScalar(a[j]*a[j]);
      }
      a += m;
    }
  } else if (type == NORM_1) {
    for (i=0; i<n; i++) {
      for (j=0; j<m; j++) {
        norms[i] += PetscAbsScalar(a[j]);
      }
      a += m;
    }
  } else if (type == NORM_INFINITY) {
    for (i=0; i<n; i++) {
      for (j=0; j<m; j++) {
        norms[i] = PetscMax(PetscAbsScalar(a[j]),norms[i]);
      }
      a += m;
    }
  } else SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_ARG_WRONG,"Unknown NormType");
  ierr = MatDenseRestoreArrayRead(A,&a);CHKERRQ(ierr);
  if (type == NORM_2) {
    for (i=0; i<n; i++) norms[i] = PetscSqrtReal(norms[i]);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseGetColumn_SeqDense(Mat A,PetscInt col,PetscScalar **vals)
{
  Mat_SeqDense *a = (Mat_SeqDense*)A->data;

  PetscFunctionBegin;
  if (A->factortype) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Not for factored matrix");
  *vals = a->v+col*a->lda;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatDenseRestoreColumn_SeqDense(Mat A,PetscScalar **vals)
{
  PetscFunctionBegin;
  *vals = 0; /* user cannot accidently use the array later */
  PetscFunctionReturn(0);
}

static struct _MatOps MatOps_Values = { MatSetValues_SeqDense,
                                        MatGetRow_SeqDense,
                                        MatRestoreRow_SeqDense,
                                        MatMult_SeqDense,
                                /*  4*/ MatMultAdd_SeqDense,
                                        MatMultTranspose_SeqDense,
                                        MatMultTransposeAdd_SeqDense,
                                        0,
                                        0,
                                        0,
                                /* 10*/ 0,
                                        MatLUFactor_SeqDense,
                                        MatCholeskyFactor_SeqDense,
                                        MatSOR_SeqDense,
                                        MatTranspose_SeqDense,
                                /* 15*/ MatGetInfo_SeqDense,
                                        MatEqual_SeqDense,
                                        MatGetDiagonal_SeqDense,
                                        MatDiagonalScale_SeqDense,
                                        MatNorm_SeqDense,
                                /* 20*/ MatAssemblyBegin_SeqDense,
                                        MatAssemblyEnd_SeqDense,
                                        MatSetOption_SeqDense,
                                        MatZeroEntries_SeqDense,
                                /* 24*/ MatZeroRows_SeqDense,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 29*/ MatSetUp_SeqDense,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 34*/ MatDuplicate_SeqDense,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 39*/ MatAXPY_SeqDense,
                                        MatCreateSubMatrices_SeqDense,
                                        0,
                                        MatGetValues_SeqDense,
                                        MatCopy_SeqDense,
                                /* 44*/ MatGetRowMax_SeqDense,
                                        MatScale_SeqDense,
                                        MatShift_Basic,
                                        0,
                                        MatZeroRowsColumns_SeqDense,
                                /* 49*/ MatSetRandom_SeqDense,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 54*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 59*/ 0,
                                        MatDestroy_SeqDense,
                                        MatView_SeqDense,
                                        0,
                                        0,
                                /* 64*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 69*/ MatGetRowMaxAbs_SeqDense,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 74*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /* 79*/ 0,
                                        0,
                                        0,
                                        0,
                                /* 83*/ MatLoad_SeqDense,
                                        0,
                                        MatIsHermitian_SeqDense,
                                        0,
                                        0,
                                        0,
                                /* 89*/ MatMatMult_SeqDense_SeqDense,
                                        MatMatMultSymbolic_SeqDense_SeqDense,
                                        MatMatMultNumeric_SeqDense_SeqDense,
                                        MatPtAP_SeqDense_SeqDense,
                                        MatPtAPSymbolic_SeqDense_SeqDense,
                                /* 94*/ MatPtAPNumeric_SeqDense_SeqDense,
                                        MatMatTransposeMult_SeqDense_SeqDense,
                                        MatMatTransposeMultSymbolic_SeqDense_SeqDense,
                                        MatMatTransposeMultNumeric_SeqDense_SeqDense,
                                        0,
                                /* 99*/ 0,
                                        0,
                                        0,
                                        MatConjugate_SeqDense,
                                        0,
                                /*104*/ 0,
                                        MatRealPart_SeqDense,
                                        MatImaginaryPart_SeqDense,
                                        0,
                                        0,
                                /*109*/ 0,
                                        0,
                                        MatGetRowMin_SeqDense,
                                        MatGetColumnVector_SeqDense,
                                        MatMissingDiagonal_SeqDense,
                                /*114*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /*119*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /*124*/ 0,
                                        MatGetColumnNorms_SeqDense,
                                        0,
                                        0,
                                        0,
                                /*129*/ 0,
                                        MatTransposeMatMult_SeqDense_SeqDense,
                                        MatTransposeMatMultSymbolic_SeqDense_SeqDense,
                                        MatTransposeMatMultNumeric_SeqDense_SeqDense,
                                        0,
                                /*134*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /*139*/ 0,
                                        0,
                                        0,
                                        0,
                                        0,
                                /*144*/ MatCreateMPIMatConcatenateSeqMat_SeqDense
};
#endif
///*@C
//   MatCreateSeqDenseCUDA - Creates a sequential dense matrix that
//   is stored in column major order (the usual Fortran 77 manner). Many
//   of the matrix operations use the BLAS and LAPACK routines.
//
//   Collective
//
//   Input Parameters:
//+  comm - MPI communicator, set to PETSC_COMM_SELF
//.  m - number of rows
//.  n - number of columns
//-  data - optional location of matrix data in column major order.  Set data=NULL for PETSc
//   to control all matrix memory allocation.
//
//   Output Parameter:
//.  A - the matrix
//
//   Notes:
//   The data input variable is intended primarily for Fortran programmers
//   who wish to allocate their own matrix memory space.  Most users should
//   set data=NULL.
//
//   Level: intermediate
//
//.seealso: MatCreate(), MatCreateDense(), MatSetValues()
//@*/
//PetscErrorCode  MatCreateSeqDenseCUDA(MPI_Comm comm,PetscInt m,PetscInt n,PetscScalar *data,Mat *A)
//{
//  PetscErrorCode ierr;
//
//  PetscFunctionBegin;
//  ierr = MatCreate(comm,A);CHKERRQ(ierr);
//  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
//  ierr = MatSetType(*A,MATSEQDENSECUDA);CHKERRQ(ierr);
//  ierr = MatSeqDenseSetPreallocation(*A,data);CHKERRQ(ierr);
//  PetscFunctionReturn(0);
//}

//PetscErrorCode MatSeqDenseSetPreallocation_SeqDense(Mat B,PetscScalar *data)
//{
//  Mat_SeqDense   *b;
//  PetscErrorCode ierr;
//
//  PetscFunctionBegin;
//  B->preallocated = PETSC_TRUE;
//
//  ierr = PetscLayoutSetUp(B->rmap);CHKERRQ(ierr);
//  ierr = PetscLayoutSetUp(B->cmap);CHKERRQ(ierr);
//
//  b       = (Mat_SeqDense*)B->data;
//  b->Mmax = B->rmap->n;
//  b->Nmax = B->cmap->n;
//  if (b->lda <= 0 || b->changelda) b->lda = B->rmap->n;
//
//  ierr = PetscIntMultError(b->lda,b->Nmax,NULL);CHKERRQ(ierr);
//  if (!data) { /* petsc-allocated storage */
//    if (!b->user_alloc) { ierr = PetscFree(b->v);CHKERRQ(ierr); }
//    ierr = PetscCalloc1((size_t)b->lda*b->Nmax,&b->v);CHKERRQ(ierr);
//    ierr = PetscLogObjectMemory((PetscObject)B,b->lda*b->Nmax*sizeof(PetscScalar));CHKERRQ(ierr);
//
//    b->user_alloc = PETSC_FALSE;
//  } else { /* user-allocated storage */
//    if (!b->user_alloc) { ierr = PetscFree(b->v);CHKERRQ(ierr); }
//    b->v          = data;
//    b->user_alloc = PETSC_TRUE;
//  }
//  B->assembled = PETSC_TRUE;
//  PetscFunctionReturn(0);
//}

typedef struct {
  PetscScalar *d_v;   /* pointer to the matrix on the GPU */
  /* factorization support */
  int         *d_fact_ipiv; /* device pivots */
  PetscScalar *d_fact_work; /* device workspace */
  int         fact_lwork;
  int         *d_fact_info; /* device info */
} Mat_SeqDenseCUDA;

PetscErrorCode MatSeqDenseCUDACopyFromGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  hipError_t      cerr;

  PetscFunctionBegin;
  ierr = PetscInfo3(A,"%s matrix %d x %d\n",A->valid_GPU_matrix == PETSC_OFFLOAD_GPU ? "Copy" : "Reusing",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_GPU) {
    ierr = PetscLogEventBegin(MAT_DenseCopyFromGPU,A,0,0,0);CHKERRQ(ierr);
    /* TODO, no lda? */
    cerr = hipMemcpy(cA->v,dA->d_v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyDeviceToHost);CHKERRCUDA(cerr);
    ierr = PetscLogGpuToCpu(cA->lda*sizeof(PetscScalar)*A->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_DenseCopyFromGPU,A,0,0,0);CHKERRQ(ierr);

    A->valid_GPU_matrix = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseCUDACopyToGPU(Mat A)
{
  Mat_SeqDense     *cA = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;
  hipError_t      cerr;

  PetscFunctionBegin;
  if (A->pinnedtocpu) PetscFunctionReturn(0);
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_UNALLOCATED) SETERRQ(PetscObjectComm((PetscObject)A),PETSC_ERR_ARG_WRONGSTATE,"Unallocated device memory");
  ierr = PetscInfo3(A,"%s matrix %d x %d\n",A->valid_GPU_matrix == PETSC_OFFLOAD_CPU ? "Copy" : "Reusing",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  if (A->valid_GPU_matrix == PETSC_OFFLOAD_CPU) {
    ierr = PetscLogEventBegin(MAT_DenseCopyToGPU,A,0,0,0);CHKERRQ(ierr);
    /* TODO, no lda? */
    cerr = hipMemcpy(dA->d_v,cA->v,cA->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyHostToDevice);CHKERRCUDA(cerr);
    ierr = PetscLogCpuToGpu(cA->lda*sizeof(PetscScalar)*A->cmap->n);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_DenseCopyToGPU,A,0,0,0);CHKERRQ(ierr);

    A->valid_GPU_matrix = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseCUDAGetArrayWrite(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;

  PetscFunctionBegin;
  *a = dA->d_v;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseCUDARestoreArrayWrite(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  *a = NULL;
  A->valid_GPU_matrix = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseCUDAGetArrayRead(Mat A, const PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseCUDARestoreArrayRead(Mat A, const PetscScalar **a)
{
  PetscFunctionBegin;
  *a = NULL;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseCUDAGetArray(Mat A, PetscScalar **a)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  *a   = dA->d_v;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseCUDARestoreArray(Mat A, PetscScalar **a)
{
  PetscFunctionBegin;
  *a = NULL;
  A->valid_GPU_matrix = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(0);
}

static PetscErrorCode MatMatSolve_SeqDenseCUDA(Mat A,Mat B,Mat X)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDense       *x = (Mat_SeqDense*)X->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscScalar        *dx;
  hipsolverHandle_t handle;
  PetscMPIInt        nrhs,info,n,lda,ldx;
  hipsolverStatus_t   cerr;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  if (!dA->d_fact_work) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  ierr = MatCopy(B,X,SAME_NONZERO_PATTERN);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  /* TODO: MatMatSolve does not have a dispatching mechanism, we may end up with a MATSEQDENSE here */
  ierr = MatDenseCUDAGetArrayWrite(X,&dx);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(X->cmap->n,&nrhs);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(x->lda,&ldx);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (A->factortype == MAT_FACTOR_LU) {
    ierr = PetscInfo2(A,"LU solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    cerr = hipsolverDnXgetrs(handle,HIPBLAS_OP_N,n,nrhs,da,lda,dA->d_fact_ipiv,dx,ldx,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  } else if (A->factortype == MAT_FACTOR_CHOLESKY) {
    ierr = PetscInfo2(A,"Cholesky solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    if (!dA->d_fact_ipiv) { /* spd */
      /* ========= Program hit hipErrorNotReady (error 34) due to "device not ready" on CUDA API call to hipEventQuery. */
      cerr = hipsolverDnXpotrs(handle,HIPBLAS_FILL_MODE_LOWER,n,nrhs,da,lda,dx,ldx,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytrs not implemented");
  } else SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Unknown factor type %d",A->factortype);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(X,&dx);CHKERRQ(ierr);
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  ierr = PetscLogGpuFlops(nrhs*(2.0*n*n - n));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA_Private(Mat A,Vec xx,Vec yy,PetscBool trans)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  const PetscScalar  *da;
  PetscScalar        *y;
  hipsolverHandle_t handle;
  PetscMPIInt        one = 1,info,n,lda;
  hipsolverStatus_t   cerr;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (A->factortype == MAT_FACTOR_NONE) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  if (!dA->d_fact_work) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,"Matrix must be factored to solve");
  ierr = VecCopy(xx,yy);CHKERRQ(ierr);
  /* may not be of type cuda */
  ierr = VecCUDAGetArrayWrite(yy,&y);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (A->factortype == MAT_FACTOR_LU) {
    ierr = PetscInfo2(A,"LU solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    cerr = hipsolverDnXgetrs(handle,trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,n,one,da,lda,dA->d_fact_ipiv,y,n,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  } else if (A->factortype == MAT_FACTOR_CHOLESKY) {
    ierr = PetscInfo2(A,"Cholesky solve %d x %d on backend\n",n,n);CHKERRQ(ierr);
    if (!dA->d_fact_ipiv) { /* spd */
      /* ========= Program hit hipErrorNotReady (error 34) due to "device not ready" on CUDA API call to hipEventQuery. */
      cerr = hipsolverDnXpotrs(handle,HIPBLAS_FILL_MODE_LOWER,n,one,da,lda,y,n,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"cusolverDnsytrs not implemented");
  } else SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Unknown factor type %d",A->factortype);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = VecCUDARestoreArrayWrite(yy,&y);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  ierr = PetscLogGpuFlops(2.0*n*n - n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolve_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Private(A,xx,yy,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatSolveTranspose_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatSolve_SeqDenseCUDA_Private(A,xx,yy,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatLUFactor_SeqDenseCUDA(Mat A,IS rperm,IS cperm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscMPIInt        m,n,lda,info;
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
  ierr = PetscInfo2(A,"LU factor %d x %d on backend\n",m,n);CHKERRQ(ierr);
  if (!dA->d_fact_ipiv) {
    ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
  }
  if (!dA->fact_lwork) {
    cerr = hipsolverDnXgetrf_bufferSize(handle,m,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
    ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
  }
  if (!dA->d_fact_info) {
    ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
  }
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  cerr = hipsolverDnXgetrf(handle,m,n,da,lda,dA->d_fact_work,dA->d_fact_ipiv,dA->d_fact_info);CHKERRCUSOLVER(cerr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
  ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
  if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_LU_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
  else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
  A->factortype = MAT_FACTOR_LU;
  ierr = PetscLogGpuFlops(2.0*n*n*m/3.0);CHKERRQ(ierr);

  A->ops->solve          = MatSolve_SeqDenseCUDA;
  A->ops->solvetranspose = MatSolveTranspose_SeqDenseCUDA;
  A->ops->matsolve       = MatMatSolve_SeqDenseCUDA;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatCholeskyFactor_SeqDenseCUDA(Mat A,IS perm,const MatFactorInfo *factinfo)
{
  Mat_SeqDense       *a = (Mat_SeqDense*)A->data;
  Mat_SeqDenseCUDA   *dA = (Mat_SeqDenseCUDA*)A->spptr;
  PetscScalar        *da;
  PetscMPIInt        n,lda,info;
  hipsolverStatus_t   cerr;
  hipsolverHandle_t handle;
  hipError_t        ccer;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  if (!A->rmap->n || !A->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUSOLVERDnGetHandle(&handle);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&n);CHKERRQ(ierr);
  ierr = PetscInfo2(A,"Cholesky factor %d x %d on backend\n",n,n);CHKERRQ(ierr);
  if (A->spd) {
    ierr = MatDenseCUDAGetArray(A,&da);CHKERRQ(ierr);
    ierr = PetscMPIIntCast(a->lda,&lda);CHKERRQ(ierr);
    if (!dA->fact_lwork) {
      cerr = hipsolverDnXpotrf_bufferSize(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
      ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
    }
    if (!dA->d_fact_info) {
      ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
    }
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cerr = hipsolverDnXpotrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);

    ierr = WaitForGPU();CHKERRCUDA(ierr);
    ierr = MatDenseCUDARestoreArray(A,&da);CHKERRQ(ierr);
    ccer = hipMemcpy(&info, dA->d_fact_info, sizeof(int), hipMemcpyDeviceToHost);CHKERRCUDA(ccer);
    if (info > 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_MAT_CH_ZRPVT,"Bad factorization: zero pivot in row %d",info-1);
    else if (info < 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Wrong argument to cuSolver %d",-info);
    A->factortype = MAT_FACTOR_CHOLESKY;
    ierr = PetscLogGpuFlops(1.0*n*n*n/3.0);CHKERRQ(ierr);
  } else { /* symmetric case */
#if 0
    /* at the time of writing this interface (cuda 10.0), cusolverDn does not implement *sytrs and *hetr* routines
       instead of erroring , we implement these factorizations using *getr*
       The code below works, and it can be activated when *sytrs routines will be available */
    if (!dA->d_fact_ipiv) {
      ccer = hipMalloc((void**)&dA->d_fact_ipiv,n*sizeof(*dA->d_fact_ipiv));CHKERRCUDA(ccer);
    }
    if (!dA->fact_lwork) {
      cerr = cusolverDnXsytrf_bufferSize(handle,n,da,lda,&dA->fact_lwork);CHKERRCUSOLVER(cerr);
      ccer = hipMalloc((void**)&dA->d_fact_work,dA->fact_lwork*sizeof(*dA->d_fact_work));CHKERRCUDA(ccer);
    }
    if (!dA->d_fact_info) {
      ccer = hipMalloc((void**)&dA->d_fact_info,sizeof(*dA->d_fact_info));CHKERRCUDA(ccer);
    }
    ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
    cerr = cusolverDnXsytrf(handle,HIPBLAS_FILL_MODE_LOWER,n,da,lda,dA->d_fact_ipiv,dA->d_fact_work,dA->fact_lwork,dA->d_fact_info);CHKERRCUSOLVER(cerr);
    ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
#endif
    ierr = MatLUFactor_SeqDenseCUDA(A,perm,perm,factinfo);CHKERRQ(ierr);
  }

  A->ops->solve          = MatSolve_SeqDenseCUDA;
  A->ops->solvetranspose = MatSolveTranspose_SeqDenseCUDA;
  A->ops->matsolve       = MatMatSolve_SeqDenseCUDA;

  ierr = PetscFree(A->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&A->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* GEMM kernel: C = op(A)*op(B), tA, tB flag transposition */
static PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(Mat A,Mat B,Mat C,PetscBool tA, PetscBool tB)
{
  Mat_SeqDense      *a = (Mat_SeqDense*)A->data;
  Mat_SeqDense      *b = (Mat_SeqDense*)B->data;
  Mat_SeqDense      *c = (Mat_SeqDense*)C->data;
  const PetscScalar *da,*db;
  PetscScalar       *dc;
  PetscScalar       one=1.0,zero=0.0;
  PetscMPIInt       m,n,k,alda,blda,clda;
  PetscErrorCode    ierr;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;

  PetscFunctionBegin;
  ierr = PetscMPIIntCast(C->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(C->cmap->n,&n);CHKERRQ(ierr);
  if (tA) {
    ierr = PetscMPIIntCast(A->rmap->n,&k);CHKERRQ(ierr);
  } else {
    ierr = PetscMPIIntCast(A->cmap->n,&k);CHKERRQ(ierr);
  }
  if (!m || !n || !k) PetscFunctionReturn(0);
  ierr = PetscInfo3(C,"Matrix-Matrix product %d x %d x %d on backend\n",m,k,n);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(B,&db);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayWrite(C,&dc);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(a->lda,&alda);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(b->lda,&blda);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(c->lda,&clda);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXgemm(cublasv2handle,tA ? HIPBLAS_OP_T : HIPBLAS_OP_N,tB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,k,&one,da,alda,db,blda,&zero,dc,clda);CHKERRCUBLAS(berr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(1.0*m*n*k + 1.0*m*n*(k-1));CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(B,&db);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayWrite(C,&dc);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_TRUE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA(Mat A,Mat B,Mat C)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA_Private(A,B,C,PETSC_FALSE,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* zz = op(A)*xx + yy
   if yy == NULL, only MatMult */
static PetscErrorCode MatMultAdd_SeqDenseCUDA_Private(Mat A,Vec xx,Vec yy,Vec zz,PetscBool trans)
{
  Mat_SeqDense      *mat = (Mat_SeqDense*)A->data;
  const PetscScalar *xarray,*da;
  PetscScalar       *zarray;
  PetscScalar       one=1.0,zero=0.0;
  PetscMPIInt       m, n, lda; /* Use PetscMPIInt as it is typedef'ed to int */
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  if (yy && yy != zz) { /* mult add */
    ierr = VecCopy_SeqCUDA(yy,zz);CHKERRQ(ierr);
  }
  if (!A->rmap->n || !A->cmap->n) {
    if (!yy) { /* mult only */
      ierr = VecSet_SeqCUDA(zz,0.0);CHKERRQ(ierr);
    }
    PetscFunctionReturn(0);
  }
  ierr = PetscInfo2(A,"Matrix-vector product %d x %d on backend\n",A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(A->cmap->n,&n);CHKERRQ(ierr);
  ierr = PetscMPIIntCast(mat->lda,&lda);CHKERRQ(ierr);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(zz,&zarray);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  berr = cublasXgemv(cublasv2handle,trans ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                     m,n,&one,da,lda,xarray,1,(yy ? &one : &zero),zarray,1);CHKERRCUBLAS(berr);
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = PetscLogGpuFlops(2.0*A->rmap->n*A->cmap->n - (yy ? 0 : A->rmap->n));CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(zz,&zarray);CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTransposeAdd_SeqDenseCUDA(Mat A,Vec xx,Vec yy,Vec zz)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,yy,zz,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMult_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_FALSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatMultTranspose_SeqDenseCUDA(Mat A,Vec xx,Vec yy)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatMultAdd_SeqDenseCUDA_Private(A,xx,NULL,yy,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseGetArrayRead_SeqDenseCUDA(Mat A,const PetscScalar *array[])
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
  ierr = MatDenseGetArray_SeqDense(A,(PetscScalar**)array);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseGetArray_SeqDenseCUDA(Mat A,PetscScalar *array[])
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
  ierr = MatDenseGetArray_SeqDense(A,array);CHKERRQ(ierr);
  A->valid_GPU_matrix = PETSC_OFFLOAD_CPU;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDenseRestoreArray_SeqDenseCUDA(Mat A,PetscScalar *array[])
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatDenseRestoreArray_SeqDense(A,array);CHKERRQ(ierr);
  ierr = MatSeqDenseCUDACopyToGPU(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatAXPY_SeqDenseCUDA(Mat Y,PetscScalar alpha,Mat X,MatStructure str)
{
  Mat_SeqDense      *x = (Mat_SeqDense*)X->data;
  Mat_SeqDense      *y = (Mat_SeqDense*)Y->data;
  const PetscScalar *dx;
  PetscScalar       *dy;
  PetscBLASInt      j,N,m,ldax,lday,one = 1;
  hipblasHandle_t    cublasv2handle;
  hipblasStatus_t    berr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  if (!X->rmap->n || !X->cmap->n) PetscFunctionReturn(0);
  ierr = PetscCUBLASGetHandle(&cublasv2handle);CHKERRQ(ierr);
  ierr = MatDenseCUDAGetArrayRead(X,&dx);CHKERRQ(ierr);
  if (alpha != 0.0) {
    ierr = MatDenseCUDAGetArray(Y,&dy);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDAGetArrayWrite(Y,&dy);CHKERRQ(ierr);
  }
  ierr = PetscBLASIntCast(X->rmap->n*X->cmap->n,&N);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(X->rmap->n,&m);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(x->lda,&ldax);CHKERRQ(ierr);
  ierr = PetscBLASIntCast(y->lda,&lday);CHKERRQ(ierr);
  ierr = PetscInfo2(Y,"Performing AXPY %d x %d on backend\n",Y->rmap->n,Y->cmap->n);CHKERRQ(ierr);
  ierr = PetscLogGpuTimeBegin();CHKERRQ(ierr);
  if (ldax>m || lday>m) {
    for (j=0; j<X->cmap->n; j++) {
      berr = cublasXaxpy(cublasv2handle,m,&alpha,dx+j*ldax,one,dy+j*lday,one);CHKERRCUBLAS(berr);
    }
  } else {
    berr = cublasXaxpy(cublasv2handle,N,&alpha,dx,one,dy,one);CHKERRCUBLAS(berr);
  }
  ierr = PetscLogGpuTimeEnd();CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUDA(ierr);
  ierr = PetscLogGpuFlops(PetscMax(2*N-1,0));CHKERRQ(ierr);
  ierr = MatDenseCUDARestoreArrayRead(X,&dx);CHKERRQ(ierr);
  if (alpha != 0.0) {
    ierr = MatDenseCUDARestoreArray(Y,&dy);CHKERRQ(ierr);
  } else {
    ierr = MatDenseCUDARestoreArrayWrite(Y,&dy);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

static PetscErrorCode MatReset_SeqDenseCUDA(Mat A)
{
  Mat_SeqDenseCUDA *dA = (Mat_SeqDenseCUDA*)A->spptr;
  hipError_t      cerr;
  PetscErrorCode   ierr;

  PetscFunctionBegin;
  if (dA) {
    cerr = hipFree(dA->d_v);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_ipiv);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_info);CHKERRCUDA(cerr);
    cerr = hipFree(dA->d_fact_work);CHKERRCUDA(cerr);
  }
  ierr = PetscFree(A->spptr);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatDestroy_SeqDenseCUDA(Mat A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatReset_SeqDenseCUDA(A);CHKERRQ(ierr);
  ierr = MatDestroy_SeqDense(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode MatSeqDenseSetPreallocation_SeqDenseCUDA(Mat B,PetscScalar *data)
{
  Mat_SeqDense      *b;
  Mat_SeqDenseCUDA* dB;
  hipError_t       cerr;
  PetscErrorCode    ierr;

  PetscFunctionBegin;
  ierr = PetscLayoutSetUp(B->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(B->cmap);CHKERRQ(ierr);
  b       = (Mat_SeqDense*)B->data;
  b->Mmax = B->rmap->n;
  b->Nmax = B->cmap->n;
  if (b->lda <= 0 || b->changelda) b->lda = B->rmap->n;
  if (b->lda < B->rmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Invalid lda %D < %D",b->lda,B->rmap->n);

  ierr = PetscIntMultError(b->lda,b->Nmax,NULL);CHKERRQ(ierr);

  ierr     = MatReset_SeqDenseCUDA(B);CHKERRQ(ierr);
  ierr     = PetscNewLog(B,&dB);CHKERRQ(ierr);
  B->spptr = dB;
  cerr     = hipMalloc((void**)&dB->d_v,b->lda*b->Nmax*sizeof(PetscScalar));CHKERRCUDA(cerr);

  if (!data) { /* petsc-allocated storage */
    if (!b->user_alloc) { ierr = PetscFree(b->v);CHKERRQ(ierr); }
    ierr = PetscCalloc1((size_t)b->lda*b->Nmax,&b->v);CHKERRQ(ierr);
    ierr = PetscLogObjectMemory((PetscObject)B,b->lda*b->Nmax*sizeof(PetscScalar));CHKERRQ(ierr);
    b->user_alloc       = PETSC_FALSE;
  } else { /* user-allocated storage */
    if (!b->user_alloc) { ierr = PetscFree(b->v);CHKERRQ(ierr); }
    b->v                = data;
    b->user_alloc       = PETSC_TRUE;
  }
  B->valid_GPU_matrix = PETSC_OFFLOAD_CPU;
  B->preallocated     = PETSC_TRUE;
  B->assembled        = PETSC_TRUE;
  PetscFunctionReturn(0);
}

PetscErrorCode MatDuplicate_SeqDenseCUDA(Mat A,MatDuplicateOption cpvalues,Mat *B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),B);CHKERRQ(ierr);
  ierr = MatSetSizes(*B,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(*B,((PetscObject)A)->type_name);CHKERRQ(ierr);
  ierr = MatDuplicateNoCreate_SeqDense(*B,A,cpvalues);CHKERRQ(ierr);
  if (cpvalues == MAT_COPY_VALUES && A->valid_GPU_matrix != PETSC_OFFLOAD_CPU) {
    Mat_SeqDense      *a = (Mat_SeqDense*)A->data;
    const PetscScalar *da;
    PetscScalar       *db;
    hipError_t       cerr;

    ierr = MatDenseCUDAGetArrayRead(A,&da);CHKERRQ(ierr);
    ierr = MatDenseCUDAGetArrayWrite(*B,&db);CHKERRQ(ierr);
    if (a->lda > A->rmap->n) {
      PetscInt j,m = A->rmap->n;

      for (j=0; j<A->cmap->n; j++) { /* it can be done better */
        cerr = hipMemcpy(db+j*m,da+j*a->lda,m*sizeof(PetscScalar),hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
      }
    } else {
      cerr = hipMemcpy(db,da,a->lda*sizeof(PetscScalar)*A->cmap->n,hipMemcpyDeviceToDevice);CHKERRCUDA(cerr);
    }
    ierr = MatDenseCUDARestoreArrayRead(A,&da);CHKERRQ(ierr);
    ierr = MatDenseCUDARestoreArrayWrite(*B,&db);CHKERRQ(ierr);
    (*B)->valid_GPU_matrix = PETSC_OFFLOAD_BOTH;
  }
  PetscFunctionReturn(0);
}

PETSC_INTERN PetscErrorCode MatGetFactor_seqdense_cuda(Mat A,MatFactorType ftype,Mat *fact)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(PetscObjectComm((PetscObject)A),fact);CHKERRQ(ierr);
  ierr = MatSetSizes(*fact,A->rmap->n,A->cmap->n,A->rmap->n,A->cmap->n);CHKERRQ(ierr);
  ierr = MatSetType(*fact,MATSEQDENSECUDA);CHKERRQ(ierr);
  if (ftype == MAT_FACTOR_LU) {
    (*fact)->ops->lufactorsymbolic = MatLUFactorSymbolic_SeqDense;
  } else {
    (*fact)->ops->choleskyfactorsymbolic = MatCholeskyFactorSymbolic_SeqDense;
  }
  (*fact)->factortype = ftype;

  ierr = PetscFree((*fact)->solvertype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(MATSOLVERCUDA,&(*fact)->solvertype);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

static PetscErrorCode MatPinToCPU_SeqDenseCUDA(Mat A,PetscBool flg)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  A->pinnedtocpu = flg;
  if (!flg) {
    /* make sure we have an up-to-date copy on the CPU */
    ierr = MatSeqDenseCUDACopyFromGPU(A);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqDenseSetPreallocation_C",MatSeqDenseSetPreallocation_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",           MatDenseGetArray_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",       MatDenseGetArrayRead_SeqDenseCUDA);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreArray_C",       MatDenseRestoreArray_SeqDenseCUDA);CHKERRQ(ierr);

    A->ops->destroy                 = MatDestroy_SeqDenseCUDA;
    A->ops->duplicate               = MatDuplicate_SeqDenseCUDA;
    A->ops->mult                    = MatMult_SeqDenseCUDA;
    A->ops->multadd                 = MatMultAdd_SeqDenseCUDA;
    A->ops->multtranspose           = MatMultTranspose_SeqDenseCUDA;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDenseCUDA;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDenseCUDA_SeqDenseCUDA;
    A->ops->axpy                    = MatAXPY_SeqDenseCUDA;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDenseCUDA;
    A->ops->lufactor                = MatLUFactor_SeqDenseCUDA;
  } else {
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatSeqDenseSetPreallocation_C",MatSeqDenseSetPreallocation_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArray_C",           MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseGetArrayRead_C",       MatDenseGetArray_SeqDense);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunction((PetscObject)A,"MatDenseRestoreArray_C",       MatDenseRestoreArray_SeqDense);CHKERRQ(ierr);

    A->ops->destroy                 = MatDestroy_SeqDense;
    A->ops->duplicate               = MatDuplicate_SeqDense;
    A->ops->mult                    = MatMult_SeqDense;
    A->ops->multadd                 = MatMultAdd_SeqDense;
    A->ops->multtranspose           = MatMultTranspose_SeqDense;
    A->ops->multtransposeadd        = MatMultTransposeAdd_SeqDense;
    A->ops->matmultnumeric          = MatMatMultNumeric_SeqDense_SeqDense;
    A->ops->mattransposemultnumeric = MatMatTransposeMultNumeric_SeqDense_SeqDense;
    A->ops->transposematmultnumeric = MatTransposeMatMultNumeric_SeqDense_SeqDense;
    A->ops->axpy                    = MatAXPY_SeqDense;
    A->ops->choleskyfactor          = MatCholeskyFactor_SeqDense;
    A->ops->lufactor                = MatLUFactor_SeqDense;
 }
  PetscFunctionReturn(0);
}

PetscErrorCode MatConvert_SeqDense_SeqDenseCUDA(Mat B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscFree(B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscStrallocpy(VECCUDA,&B->defaultvectype);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQDENSECUDA);CHKERRQ(ierr);

  ierr = MatPinToCPU_SeqDenseCUDA(B,PETSC_FALSE);CHKERRQ(ierr);
  B->ops->pintocpu = MatPinToCPU_SeqDenseCUDA;
  B->valid_GPU_matrix = PETSC_OFFLOAD_UNALLOCATED;

  PetscFunctionReturn(0);
}

/*MC
   MATSEQDENSECUDA - MATSEQDENSECUDA = "seqdensecuda" - A matrix type to be used for sequential dense matrices on GPUs.

   Options Database Keys:
. -mat_type seqdensecuda - sets the matrix type to "seqdensecuda" during a call to MatSetFromOptions()

  Level: beginner

.seealso: MatCreateSeqDenseCuda()

M*/

PETSC_EXTERN PetscErrorCode MatCreate_SeqDenseCUDA(Mat B)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate_SeqDense(B);CHKERRQ(ierr);
  ierr = MatConvert_SeqDense_SeqDenseCUDA(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
