static char help[] = "Benchmarking hipPointerGetAttributes() time\n";
/*
  Running example on Summit at OLCF:
  # run with total 1 resource set (RS) (-n1), 1 RS per node (-r1), 1 MPI rank (-a1), 7 cores (-c7) and 1 GPU (-g1) per RS
  $ jsrun -n1 -a1 -c7 -g1 -r1  ./ex2cu
    Average hipPointerGetAttributes() time = 0.29 microseconds
*/
#include <petscsys.h>
#include <petscdevice.h>

int main(int argc,char **argv)
{
  PetscInt                     i,n=2000;
  hipError_t                  cerr;
  PetscScalar                  **ptrs;
  PetscLogDouble               tstart,tend,time;
  struct hipPointerAttribute_t attr;

  CHKERRQ(PetscInitialize(&argc,&argv,(char*)0,help));
  CHKERRQ(PetscOptionsGetInt(NULL,NULL,"-n",&n,NULL));

  CHKERRQ(PetscMalloc1(n,&ptrs));
  for (i=0; i<n; i++) {
    if (i%2) CHKERRQ(PetscMalloc1(i+16,&ptrs[i]));
    else CHKERRCUDA(hipMalloc((void**)&ptrs[i],(i+16)*sizeof(PetscScalar)));
  }

  CHKERRQ(PetscTime(&tstart));
  for (i=0; i<n; i++) {
    cerr = hipPointerGetAttributes(&attr,ptrs[i]);
    if (cerr) hipGetLastError();
  }
  CHKERRQ(PetscTime(&tend));
  time = (tend-tstart)*1e6/n;

  CHKERRQ(PetscPrintf(PETSC_COMM_WORLD,"Average hipPointerGetAttributes() time = %.2f microseconds\n",time));

  for (i=0; i<n; i++) {
    if (i%2) CHKERRQ(PetscFree(ptrs[i]));
    else CHKERRCUDA(hipFree(ptrs[i]));
  }
  CHKERRQ(PetscFree(ptrs));

  CHKERRQ(PetscFinalize());
  return 0;
}

/*TEST
  build:
    requires: cuda

  test:
    requires: cuda
    args: -n 2
    output_file: output/empty.out
    filter: grep "DOES_NOT_EXIST"

TEST*/
