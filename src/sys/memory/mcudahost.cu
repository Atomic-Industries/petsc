#include <petscsys.h>             /*I   "petscsys.h"   I*/

static PetscErrorCode PetscCudaHostMalloc(size_t a,PetscBool clear,int lineno,const char function[],const char filename[],void **result)
{
  PetscErrorCode ierr;
  ierr = hipHostMalloc(result,a);CHKERRCUDA(ierr);
  return 0;
}

static PetscErrorCode PetscCudaHostFree(void *aa,int lineno,const char function[],const char filename[])
{
  PetscErrorCode ierr;
  ierr = hipHostFree(aa);CHKERRCUDA(ierr);
  return 0;
}

static PetscErrorCode PetscCudaHostRealloc(size_t a,int lineno,const char function[],const char filename[],void **result)
{
  SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"CUDA has no Realloc()");
  return 0;
}

static PetscErrorCode (*PetscMallocOld)(size_t,PetscBool,int,const char[],const char[],void**);
static PetscErrorCode (*PetscFreeOld)(void*,int,const char[],const char[]);

/*@C
   PetscMallocSetCUDAHost - Set PetscMalloc to use CUDAHostMalloc
     Switch the current malloc and free routines to the CUDA malloc and free routines

   Not Collective

   Level: developer

   Notes:
     This provides a way to use the CUDA malloc and free routines temporarily. One
     can switch back to the previous choice by calling PetscMallocResetCUDAHost().

.seealso: PetscMallocResetCUDAHost()
@*/
PetscErrorCode PetscMallocSetCUDAHost(void)
{
  PetscFunctionBegin;
  /* Save the previous choice */
  PetscMallocOld = PetscTrMalloc;
  PetscFreeOld   = PetscTrFree;
  PetscTrMalloc  = PetscCudaHostMalloc;
  PetscTrFree    = PetscCudaHostFree;
  PetscFunctionReturn(0);
}

/*@C
   PetscMallocResetCUDAHost - Reset the changes made by PetscMallocSetCUDAHost

   Not Collective

   Level: developer

.seealso: PetscMallocSetCUDAHost()
@*/
PetscErrorCode PetscMallocResetCUDAHost(void)
{
  PetscFunctionBegin;
  PetscTrMalloc = PetscMallocOld;
  PetscTrFree   = PetscFreeOld;
  PetscFunctionReturn(0);
}
