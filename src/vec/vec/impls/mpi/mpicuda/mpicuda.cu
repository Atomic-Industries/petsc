#include "hip/hip_runtime.h"

/*
   This file contains routines for parallel CUDA vector operations.
 */
#define PETSC_SKIP_SPINLOCK
#define PETSC_SKIP_CXX_COMPLEX_FIX

#include <petscconf.h>
#include <../src/vec/vec/impls/mpi/pvecimpl.h>   /*I  "petscvec.h"   I*/
#include <../src/vec/vec/impls/seq/seqcuda/cudavecimpl.h>

PetscErrorCode VecDestroy_MPICUDA(Vec v)
{
  PetscErrorCode ierr;
  hipError_t    err;

  PetscFunctionBegin;
  if (((Vec_CUDA*)v->spptr)->array_allocated) {
    err = hipFree(((Vec_CUDA*)v->spptr)->array_allocated);CHKERRCUDA(err);
    ((Vec_CUDA*)v->spptr)->array_allocated = NULL;
  }
  if (((Vec_CUDA*)v->spptr)->stream) {
    err = hipStreamDestroy(((Vec_CUDA*)v->spptr)->stream);CHKERRCUDA(err);
  }
  ierr = PetscFree(v->spptr);CHKERRQ(ierr);
  ierr = VecDestroy_MPI(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecNorm_MPICUDA(Vec xin,NormType type,PetscReal *z)
{
  PetscReal      sum,work = 0.0;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr  = VecNorm_SeqCUDA(xin,NORM_2,&work);
    work *= work;
    ierr  = MPIU_Allreduce(&work,&sum,1,MPIU_REAL,MPIU_SUM,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
    *z    = PetscSqrtReal(sum);
  } else if (type == NORM_1) {
    /* Find the local part */
    ierr = VecNorm_SeqCUDA(xin,NORM_1,&work);CHKERRQ(ierr);
    /* Find the global max */
    ierr = MPIU_Allreduce(&work,z,1,MPIU_REAL,MPIU_SUM,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    /* Find the local max */
    ierr = VecNorm_SeqCUDA(xin,NORM_INFINITY,&work);CHKERRQ(ierr);
    /* Find the global max */
    ierr = MPIU_Allreduce(&work,z,1,MPIU_REAL,MPIU_MAX,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    PetscReal temp[2];
    ierr = VecNorm_SeqCUDA(xin,NORM_1,temp);CHKERRQ(ierr);
    ierr = VecNorm_SeqCUDA(xin,NORM_2,temp+1);CHKERRQ(ierr);
    temp[1] = temp[1]*temp[1];
    ierr = MPIU_Allreduce(temp,z,2,MPIU_REAL,MPIU_SUM,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
    z[1] = PetscSqrtReal(z[1]);
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecDot_MPICUDA(Vec xin,Vec yin,PetscScalar *z)
{
  PetscScalar    sum,work;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecDot_SeqCUDA(xin,yin,&work);CHKERRQ(ierr);
  ierr = MPIU_Allreduce(&work,&sum,1,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
  *z   = sum;
  PetscFunctionReturn(0);
}

PetscErrorCode VecTDot_MPICUDA(Vec xin,Vec yin,PetscScalar *z)
{
  PetscScalar    sum,work;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecTDot_SeqCUDA(xin,yin,&work);CHKERRQ(ierr);
  ierr = MPIU_Allreduce(&work,&sum,1,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
  *z   = sum;
  PetscFunctionReturn(0);
}

PetscErrorCode VecMDot_MPICUDA(Vec xin,PetscInt nv,const Vec y[],PetscScalar *z)
{
  PetscScalar    awork[128],*work = awork;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (nv > 128) {
    ierr = PetscMalloc1(nv,&work);CHKERRQ(ierr);
  }
  ierr = VecMDot_SeqCUDA(xin,nv,y,work);CHKERRQ(ierr);
  ierr = MPIU_Allreduce(work,z,nv,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)xin));CHKERRQ(ierr);
  if (nv > 128) {
    ierr = PetscFree(work);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*MC
   VECMPICUDA - VECMPICUDA = "mpi:cuda" - The basic parallel vector, modified to use CUDA

   Options Database Keys:
. -vec_type mpi:cuda - sets the vector type to VECMPICUDA during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateMPIWithArray(), VECMPI, VecType, VecCreateMPI()
M*/

PETSC_EXTERN PetscErrorCode VecDuplicate_MPI(Vec,Vec*);

PetscErrorCode VecDuplicate_MPICUDA(Vec win,Vec *v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecDuplicate_MPI(win,v);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECMPICUDA);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecDotNorm2_MPICUDA(Vec s,Vec t,PetscScalar *dp,PetscScalar *nm)
{
  PetscErrorCode ierr;
  PetscScalar    work[2],sum[2];

  PetscFunctionBegin;
  ierr = VecDotNorm2_SeqCUDA(s,t,work,work+1);CHKERRQ(ierr);
  ierr = MPIU_Allreduce(&work,&sum,2,MPIU_SCALAR,MPIU_SUM,PetscObjectComm((PetscObject)s));CHKERRQ(ierr);
  *dp  = sum[0];
  *nm  = sum[1];
  PetscFunctionReturn(0);
}

extern "C" PetscErrorCode VecGetArrayWrite_SeqCUDA(Vec,PetscScalar**);

PetscErrorCode VecPinToCPU_MPICUDA(Vec V,PetscBool pin)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  V->pinnedtocpu = pin;
  if (pin) {
    ierr = VecCUDACopyFromGPU(V);CHKERRQ(ierr);
    V->offloadmask = PETSC_OFFLOAD_CPU; /* since the CPU code will likely change values in the vector */
    V->ops->dotnorm2               = NULL;
    V->ops->waxpy                  = VecWAXPY_Seq;
    V->ops->dot                    = VecDot_MPI;
    V->ops->mdot                   = VecMDot_MPI;
    V->ops->tdot                   = VecTDot_MPI;
    V->ops->norm                   = VecNorm_MPI;
    V->ops->scale                  = VecScale_Seq;
    V->ops->copy                   = VecCopy_Seq;
    V->ops->set                    = VecSet_Seq;
    V->ops->swap                   = VecSwap_Seq;
    V->ops->axpy                   = VecAXPY_Seq;
    V->ops->axpby                  = VecAXPBY_Seq;
    V->ops->maxpy                  = VecMAXPY_Seq;
    V->ops->aypx                   = VecAYPX_Seq;
    V->ops->axpbypcz               = VecAXPBYPCZ_Seq;
    V->ops->pointwisemult          = VecPointwiseMult_Seq;
    V->ops->setrandom              = VecSetRandom_Seq;
    V->ops->placearray             = VecPlaceArray_Seq;
    V->ops->replacearray           = VecReplaceArray_Seq;
    V->ops->resetarray             = VecResetArray_Seq;
    V->ops->dot_local              = VecDot_Seq;
    V->ops->tdot_local             = VecTDot_Seq;
    V->ops->norm_local             = VecNorm_Seq;
    V->ops->mdot_local             = VecMDot_Seq;
    V->ops->pointwisedivide        = VecPointwiseDivide_Seq;
    V->ops->getlocalvector         = NULL;
    V->ops->restorelocalvector     = NULL;
    V->ops->getlocalvectorread     = NULL;
    V->ops->restorelocalvectorread = NULL;
    V->ops->getarraywrite          = NULL;
  } else {
    V->ops->dotnorm2               = VecDotNorm2_MPICUDA;
    V->ops->waxpy                  = VecWAXPY_SeqCUDA;
    V->ops->duplicate              = VecDuplicate_MPICUDA;
    V->ops->dot                    = VecDot_MPICUDA;
    V->ops->mdot                   = VecMDot_MPICUDA;
    V->ops->tdot                   = VecTDot_MPICUDA;
    V->ops->norm                   = VecNorm_MPICUDA;
    V->ops->scale                  = VecScale_SeqCUDA;
    V->ops->copy                   = VecCopy_SeqCUDA;
    V->ops->set                    = VecSet_SeqCUDA;
    V->ops->swap                   = VecSwap_SeqCUDA;
    V->ops->axpy                   = VecAXPY_SeqCUDA;
    V->ops->axpby                  = VecAXPBY_SeqCUDA;
    V->ops->maxpy                  = VecMAXPY_SeqCUDA;
    V->ops->aypx                   = VecAYPX_SeqCUDA;
    V->ops->axpbypcz               = VecAXPBYPCZ_SeqCUDA;
    V->ops->pointwisemult          = VecPointwiseMult_SeqCUDA;
    V->ops->setrandom              = VecSetRandom_SeqCUDA;
    V->ops->placearray             = VecPlaceArray_SeqCUDA;
    V->ops->replacearray           = VecReplaceArray_SeqCUDA;
    V->ops->resetarray             = VecResetArray_SeqCUDA;
    V->ops->dot_local              = VecDot_SeqCUDA;
    V->ops->tdot_local             = VecTDot_SeqCUDA;
    V->ops->norm_local             = VecNorm_SeqCUDA;
    V->ops->mdot_local             = VecMDot_SeqCUDA;
    V->ops->destroy                = VecDestroy_MPICUDA;
    V->ops->pointwisedivide        = VecPointwiseDivide_SeqCUDA;
    V->ops->getlocalvector         = VecGetLocalVector_SeqCUDA;
    V->ops->restorelocalvector     = VecRestoreLocalVector_SeqCUDA;
    V->ops->getlocalvectorread     = VecGetLocalVector_SeqCUDA;
    V->ops->restorelocalvectorread = VecRestoreLocalVector_SeqCUDA;
    V->ops->getarraywrite          = VecGetArrayWrite_SeqCUDA;
  }
  PetscFunctionReturn(0);
}

PetscErrorCode VecCreate_MPICUDA(Vec vv)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;

  PetscFunctionBegin;
  if (!vv->data) {ierr = VecSetType(vv,VECMPI ":~");CHKERRQ(ierr);}

  ierr = PetscNewLog(vv,&veccuda);CHKERRQ(ierr);
  vv->spptr = (void*)veccuda;
  veccuda->stream = 0;  /* using default stream */
  veccuda->hostDataRegisteredAsPageLocked = PETSC_FALSE;
  ierr = VecPinToCPU_MPICUDA(vv,PETSC_FALSE);CHKERRQ(ierr);
  vv->ops->pintocpu = VecPinToCPU_MPICUDA;
  ierr = PetscObjectChangeTypeName((PetscObject)vv,VECMPICUDA);CHKERRQ(ierr);

  err = hipMalloc((void**)&veccuda->array_allocated,sizeof(PetscScalar)*((PetscBLASInt)vv->map->n));CHKERRCUDA(err);
  veccuda->array = veccuda->array_allocated;
  ierr = VecSet(vv,0.0);CHKERRQ(ierr);
  vv->offloadmask = (PetscOffloadMask)(vv->offloadmask | PETSC_OFFLOAD_GPU);
  PetscFunctionReturn(0);
}

PetscErrorCode VecCreate_CUDA(Vec v)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MPI_Comm_size(PetscObjectComm((PetscObject)v),&size);CHKERRQ(ierr);
  if (size == 1) {
    ierr = VecSetType(v,VECSEQCUDA);CHKERRQ(ierr);
  } else {
    ierr = VecSetType(v,VECMPICUDA);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*@C
   VecCreateMPICUDAWithArray - Creates a parallel, array-style vector,
   where the user provides the GPU array space to store the vector values.

   Collective

   Input Parameters:
+  comm  - the MPI communicator to use
.  bs    - block size, same meaning as VecSetBlockSize()
.  n     - local vector length, cannot be PETSC_DECIDE
.  N     - global vector length (or PETSC_DECIDE to have calculated)
-  array - the user provided GPU array to store the vector values, must be NULL or contain valid values

   Output Parameter:
.  vv - the vector

   Notes:
   Use VecDuplicate() or VecDuplicateVecs() to form additional vectors of the
   same type as an existing vector.

   If the user-provided array is NULL, then VecCUDAPlaceArray() can be used
   at a later stage to SET the array for storing the vector values.

   PETSc does NOT free the array when the vector is destroyed via VecDestroy().
   The user should not free the array until the vector is destroyed.

   Level: intermediate

.seealso: VecCreateSeqCUDAWithArray(), VecCreateMPIWithArray(), VecCreateSeqWithArray(),
          VecCreate(), VecDuplicate(), VecDuplicateVecs(), VecCreateGhost(),
          VecCreateMPI(), VecCreateGhostWithArray(), VecPlaceArray()

@*/
PetscErrorCode  VecCreateMPICUDAWithArray(MPI_Comm comm,PetscInt bs,PetscInt n,PetscInt N,const PetscScalar array[],Vec *vv)
{
  PetscErrorCode ierr;
  hipError_t    err;
  Vec_CUDA       *veccuda;

  PetscFunctionBegin;
  if (n == PETSC_DECIDE) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"Must set local size of vector");
  ierr = VecCreate(comm,vv);CHKERRQ(ierr);
  ierr = VecSetSizes(*vv,n,N);CHKERRQ(ierr);
  ierr = VecSetBlockSize(*vv,bs);CHKERRQ(ierr);
  ierr = VecSetType(*vv,VECMPICUDA);CHKERRQ(ierr);

  veccuda = (Vec_CUDA*)(*vv)->spptr;
  if (veccuda->array_allocated) {
    err = hipFree(veccuda->array_allocated);CHKERRCUDA(err);
    veccuda->array_allocated = NULL;
  }
  veccuda->array = (PetscScalar*)array;
  if (array) (*vv)->offloadmask = PETSC_OFFLOAD_GPU;
  /* TODO: turn off GPU flag */
  PetscFunctionReturn(0);
}

